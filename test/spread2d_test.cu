#include <iostream>
#include <iomanip>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <complex>
#include <algorithm>
#include "../src/cuspreadinterp.h"
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT upsampfac=2.0;
	int N1, N2, M;
	if (argc<5) {
		fprintf(stderr,
			"Usage: spread2d_test method nupts_distr nf1 nf2 [maxsubprobsize [M [tol [kerevalmeth]]]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven,\n"
			"    2: sub-problem, or\n"
			"    3: sub-problem with Paul's idea.\n"
			"  nupts_distr: The distribution of the points; one of\n"
			"    0: uniform, or\n"
			"    1: concentrated in a small region.\n"
			"  nf1, nf2: The size of the 2D array.\n"
			"  maxsubprobsize: Maximum size of subproblems (default 65536).\n"
			"  M: The number of non-uniform points (default nf1 * nf2 / 4).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n"
			"  kerevalmeth: Kernel evaluation method; one of\n"
			"     0: Exponential of square root (default), or\n"
			"     1: Horner evaluation.\n");
		return 1;
	}
	double w;
	int method;
	sscanf(argv[1],"%d",&method);

	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	int maxsubprobsize=65536;
	if(argc>5){
		sscanf(argv[5],"%d",&maxsubprobsize);
	}

	N1 = (int) nf1/upsampfac;
	N2 = (int) nf2/upsampfac;
	M = N1*N2;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int kerevalmeth=0;
	if(argc>8){
		sscanf(argv[8],"%d",&kerevalmeth);
	}

	int ier;
	int dim=2;

	CUFINUFFT_PLAN dplan = new CUFINUFFT_PLAN_S;
        // Zero out your struct, (sets all pointers to NULL, crucial)
        memset(dplan, 0, sizeof(*dplan));
	ier = CUFINUFFT_DEFAULT_OPTS(1, dim, &(dplan->opts));

	dplan->opts.gpu_method           = method;
	dplan->opts.gpu_maxsubprobsize   = maxsubprobsize;
	dplan->opts.gpu_kerevalmeth      = kerevalmeth;
	dplan->opts.gpu_sort             = 1;   // ahb changed from 0
	dplan->opts.gpu_spreadinterponly = 1;
	dplan->opts.gpu_binsizex         = 32; //binsize needs to be set here, since
                                           //SETUP_BINSIZE() is not called in 
                                           //spread, interp only wrappers.
	dplan->opts.gpu_binsizey         = 32;
	ier = setup_spreader_for_nufft(dplan->spopts, tol, dplan->opts);

	cout<<scientific<<setprecision(3);

	FLT *x, *y;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*sizeof(CPX));

	FLT *d_x, *d_y;
	CUCPX *d_c, *d_fw;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fw,nf1*nf2*sizeof(CUCPX)));

	switch(nupts_distribute){
		// Making data
		case 0: //uniform
			{
				for (int i = 0; i < M; i++) {
					x[i] = M_PI*randm11();// x in [-pi,pi)
					y[i] = M_PI*randm11();
					c[i].real(randm11());
					c[i].imag(randm11());
				}
			}
			break;
		case 1: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = M_PI*rand01()/(nf1*2/32);
					y[i] = M_PI*rand01()/(nf2*2/32);
					c[i].real(randm11());
					c[i].imag(randm11());
				}
			}
			break;
		default:
			cerr << "not valid nupts distr" << endl;
	}

	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c,c,M*sizeof(CUCPX),hipMemcpyHostToDevice));

	CNTime timer;
	/*warm up gpu*/
	timer.restart();
	ier = CUFINUFFT_SPREAD2D(nf1, nf2, d_fw, M, d_x, d_y, d_c, dplan);
	if(ier != 0 ){
		cout<<"error: cnufftspread2d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (%.3g NU pts/s)\n",
			dplan->opts.gpu_method,M,nf1*nf2,t,M/t);

	checkCudaErrors(hipMemcpy(fw,d_fw,nf1*nf2*sizeof(CUCPX),
		hipMemcpyDeviceToHost));
#ifdef RESULT
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % dplan->opts.gpu_binsizey == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % dplan->opts.gpu_binsizex == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fw);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_c);
	hipFree(d_fw);
	return 0;
}
