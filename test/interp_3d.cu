#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/cuspreadinterp.h"
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2, nf3;
	FLT sigma = 2.0;
	int N1, N2, N3, M;
	if (argc<5) {
		fprintf(stderr,
			"Usage: interp3d method nupts_distr nf1 nf2 nf3 [M [tol [sort]]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven, or\n"
			"    2: sub-problem.\n"
			"  nupts_distr: The distribution of the points; one of\n"
			"    0: uniform, or\n"
			"    1: concentrated in a small region.\n"
			"  nf1, nf2, nf3: The size of the 3D array.\n"
			"  M: The number of non-uniform points (default nf1 * nf2 * nf3 / 8).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n"
			"  sort: One of\n"
			"     0: do not sort the points, or\n"
			"     1: sort the points (default).\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[5],"%lf",&w); nf3 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	N3 = (int) nf3/sigma;
	M = N1*N2*N3;// let density always be 1
	if(argc>6){
		sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		if(M == 0) M=N1*N2*N3;
	}

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int sort=1;
	if(argc>8){
		sscanf(argv[8],"%d",&sort);
	}
	int ier;

	int ns=std::ceil(-log10(tol/10.0));
	int maxsubprobsize;

	cout<<scientific<<setprecision(3);


	FLT *x, *y, *z;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*nf3*sizeof(CPX));

	switch(nupts_distribute){
		// Making data
		case 1: //uniform
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*randm11(), nf2, 1);
					z[i] = RESCALE(M_PI*randm11(), nf3, 1);
					//cout << x[i] << "," << y[i] << "," << z[i] << endl;
				}
				maxsubprobsize = 65536;
			}
			break;
		case 2: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf2, 1);
					z[i] = RESCALE(M_PI*rand01()/(nf3*2/32), nf3, 1);
				}
				maxsubprobsize = 1024;
			}
			break;
		default:
			cerr<<"error: nupts distr should be 1,2" << endl;
			return 1;
	}
	for(int i=0; i<nf1*nf2*nf3; i++){
		fw[i].real(1.0);
		fw[i].imag(0.0);
	}

	int dim=3;
	cufinufft_plan dplan;
	ier = cufinufft_default_opts(2, dim, dplan.opts);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	ier = setup_spreader_for_nufft(dplan.spopts, tol, dplan.opts);

	dplan.opts.upsampfac=sigma;
	dplan.opts.gpu_method=method;
	dplan.opts.gpu_kerevalmeth=1;
	dplan.opts.gpu_sort=sort;
	dplan.spopts.pirange=0;
	if(dplan.opts.gpu_method == 2)
	{
		dplan.opts.gpu_binsizex=16;
		dplan.opts.gpu_binsizey=16;
		dplan.opts.gpu_binsizez=2;
		dplan.opts.gpu_maxsubprobsize=maxsubprobsize;
	}
	if(dplan.opts.gpu_method == 1)
	{
		dplan.opts.gpu_binsizex=16;
		dplan.opts.gpu_binsizey=8;
		dplan.opts.gpu_binsizez=4;
	}

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() <<" s"<<endl<<endl;
#endif

#ifdef INFO
	cout<<"[info  ] Interpolating  ["<<nf1<<"x"<<nf2<<"x"<<nf3<<
		"] uniform points to "<<M<<"nupts"<<endl;
#endif
	timer.restart();
	ier = cufinufft_interp3d(N1, N2, N3, nf1, nf2, nf3, fw, M, x, y, z, c, tol,
		&dplan);
	if(ier != 0 ){
		cout<<"error: cnufftinterp3d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld U pts to #%d NU pts in %.3g s (\t%.3g U pts/s)\n",
			dplan.opts.gpu_method,nf1*nf2*nf3,M,t,M/t);
#ifdef RESULT
	cout<<"[result-input]"<<endl;
	for(int j=0; j<10; j++){
		printf(" (%2.3g,%2.3g)",c[j].real(),c[j].imag() );
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(z);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
