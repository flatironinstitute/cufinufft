#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>

#include <cufinufft.h>
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, M, N, ntransf, ntransfcufftplan;
	if (argc<4) {
		fprintf(stderr,
			"Usage: cufinufft2d1many_test method N1 N2 [ntransf [ntransfcufftplan [M [tol]]]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven,\n"
			"    2: sub-problem, or\n"
			"    3: sub-problem with Paul's idea.\n"
			"  N1, N2: The size of the 2D array.\n"
			"  ntransf: Number of inputs (default 2 ^ 27 / (N1 * N2)).\n"
			"  ntransfcufftplan: Number of simultaneous transforms (default min(8, ntransf)).\n"
			"  M: The number of non-uniform points (default N1 * N2).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	N = N1*N2;
	M = N1*N2*2;// let density always be 2
	ntransf = pow(2,28)/M;
	if(argc>4){
		sscanf(argv[4],"%d",&ntransf);
	}
	ntransfcufftplan = min(8, ntransf);
	if(argc>5){
		sscanf(argv[5],"%d",&ntransfcufftplan);
	}

	if(argc>6){
		sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;

	printf("#modes = %d, #inputs = %d, #NUpts = %d\n", N, ntransf, M);

	FLT *x, *y;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*ntransf*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*ntransf*sizeof(CPX));

	FLT *d_x, *d_y;
	CUCPX *d_c, *d_fk;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*ntransf*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fk,N1*N2*ntransf*sizeof(CUCPX)));


	// Making data
	for (int i=0; i<M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
	}

	for(int i=0; i<M*ntransf; i++){
		c[i].real(randm11());
		c[i].imag(randm11());
	}

	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c,c,M*ntransf*sizeof(CUCPX),hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	float milliseconds = 0;
	double totaltime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*warm up gpu*/
	hipEventRecord(start);
	char *a;
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tWarm up GPU \t\t %.3g s\n", milliseconds/1000);
#endif

	cufinufft_plan dplan;
	int dim = 2;
	int type = 1;
	ier=cufinufft_default_opts(type, dim, dplan.opts);
	dplan.opts.gpu_method=method;

	int nmodes[3];
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = 1;
	hipEventRecord(start);
	ier=cufinufft_makeplan(type, dim, nmodes, iflag, ntransf, tol, 
		ntransfcufftplan, &dplan);
	if (ier!=0){
		printf("err: cufinufft2d_plan\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft_setNUpts(M, d_x, d_y, NULL, 0, NULL, NULL, NULL, &dplan);
	if (ier!=0){
		printf("err: cufinufft_setNUpts\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft_exec(d_c, d_fk, &dplan);
	if (ier!=0){
		printf("err: cufinufft2d1_exec\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft_destroy(&dplan);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);

	checkCudaErrors(hipMemcpy(fk,d_fk,N1*N2*ntransf*sizeof(CUCPX),
		hipMemcpyDeviceToHost));
#if 0
	for(int i=0; i<ntransf; i+=10){
		int nt1 = (int)(0.37*N1), nt2 = (int)(0.26*N2);  // choose some mode index to check
		CPX Ft = CPX(0,0), J = IMA*(FLT)iflag;
		for (BIGINT j=0; j<M; ++j)
			Ft += c[j+i*M] * exp(J*(nt1*x[j]+nt2*y[j]));   // crude direct
		int it = N1/2+nt1 + N1*(N2/2+nt2);   // index in complex F as 1d array
		printf("[gpu   ] one mode: abs err in F[%ld,%ld] is %.3g\n",(int)nt1,(int)nt2,abs(Ft-fk[it+i*N]));
		printf("[gpu   ] one mode: rel err in F[%ld,%ld] is %.3g\n",(int)nt1,(int)nt2,abs(Ft-fk[it+i*N])/infnorm(N,fk+i*N));
	}
#endif
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		//        if( j % opts.gpu_binsizey == 0)
		//                printf("\n");
		for (int i=0; i<nf1; i++){
			//                if( i % opts.gpu_binsizex == 0 && i!=0)
			//                        printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
#endif	
	printf("[totaltime] %.3g us, speed %.3g NUpts/s\n", totaltime*1000, M*ntransf/totaltime*1000);
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fk);
	return 0;
}
