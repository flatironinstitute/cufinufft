#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/cuspreadinterp.h"
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2, nf3;
	FLT sigma = 2.0;
	int N1, N2, N3, M;
	if (argc<5) {
		fprintf(stderr,
			"Usage: interp3d method nupts_distr nf1 nf2 nf3 [M [tol [sort]]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven, or\n"
			"    2: sub-problem.\n"
			"  nupts_distr: The distribution of the points; one of\n"
			"    0: uniform, or\n"
			"    1: concentrated in a small region.\n"
			"  nf1, nf2, nf3: The size of the 3D array.\n"
			"  M: The number of non-uniform points (default nf1 * nf2 * nf3 / 8).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n"
			"  sort: One of\n"
			"     0: do not sort the points, or\n"
			"     1: sort the points (default).\n");
		return 1;
	}
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[5],"%lf",&w); nf3 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	N3 = (int) nf3/sigma;
	M = N1*N2*N3;// let density always be 1
	if(argc>6){
		sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		if(M == 0) M=N1*N2*N3;
	}

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int sort=1;
	if(argc>8){
		sscanf(argv[8],"%d",&sort);
	}
	int ier;

	int ns=std::ceil(-log10(tol/10.0));

	cout<<scientific<<setprecision(3);


	FLT *x, *y, *z;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*nf3*sizeof(CPX));

	FLT *d_x, *d_y, *d_z;
	CUCPX *d_c, *d_fw;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_z,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fw,nf1*nf2*nf3*sizeof(CUCPX)));


	int dim=3;
	CUFINUFFT_PLAN dplan = new CUFINUFFT_PLAN_S;
	// Zero out your struct, (sets all pointers to NULL, crucial)
        memset(dplan, 0, sizeof(*dplan));
	ier = CUFINUFFT_DEFAULT_OPTS(2, dim, &(dplan->opts));
	dplan->opts.gpu_method           = method;
	dplan->opts.gpu_maxsubprobsize   = 1024;
	dplan->opts.gpu_kerevalmeth      = 0;      // not in cmd-line args
	dplan->opts.gpu_sort             = sort;
	dplan->opts.gpu_spreadinterponly = 1;

	//binsize needs to be set here, since SETUP_BINSIZE() is not called in spread, 
	//interp only wrappers.
	if(dplan->opts.gpu_method == 1)
	{
		dplan->opts.gpu_binsizex=16;
		dplan->opts.gpu_binsizey=16;
		dplan->opts.gpu_binsizez=2;
	}
	if(dplan->opts.gpu_method == 2)
	{
		dplan->opts.gpu_binsizex=16;
		dplan->opts.gpu_binsizey=16;
		dplan->opts.gpu_binsizez=2;
	}
	ier = setup_spreader_for_nufft(dplan->spopts, tol, dplan->opts);

	switch(nupts_distribute){
		// Making data
		case 0: //uniform
			{
				for (int i = 0; i < M; i++) {
					x[i] = M_PI*randm11();// x in [-pi,pi)
					y[i] = M_PI*randm11();
					z[i] = M_PI*randm11();
					//cout << x[i] << "," << y[i] << "," << z[i] << endl;
				}
			}
			break;
		case 1: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = M_PI*rand01()/(nf1*2/32);// x in [-pi,pi)
					y[i] = M_PI*rand01()/(nf2*2/32);
					z[i] = M_PI*rand01()/(nf3*2/32);
				}
			}
			break;
		default:
			cerr<<"error: nupts distr should be 0,1" << endl;
			return 1;
	}
	for(int i=0; i<nf1*nf2*nf3; i++){
		fw[i].real(1.0);
		fw[i].imag(0.0);
	}

	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_z,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fw,fw,nf1*nf2*nf3*sizeof(CUCPX),hipMemcpyHostToDevice));

	CNTime timer;
	timer.restart();
	ier = CUFINUFFT_INTERP3D(nf1, nf2, nf3, d_fw, M, d_x, d_y, d_z, d_c, dplan);
	if(ier != 0 ){
		cout<<"error: cnufftinterp3d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld U pts to #%d NU pts in %.3g s (\t%.3g NU pts/s)\n",
			dplan->opts.gpu_method,nf1*nf2*nf3,M,t,M/t);
	checkCudaErrors(hipMemcpy(c,d_c,M*sizeof(CUCPX),hipMemcpyDeviceToHost));
#ifdef RESULT
	cout<<"[result-input]"<<endl;
	for(int j=0; j<10; j++){
		printf(" (%2.3g,%2.3g)",c[j].real(),c[j].imag() );
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(z);
	hipHostFree(c);
	hipHostFree(fw);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_c);
	hipFree(d_fw);
	return 0;
}
