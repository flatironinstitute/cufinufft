#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>

#include <cufinufft.h>
#include <profile.h>
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, N3, M;
	if (argc<4) {
		fprintf(stderr,
			"Usage: cufinufft3d2_test method N1 N2 N3 [M [tol]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven, or\n"
			"    2: sub-problem.\n"
			"  N1, N2, N3: The size of the 3D array.\n"
			"  M: The number of non-uniform points (default N1 * N2 * N3).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); N3 = (int)w;  // so can read 1e6 right!
	M = N1*N2*N3;// let density always be 1
	if(argc>5){
		sscanf(argv[5],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y, *z;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*N3*sizeof(CPX));

	FLT *d_x, *d_y, *d_z;
	CUCPX *d_c, *d_fk;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_z,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fk,N1*N2*N3*sizeof(CUCPX)));

	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
		z[i] = M_PI*randm11();
	}

	for(int i=0; i<N1*N2*N3; i++){
		fk[i].real(1.0);
		fk[i].imag(1.0);
	}

	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_z,z,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fk,fk,N1*N2*N3*sizeof(CPX),
		hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	float milliseconds = 0;
	float totaltime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*warm up gpu*/
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("Warm Up",1);
		char *a;
		checkCudaErrors(hipMalloc(&a,1));
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tWarm up GPU \t\t %.3g s\n", milliseconds/1000);
#endif

	cufinufft_plan dplan;
	int dim = 3;
	int type = 2;
	ier=cufinufft_default_opts(type, dim, &dplan.opts);
	dplan.opts.gpu_method=method;
	dplan.opts.gpu_binsizex = 16;
	dplan.opts.gpu_binsizey = 16;
	dplan.opts.gpu_binsizez = 2;
	dplan.opts.gpu_maxsubprobsize = 1024;

	int nmodes[3];
	int ntransf = 1;
	int maxbatchsize = 1;
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = N3;

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft3d_plan",2);
		ier=cufinufft_makeplan(type, dim, nmodes, iflag, ntransf, tol, 
			maxbatchsize, &dplan);
		if (ier!=0){
			printf("err: cufinufft_makeplan\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft_setNUpts",3);
		ier=cufinufft_setNUpts(M, d_x, d_y, d_z, 0, NULL, NULL, NULL, &dplan);
		if (ier!=0){
			printf("err: cufinufft_setNUpts\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft_exec",4);
		ier=cufinufft_exec(d_c, d_fk, &dplan);
		if (ier!=0){
			printf("err: cufinufft_exec\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft3d_destroy",5);
		ier=cufinufft_destroy(&dplan);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);

	checkCudaErrors(hipMemcpy(c,d_c,M*sizeof(CUCPX),hipMemcpyDeviceToHost));
	
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (\t%.3g NU pts/s)\n",
			dplan.opts.gpu_method,M,N1*N2*N3,totaltime/1000,M/totaltime*1000);
#if 1
	int jt = M/2;          // check arbitrary choice of one targ pt
	CPX J = IMA*(FLT)iflag;
	CPX ct = CPX(0,0);
	int m=0;
	for (int m3=-(N3/2); m3<=(N3-1)/2; ++m3)  // loop in correct order over F
		for (int m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
			for (int m1=-(N1/2); m1<=(N1-1)/2; ++m1)
				ct += fk[m++] * exp(J*(m1*x[jt] + m2*y[jt] + m3*z[jt]));   // crude direct
	printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,
		abs(c[jt]-ct)/infnorm(M,c));
#endif	
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(z);
	hipHostFree(c);
	hipHostFree(fk);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_c);
	hipFree(d_fk);
	return 0;
}
