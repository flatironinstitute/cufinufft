#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <complex>
#include <profile.h>

#include <cufinufft_eitherprec.h>

#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, N3, M;
	if (argc<4) {
		fprintf(stderr,
			"Usage: cufinufft3d2_test method N1 N2 N3 [M [tol]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven, or\n"
			"    2: sub-problem.\n"
			"  N1, N2, N3: The size of the 3D array.\n"
			"  M: The number of non-uniform points (default N1 * N2 * N3).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n");
		return 1;
	}
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); N3 = (int)w;  // so can read 1e6 right!
	M = N1*N2*N3;// let density always be 1
	if(argc>5){
		sscanf(argv[5],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y, *z;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*N3*sizeof(CPX));

	FLT *d_x, *d_y, *d_z;
	CUCPX *d_c, *d_fk;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_z,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fk,N1*N2*N3*sizeof(CUCPX)));

	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
		z[i] = M_PI*randm11();
	}

	for(int i=0; i<N1*N2*N3; i++){
		fk[i].real(randm11());
		fk[i].imag(randm11());
	}

	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_z,z,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fk,fk,N1*N2*N3*sizeof(CPX),
		hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	float milliseconds = 0;
	float totaltime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    // warm up CUFFT (is slow, takes around 0.2 sec... )
	hipEventRecord(start);
	{
		int nf1=1;
		hipfftHandle fftplan;
		hipfftPlan1d(&fftplan,nf1,CUFFT_TYPE,1);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] dummy warmup call to CUFFT\t %.3g s\n", milliseconds/1000);

        // now to the test...
	CUFINUFFT_PLAN dplan;
	int dim = 3;
	int type = 2;

	// Here we setup our own opts, for gpu_method.
	cufinufft_opts opts;
	ier=CUFINUFFT_DEFAULT_OPTS(type, dim, &opts);
	if(ier!=0){
	  printf("err %d: CUFINUFFT_DEFAULT_OPTS\n", ier);
	  return ier;
	}
	opts.gpu_method=method;

	int nmodes[3];
	int ntransf = 1;
	int maxbatchsize = 1;
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = N3;

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft3d_plan",2);
		ier=CUFINUFFT_MAKEPLAN(type, dim, nmodes, iflag, ntransf, tol,
				       maxbatchsize, &dplan, &opts);
		if (ier!=0){
			printf("err: cufinufft_makeplan\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft_setpts",3);
		ier=CUFINUFFT_SETPTS(M, d_x, d_y, d_z, 0, NULL, NULL, NULL, dplan);
		if (ier!=0){
		  printf("err: cufinufft_setpts\n");
		  return ier;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft_execute",4);
		ier=CUFINUFFT_EXECUTE(d_c, d_fk, dplan);
		if (ier!=0){
		  printf("err: cufinufft_execute\n");
		  return ier;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	float exec_ms =	milliseconds;
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft3d_destroy",5);
		ier=CUFINUFFT_DESTROY(dplan);
		if (ier!=0){
		  printf("err: cufinufft_destroy\n");
		  return ier;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);

	checkCudaErrors(hipMemcpy(c,d_c,M*sizeof(CUCPX),hipMemcpyDeviceToHost));

	printf("[Method %d] %ld U pts to %d NU pts in %.3g s:\t%.3g NU pts/s\n",
			opts.gpu_method,N1*N2*N3,M,totaltime/1000,M/totaltime*1000);
        printf("\t\t\t\t\t(exec-only thoughput: %.3g NU pts/s)\n",M/exec_ms*1000);

	int jt = M/2;          // check arbitrary choice of one targ pt
	CPX J = IMA*(FLT)iflag;
	CPX ct = CPX(0,0);
	int m=0;
	for (int m3=-(N3/2); m3<=(N3-1)/2; ++m3)  // loop in correct order over F
		for (int m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
			for (int m1=-(N1/2); m1<=(N1-1)/2; ++m1)
				ct += fk[m++] * exp(J*(m1*x[jt] + m2*y[jt] + m3*z[jt]));   // crude direct
	printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,
		abs(c[jt]-ct)/infnorm(M,c));

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(z);
	hipHostFree(c);
	hipHostFree(fk);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_c);
	hipFree(d_fk);
	return 0;
}
