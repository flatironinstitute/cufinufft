#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <complex>

#include <cufinufft_eitherprec.h>

#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, N3, M, N;
	if (argc<4) {
		fprintf(stderr,
			"Usage: cufinufft3d1_test method N1 N2 N3 [M [tol]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven,\n"
			"    2: sub-problem, or\n"
			"    4: block gather.\n"
			"  N1, N2, N3: The size of the 3D array.\n"
			"  M: The number of non-uniform points (default N1 * N2 * N3).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n");
		return 1;
	}
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); N3 = (int)w;  // so can read 1e6 right!

	M = N1*N2*N3;// let density always be 1
	if(argc>5){
		sscanf(argv[5],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y, *z;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*N3*sizeof(CPX));

	FLT *d_x, *d_y, *d_z;
	CUCPX *d_c, *d_fk;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_z,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fk,N1*N2*N3*sizeof(CUCPX)));

	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
		z[i] = M_PI*randm11();
		c[i].real(randm11());
		c[i].imag(randm11());
	}

	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_z,z,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c,c,M*sizeof(CUCPX),hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	float milliseconds = 0;
	float totaltime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// warm up CUFFT (is slow, takes around 0.2 sec... )
	hipEventRecord(start);
	{
		int nf1=1;
		hipfftHandle fftplan;
		hipfftPlan1d(&fftplan,nf1,CUFFT_TYPE,1);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] dummy warmup call to CUFFT\t %.3g s\n", milliseconds/1000);

        // now to the test...
	CUFINUFFT_PLAN dplan;
	int dim = 3;
	int type = 1;

	// Here we setup our own opts, for gpu_method and gpu_kerevalmeth.
	cufinufft_opts opts;
	ier=CUFINUFFT_DEFAULT_OPTS(type, dim, &opts);
	if(ier!=0){
	  printf("err %d: CUFINUFFT_DEFAULT_OPTS\n", ier);
	  return ier;
	}
	opts.gpu_method=method;
	opts.gpu_kerevalmeth=1;

	int nmodes[3];
	int ntransf = 1;
	int maxbatchsize = 1;
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = N3;
	hipEventRecord(start);
	ier=CUFINUFFT_MAKEPLAN(type, dim, nmodes, iflag, ntransf, tol,
			       maxbatchsize, &dplan, &opts);
	if (ier!=0){
	  printf("err: cufinufft_makeplan\n");
	  return ier;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);


	hipEventRecord(start);
	ier=CUFINUFFT_SETPTS(M, d_x, d_y, d_z, 0, NULL, NULL, NULL, dplan);
	if (ier!=0){
	  printf("err: cufinufft_setpts\n");
	  return ier;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=CUFINUFFT_EXECUTE(d_c, d_fk, dplan);
	if (ier!=0){
	  printf("err: cufinufft_execute\n");
	  return ier;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	float exec_ms =	milliseconds;
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=CUFINUFFT_DESTROY(dplan);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);

	checkCudaErrors(hipMemcpy(fk,d_fk,N1*N2*N3*sizeof(CUCPX),
		hipMemcpyDeviceToHost));

	printf("[Method %d] %ld NU pts to %d U pts in %.3g s:\t%.3g NU pts/s\n",
			opts.gpu_method,M,N1*N2*N3,totaltime/1000,M/totaltime*1000);
        printf("\t\t\t\t\t(exec-only thoughput: %.3g NU pts/s)\n",M/exec_ms*1000);

	int nt1 = (int)(0.37*N1), nt2 = (int)(0.26*N2), nt3 = (int) (0.13*N3);  // choose some mode index to check
	CPX Ft = CPX(0,0), J = IMA*(FLT)iflag;
	for (int j=0; j<M; ++j)
		Ft += c[j] * exp(J*(nt1*x[j]+nt2*y[j]+nt3*z[j]));   // crude direct
	int it = N1/2+nt1 + N1*(N2/2+nt2) + N1*N2*(N3/2+nt3);   // index in complex F as 1d array
	N = N1*N2*N3;
//	printf("[gpu   ] one mode: abs err in F[%ld,%ld,%ld] is %.3g\n",(int)nt1,
//		(int)nt2, (int)nt3, (abs(Ft-fk[it])));
	printf("[gpu   ] one mode: rel err in F[%ld,%ld,%ld] is %.3g\n",(int)nt1,
		(int)nt2, (int)nt3, abs(Ft-fk[it])/infnorm(N,fk));

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(z);
	hipHostFree(c);
	hipHostFree(fk);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_c);
	hipFree(d_fk);

	return 0;
}
