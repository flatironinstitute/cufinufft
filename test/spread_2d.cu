#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <algorithm>
#include "../src/cuspreadinterp.h"
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT upsampfac=2.0;
	int N1, N2, M;
	if (argc<5) {
		fprintf(stderr,"Usage: spread2d method maxsubprob nupts_distr N1 N2 [M [tol [kerevalmeth]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: nupts driven\n");
		fprintf(stderr,"method 2: sub-problem\n");
		fprintf(stderr,"method 3: sub-problem with paul's idea\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int maxsubprobsize;
	sscanf(argv[2],"%d",&maxsubprobsize);

	int nupts_distribute;
	sscanf(argv[3],"%d",&nupts_distribute);
	sscanf(argv[4],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[5],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/upsampfac;
	N2 = (int) nf2/upsampfac;
	M = N1*N2;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int kerevalmeth=0;
	if(argc>8){
		sscanf(argv[8],"%d",&kerevalmeth);
	}

	int ier;

	int dim=2;
	int ns=std::ceil(-log10(tol/10.0));
	cufinufft_plan dplan;
	ier = cufinufft_default_opts(1, dim, dplan.opts);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	ier = setup_spreader_for_nufft(dplan.spopts, tol, dplan.opts);
	dplan.opts.gpu_method=method;
	dplan.opts.upsampfac=upsampfac;
	dplan.opts.gpu_maxsubprobsize=maxsubprobsize;
	dplan.opts.gpu_kerevalmeth=kerevalmeth;
	dplan.spopts.pirange=0;

	cout<<scientific<<setprecision(3);


	FLT *x, *y;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*sizeof(CPX));

	switch(nupts_distribute){
		// Making data
		case 1: //uniform
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*randm11(), nf2, 1);
					c[i].real(randm11());
					c[i].imag(randm11());
				}
			}
		case 2: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*rand01(), nf1, 1)/2.0 - 0.5;// x in [-pi,pi)
					y[i] = RESCALE(M_PI*rand01(), nf2, 1)/2.0 - 0.5;
					if(method == 6){
						x[i] = x[i] > nf1-0.5 ? x[i] - nf1 : x[i];
						y[i] = y[i] > nf2-0.5 ? y[i] - nf2 : y[i];// x in [-pi,pi)
					}
					c[i].real(randm11());
					c[i].imag(randm11());
				}
			}
			break;
	}

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() 
		<<" s"<<endl<<endl;
#endif

#ifdef INFO
	cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"
		<<endl;
#endif
	timer.restart();
	ier = cufinufft_spread2d(N1, N2, nf1, nf2, fw, M, x, y, c, &dplan);
	if(ier != 0 ){
		cout<<"error: cnufftspread2d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (%.3g NU pts/s)\n",
			dplan.opts.gpu_method,M,nf1*nf2,t,M/t);
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % dplan.opts.gpu_binsizey == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % dplan.opts.gpu_binsizex == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
