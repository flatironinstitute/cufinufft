#include <iostream>
#include <iomanip>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/cuspreadinterp.h"
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2, nf3;
	FLT sigma = 2.0;
	int N1, N2, N3, M;
	if (argc<6) {
		fprintf(stderr,
			"Usage: spread3d_test method nupts_distr nf1 nf2 nf3 [maxsubprobsize [M [tol [kerevalmeth [sort]]]]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven,\n"
			"    2: sub-problem, or\n"
			"    4: block gather (each nf must be multiple of 8).\n"
			"  nupts_distr: The distribution of the points; one of\n"
			"    0: uniform, or\n"
			"    1: concentrated in a small region.\n"
			"  nf1, nf2, nf3: The size of the 3D array.\n"
			"  maxsubprobsize: Maximum size of subproblems (default 65536).\n"
			"  M: The number of non-uniform points (default nf1 * nf2 * nf3 / 8).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n"
			"  kerevalmeth: Kernel evaluation method; one of\n"
			"     0: Exponential of square root (default), or\n"
			"     1: Horner evaluation.\n"
			"  sort: One of\n"
			"     0: do not sort the points, or\n"
			"     1: sort the points (default).\n");
		return 1;
	}
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[5],"%lf",&w); nf3 = (int)w;  // so can read 1e6 right!

	int maxsubprobsize=1024;
	if(argc>6){
		sscanf(argv[6],"%d",&maxsubprobsize);
	}
	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	N3 = (int) nf3/sigma;
	M = N1*N2*N3;// let density always be 1
	if(argc>7){
		sscanf(argv[7],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		//if(M == 0) M=N1*N2;
	}

	FLT tol=1e-6;
	if(argc>8){
		sscanf(argv[8],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int kerevalmeth=0;
	if(argc>9){
		sscanf(argv[9],"%d",&kerevalmeth);
	}

	int sort=1;
	if(argc>10){
		sscanf(argv[10],"%d",&sort);
	}

	int ier;
	FLT *x, *y, *z;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*nf3*sizeof(CPX));

	FLT *d_x, *d_y, *d_z;
	CUCPX *d_c, *d_fw;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_z,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fw,nf1*nf2*nf3*sizeof(CUCPX)));

	int dim=3;
	CUFINUFFT_PLAN dplan = new CUFINUFFT_PLAN_S;
	// Zero out your struct, (sets all pointers to NULL, crucial)
        memset(dplan, 0, sizeof(*dplan));
	ier = CUFINUFFT_DEFAULT_OPTS(1, dim, &(dplan->opts));

	dplan->opts.gpu_method          =method;
	dplan->opts.gpu_maxsubprobsize  =maxsubprobsize;
	dplan->opts.gpu_kerevalmeth     =kerevalmeth;
	dplan->opts.gpu_sort            =sort;
	dplan->opts.gpu_spreadinterponly=1;
	ier = setup_spreader_for_nufft(dplan->spopts, tol, dplan->opts);

	//binsize, obinsize need to be set here, since SETUP_BINSIZE() is not 
	//called in spread, interp only wrappers.
	if(dplan->opts.gpu_method == 4)
	{
		dplan->opts.gpu_binsizex=4;
		dplan->opts.gpu_binsizey=4;
		dplan->opts.gpu_binsizez=4;
		dplan->opts.gpu_obinsizex=8;
		dplan->opts.gpu_obinsizey=8;
		dplan->opts.gpu_obinsizez=8;
		dplan->opts.gpu_maxsubprobsize=maxsubprobsize;
	}
	if(dplan->opts.gpu_method == 2)
	{
		dplan->opts.gpu_binsizex=16;
		dplan->opts.gpu_binsizey=16;
		dplan->opts.gpu_binsizez=2;
		dplan->opts.gpu_maxsubprobsize=maxsubprobsize;
	}
	if(dplan->opts.gpu_method == 1)
	{
		dplan->opts.gpu_binsizex=16;
		dplan->opts.gpu_binsizey=16;
		dplan->opts.gpu_binsizez=2;
	}

	cout<<scientific<<setprecision(3);

	switch(nupts_distribute){
		// Making data
		case 0: //uniform
			{
				for (int i = 0; i < M; i++) {
					x[i] = M_PI*randm11();
					y[i] = M_PI*randm11();
					z[i] = M_PI*randm11();
					c[i].real(randm11());
					c[i].imag(randm11());
				}
			}
			break;
		case 1: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = M_PI*rand01()/nf1*16;
					y[i] = M_PI*rand01()/nf2*16;
					z[i] = M_PI*rand01()/nf3*16;
					c[i].real(randm11());
					c[i].imag(randm11());
				}
			}
			break;
		default:
			cerr << "not valid nupts distr" << endl;
			return 1;
	}

	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_z,z,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c,c,M*sizeof(CUCPX),hipMemcpyHostToDevice));

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
	// cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec()
	//	<<" s"<<endl<<endl;



	timer.restart();
	ier = CUFINUFFT_SPREAD3D(nf1, nf2, nf3, d_fw, M, d_x, d_y, d_z, d_c, dplan);
	if(ier != 0 ){
		cout<<"error: cnufftspread3d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (%.3g NU pts/s)\n",
			dplan->opts.gpu_method,M,nf1*nf2*nf3,t,M/t);
#ifdef RESULT
	cout<<"[result-input]"<<endl;
	for(int k=0; k<nf3; k++){
		for(int j=0; j<nf2; j++){
			for (int i=0; i<nf1; i++){
				if( i % dplan->opts.gpu_binsizex == 0 && i!=0)
					printf(" |");
				printf(" (%2.3g,%2.3g)",fw[i+j*nf1+k*nf2*nf1].real(),
					fw[i+j*nf1+k*nf2*nf1].imag() );
			}
			cout<<endl;
		}
		cout<<"----------------------------------------------------------------"<<endl;
	}
#endif

	hipDeviceReset();
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(z);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
