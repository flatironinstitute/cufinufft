#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <complex>

#include <cufinufft_eitherprec.h>

#include <profile.h>
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, M;
	if (argc<4) {
		fprintf(stderr,
			"Usage: cufinufft2d2_test method N1 N2 [M [tol]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven, or\n"
			"    2: sub-problem.\n"
			"  N1, N2: The size of the 2D array.\n"
			"  M: The number of non-uniform points (default N1 * N2).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n");
		return 1;
	}
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	M = N1*N2;// let density always be 1
	if(argc>4){
		sscanf(argv[4],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>5){
		sscanf(argv[5],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*sizeof(CPX));

	FLT *d_x, *d_y;
	CUCPX *d_c, *d_fk;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fk,N1*N2*sizeof(CUCPX)));
	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
	}
	for(int i=0; i<N1*N2; i++){
		fk[i].real(randm11());
		fk[i].imag(randm11());
	}
	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fk, fk, N1*N2*sizeof(CPX),
		hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	float milliseconds = 0;
        float totaltime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// warm up CUFFT (is slow, takes around 0.2 sec... )
	hipEventRecord(start);
	{
		int nf1=1;
		hipfftHandle fftplan;
		hipfftPlan1d(&fftplan,nf1,CUFFT_TYPE,1);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] dummy warmup call to CUFFT\t %.3g s\n", milliseconds/1000);

        // now to the test...
	CUFINUFFT_PLAN dplan;
	int dim = 2;
	int type = 2;

	// Here we setup our own opts, for gpu_method.
	cufinufft_opts opts;
	ier=CUFINUFFT_DEFAULT_OPTS(type, dim, &opts);
	if(ier!=0){
	  printf("err %d: CUFINUFFT_DEFAULT_OPTS\n", ier);
	  return ier;
	}
	opts.gpu_method=method;

	int nmodes[3];
	int ntransf = 1;
	int maxbatchsize = 1;
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = 1;
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_plan",2);
		ier=CUFINUFFT_MAKEPLAN(type, dim, nmodes, iflag, ntransf, tol,
				       maxbatchsize, &dplan, &opts);
		if (ier!=0){
			printf("err: cufinufft2d_plan\n");
			return ier;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_setNUpts",3);
		ier=CUFINUFFT_SETPTS(M, d_x, d_y, NULL, 0, NULL, NULL, NULL, dplan);
		if (ier!=0){
			printf("err: cufinufft_setpts\n");
			return ier;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d2_exec",4);
		ier=CUFINUFFT_EXECUTE(d_c, d_fk, dplan);
		if (ier!=0){
			printf("err: cufinufft2d2_exec\n");
			return ier;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	float exec_ms = milliseconds;
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_destroy",5);
		ier=CUFINUFFT_DESTROY(dplan);
		if(ier!=0){
		  printf("err %d: cufinufft2d2_destroy\n", ier);
		  return ier;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);

        printf("[Method %d] %d U pts to %d NU pts in %.3g s:      %.3g NU pts/s\n", opts.gpu_method,N1*N2,M,totaltime/1000,M/totaltime*1000);
        printf("\t\t\t\t\t(exec-only thoughput: %.3g NU pts/s)\n",M/exec_ms*1000);


	checkCudaErrors(hipMemcpy(c,d_c,M*sizeof(CUCPX),hipMemcpyDeviceToHost));
	int jt = M/2;          // check arbitrary choice of one targ pt
	CPX J = IMA*(FLT)iflag;
	CPX ct = CPX(0,0);
	int m=0;
	for (int m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
		for (int m1=-(N1/2); m1<=(N1-1)/2; ++m1)
			ct += fk[m++] * exp(J*(m1*x[jt] + m2*y[jt]));   // crude direct
	printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,abs(c[jt]-ct)/infnorm(M,c));

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fk);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_c);
	hipFree(d_fk);
	return 0;
}
