#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>

#include <cufinufft.h>
#include <profile.h>
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, M;
	if (argc<4) {
		fprintf(stderr,
			"Usage: cufinufft2d2_test method N1 N2 [M [tol]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven, or\n"
			"    2: sub-problem.\n"
			"  N1, N2: The size of the 2D array.\n"
			"  M: The number of non-uniform points (default N1 * N2).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	M = N1*N2;// let density always be 1
	if(argc>4){
		sscanf(argv[4],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>5){
		sscanf(argv[5],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*sizeof(CPX));

	FLT *d_x, *d_y;
	CUCPX *d_c, *d_fk;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fk,N1*N2*sizeof(CUCPX)));
	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
	}
	for(int i=0; i<N1*N2; i++){
		fk[i].real(1.0);
		fk[i].imag(1.0);
	}
	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fk, fk, N1*N2*sizeof(CPX), 
		hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	float milliseconds = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*warm up gpu*/
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("Warm Up",1);
		char *a;
		checkCudaErrors(hipMalloc(&a,1));
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tWarm up GPU \t\t %.3g s\n", milliseconds/1000);
#endif

	cufinufft_plan dplan;
	int dim = 2;
	int type = 2;
	ier=cufinufft_default_opts(type, dim, dplan.opts);
	dplan.opts.gpu_method=method;

	int nmodes[3];
	int ntransf = 1;
	int ntransfcufftplan = 1;
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = 1;
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_plan",2);
		ier=cufinufft_makeplan(type, dim, nmodes, iflag, ntransf, tol, 
			ntransfcufftplan, &dplan);
		if (ier!=0){
			printf("err: cufinufft2d_plan\n");
		}
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);
#endif
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_setNUpts",3);
		ier=cufinufft_setNUpts(M, d_x, d_y, NULL, 0, NULL, NULL, NULL, &dplan);
		if (ier!=0){
			printf("err: cufinufft_setNUpts\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d2_exec",4);
		ier=cufinufft_exec(d_c, d_fk, &dplan);
		if (ier!=0){
			printf("err: cufinufft2d2_exec\n");
		}
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);
#endif
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_destroy",5);
		ier=cufinufft_destroy(&dplan);
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);
#endif
	checkCudaErrors(hipMemcpy(c,d_c,M*sizeof(CUCPX),hipMemcpyDeviceToHost));
	int jt = M/2;          // check arbitrary choice of one targ pt
	CPX J = IMA*(FLT)iflag;
	CPX ct = CPX(0,0);
	int m=0;
	for (int m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
		for (int m1=-(N1/2); m1<=(N1-1)/2; ++m1)
			ct += fk[m++] * exp(J*(m1*x[jt] + m2*y[jt]));   // crude direct
	printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,abs(c[jt]-ct)/infnorm(M,c));
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		//        if( j % opts.gpu_binsizey == 0)
		//                printf("\n");
		for (int i=0; i<nf1; i++){
			//                if( i % opts.gpu_binsizex == 0 && i!=0)
			//                        printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
#endif	
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fk);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_c);
	hipFree(d_fk);
	return 0;
}
