#include "hip/hip_runtime.h"
#include "../finufft/finufft.h"
//#include "../src/cufinufft.h"
#include "../finufft/dirft.h"
#include "../finufft/spreadinterp.h"
#include <math.h>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime_api.h>

// how big a problem to do full direct DFT check in 2D...
#define BIGPROB 1e8

// for omp rand filling
#define CHUNK 1000000

int main(int argc, char* argv[])
/* Test executable for finufft in 2d, all 3 types

   Usage: finufft2d_test [Nmodes1 Nmodes2 [Nsrc [tol [debug [spread_sort [upsampfac]]]]]]

   debug = 0: rel errors and overall timing, 1: timing breakdowns
           2: also spreading output

   Example: finufft2d_test 1000 1000 1000000 1e-12 1 2 2.0

   Barnett 2/1/17
*/
{
  BIGINT M = 1e6, N1 = 1000, N2 = 500;  // defaults: M = # srcs, N1,N2 = # modes
  double w, tol = 1e-6;          // default
  double upsampfac = 2.0;    // default
  nufft_opts opts; finufft_default_opts(opts);
  opts.debug = 0;            // 1 to see some timings
  opts.fftw = FFTW_MEASURE;  // change from usual FFTW_ESTIMATE        ***
  int isign = +1;             // choose which exponential sign to test
  if (argc>1) {
    sscanf(argv[1],"%lf",&w); N1 = (BIGINT)w;
    sscanf(argv[2],"%lf",&w); N2 = (BIGINT)w;
  }
  if (argc>3) { sscanf(argv[3],"%lf",&w); M = (BIGINT)w; }
  if (argc>4) {
    sscanf(argv[4],"%lf",&tol);
    if (tol<=0.0) { printf("tol must be positive!\n"); return 1; }
  }
  if (argc>5) sscanf(argv[5],"%d",&opts.debug);
  opts.spread_debug = (opts.debug>1) ? 1 : 0;  // see output from spreader
  if (argc>6) sscanf(argv[6],"%d",&opts.spread_sort);
  if (argc>7) sscanf(argv[7],"%lf",&upsampfac);
  opts.upsampfac=(FLT)upsampfac;
  if (argc==1 || argc==2 || argc>8) {
    fprintf(stderr,"Usage: finufft2d_test [N1 N2 [Nsrc [tol [debug [spread_sort [upsampfac]]]]]]\n");
    return 1;
  }
  cout << scientific << setprecision(3);
  BIGINT N = N1*N2;

  FLT *x = (FLT *)malloc(sizeof(FLT)*M);        // NU pts x coords
  FLT *y = (FLT *)malloc(sizeof(FLT)*M);        // NU pts y coords
  CPX* c = (CPX*)malloc(sizeof(CPX)*M);   // strengths 
  CPX* Fcpu = (CPX*)malloc(sizeof(CPX)*N);   // mode ampls
  CPX* Fgpu = (CPX*)malloc(sizeof(CPX)*N);   // mode ampls
#pragma omp parallel
  {
    unsigned int se=MY_OMP_GET_THREAD_NUM();  // needed for parallel random #s
#pragma omp for schedule(dynamic,CHUNK)
    for (BIGINT j=0; j<M; ++j) {
      x[j] = M_PI*randm11r(&se);
      y[j] = M_PI*randm11r(&se);
      c[j] = crandm11r(&se);
    }
  }

  printf("test 2d type-1:\n"); // -------------- type 1
  CNTime timer; timer.start();
  int ier;
  double ti;
  ier = finufft2d1(M,x,y,c,isign,tol,N1,N2,Fcpu,opts);
  ti=timer.elapsedsec();
  if (ier!=0) {
    printf("error (ier=%d)!\n",ier);
  } else
    printf("[cpu   ] %ld NU pts to (%ld,%ld) modes in %.3g s \t%.3g NU pts/s\n\n",
	   (int64_t)M,(int64_t)N1,(int64_t)N2,ti,M/ti);
  char *a;
  timer.restart();
  checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
  printf("[time  ] (warm up) First cudamalloc call %.3g s\n", timer.elapsedsec());
#endif
  timer.restart();
  ier = finufft2d1_gpu(M,x,y,c,isign,tol,N1,N2,Fgpu,opts);
  ti=timer.elapsedsec();
  if (ier!=0) {
    printf("error (ier=%d)!\n",ier);
  } else
    printf("[gpu   ] %ld NU pts to (%ld,%ld) modes in %.3g s \t%.3g NU pts/s\n",
	   (int64_t)M,(int64_t)N1,(int64_t)N2,ti,M/ti);

  BIGINT nt1 = (BIGINT)(0.37*N1), nt2 = (BIGINT)(0.26*N2);  // choose some mode index to check
  CPX Ft = CPX(0,0), J = IMA*(FLT)isign;
  for (BIGINT j=0; j<M; ++j)
    Ft += c[j] * exp(J*(nt1*x[j]+nt2*y[j]));   // crude direct
  BIGINT it = N1/2+nt1 + N1*(N2/2+nt2);   // index in complex F as 1d array
  
  CPX* Ftt = (CPX*)malloc(sizeof(CPX)*N);
  if ((int64_t)M*N<=BIGPROB) {                   // also check vs full direct eval
    dirft2d1(M,x,y,c,isign,N1,N2,Ftt);
  }

  printf("\n[cpu   ] one mode: abs err in F[%ld,%ld] is %.3g\n",(int64_t)nt1,(int64_t)nt2,abs(Ft-Fcpu[it]));
  printf("[cpu   ] one mode: rel err in F[%ld,%ld] is %.3g\n",(int64_t)nt1,(int64_t)nt2,abs(Ft-Fcpu[it])/infnorm(N,Fcpu));
  if ((int64_t)M*N<=BIGPROB) {                   // also check vs full direct eval
    printf("[cpu   ]dirft2d: rel l2-err of result F is %.3g\n",relerrtwonorm(N,Ftt,Fcpu));
  }
#if 1
  printf("[gpu   ] one mode: abs err in F[%ld,%ld] is %.3g\n",(int64_t)nt1,(int64_t)nt2,abs(Ft-Fgpu[it]));
  printf("[gpu   ] one mode: rel err in F[%ld,%ld] is %.3g\n",(int64_t)nt1,(int64_t)nt2,abs(Ft-Fgpu[it])/infnorm(N,Fgpu));
  if ((int64_t)M*N<=BIGPROB) {                   // also check vs full direct eval
    printf("[gpu   ]dirft2d: rel l2-err of result F is %.3g\n",relerrtwonorm(N,Ftt,Fgpu));
    free(Ftt);
  }
#endif

  printf("\ntest 2d type-2:\n"); // -------------- type 2
  CPX* F = (CPX*)malloc(sizeof(CPX)*N);   // mode ampls
  CPX* ccpu = (CPX*)malloc(sizeof(CPX)*M);   // strengths 
  CPX* cgpu = (CPX*)malloc(sizeof(CPX)*M);   // strengths 
// since x, y have been modified by gpu code
#pragma omp parallel
  {
    unsigned int se=MY_OMP_GET_THREAD_NUM();  // needed for parallel random #s
#pragma omp for schedule(dynamic,CHUNK)
    for (BIGINT j=0; j<M; ++j) {
      x[j] = M_PI*randm11r(&se);
      y[j] = M_PI*randm11r(&se);
    }
  }
#pragma omp parallel
  {
    unsigned int se=MY_OMP_GET_THREAD_NUM();
#pragma omp for schedule(dynamic,CHUNK)
    for (BIGINT m=0; m<N; ++m) F[m] = crandm11r(&se);
  }
  timer.restart();
  ier = finufft2d2(M,x,y,ccpu,isign,tol,N1,N2,F,opts);
  ti=timer.elapsedsec();
  if (ier!=0) {
    printf("error (ier=%d)!\n",ier);
  } else
    printf("[cpu   ] (%ld,%ld) modes to %ld NU pts in %.3g s \t%.3g NU pts/s\n\n",(int64_t)N1,(int64_t)N2,(int64_t)M,ti,M/ti);
  timer.restart();
  ier = finufft2d2_gpu(M,x,y,cgpu,isign,tol,N1,N2,F,opts);
  ti=timer.elapsedsec();
  if (ier!=0) {
    printf("error (ier=%d)!\n",ier);
  } else
    printf("\n[gpu   ] %ld NU pts to (%ld,%ld) modes in %.3g s \t%.3g NU pts/s\n",
	   (int64_t)M,(int64_t)N1,(int64_t)N2,ti,M/ti);

  BIGINT jt = M/2;          // check arbitrary choice of one targ pt
  CPX ct = CPX(0,0);
  BIGINT m=0;
  for (BIGINT m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
    for (BIGINT m1=-(N1/2); m1<=(N1-1)/2; ++m1)
      ct += F[m++] * exp(J*(m1*x[jt] + m2*y[jt]));   // crude direct

  CPX* ctt = (CPX*)malloc(sizeof(CPX)*M);
  if ((int64_t)M*N<=BIGPROB) {                  // also full direct eval
    dirft2d2(M,x,y,ctt,isign,N1,N2,F);
  }

  printf("\n[cpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,abs(ccpu[jt]-ct)/infnorm(M,ccpu));
  if ((int64_t)M*N<=BIGPROB) {                  // also full direct eval
    printf("[cpu   ] dirft2d: rel l2-err of result c is %.3g\n",relerrtwonorm(M,ctt,ccpu));
  }
  printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,abs(ct-cgpu[jt])/infnorm(M,cgpu));
  if ((int64_t)M*N<=BIGPROB) {                  // also full direct eval
    printf("[gpu   ] dirft2d: rel l2-err of result c is %.3g\n",relerrtwonorm(M,ctt,cgpu));
    free(ctt);
  }
#if 0
  printf("test 2d type-3:\n"); // -------------- type 3
  // reuse the strengths c, interpret N as number of targs:
#pragma omp parallel
  {
    unsigned int se=MY_OMP_GET_THREAD_NUM();
#pragma omp for schedule(dynamic,CHUNK)
    for (BIGINT j=0; j<M; ++j) {
      x[j] = 2.0 + M_PI*randm11r(&se);      // new x_j srcs, offset from origin
      y[j] = -3.0 + M_PI*randm11r(&se);     // " y_j
    }
  }
  FLT* s = (FLT*)malloc(sizeof(FLT)*N);    // targ freqs (1-cmpt)
  FLT* t = (FLT*)malloc(sizeof(FLT)*N);    // targ freqs (2-cmpt)
  FLT S1 = (FLT)N1/2;                   // choose freq range sim to type 1
  FLT S2 = (FLT)N2/2;
#pragma omp parallel
  {
    unsigned int se=MY_OMP_GET_THREAD_NUM();
#pragma omp for schedule(dynamic,CHUNK)
    for (BIGINT k=0; k<N; ++k) {
      s[k] = S1*(1.7 + randm11r(&se));    //S*(1.7 + k/(FLT)N); // offset the freqs
      t[k] = S2*(-0.5 + randm11r(&se));
    }
  }
  timer.restart();
  ier = finufft2d3(M,x,y,c,isign,tol,N,s,t,F,opts);
  ti=timer.elapsedsec();
  if (ier!=0) {
    printf("error (ier=%d)!\n",ier);
  } else
    printf("\t%ld NU to %ld NU in %.3g s   %.3g srcs/s, %.3g targs/s\n",(int64_t)M,(int64_t)N,ti,M/ti,N/ti);

  BIGINT kt = N/2;          // check arbitrary choice of one targ pt
  Ft = CPX(0,0);
  for (BIGINT j=0;j<M;++j)
    Ft += c[j] * exp(IMA*(FLT)isign*(s[kt]*x[j] + t[kt]*y[j]));
  printf("one targ: rel err in F[%ld] is %.3g\n",(int64_t)kt,abs(Ft-F[kt])/infnorm(N,F));
  if (((int64_t)M)*N<=BIGPROB) {                  // also full direct eval
    CPX* Ft = (CPX*)malloc(sizeof(CPX)*N);
    dirft2d3(M,x,y,c,isign,N,s,t,Ft);       // writes to F
    printf("dirft2d: rel l2-err of result F is %.3g\n",relerrtwonorm(N,Ft,F));
    //cout<<"s t, F, Ft, F/Ft:\n"; for (int k=0;k<N;++k) cout<<s[k]<<" "<<t[k]<<", "<<F[k]<<",\t"<<Ft[k]<<",\t"<<F[k]/Ft[k]<<endl;
    free(Ft);
  }
#endif
  free(x); free(y); free(c); free(Fgpu); free(Fcpu); free(F); free(ccpu); free(cgpu);//free(s); free(t);
  return ier;
}
