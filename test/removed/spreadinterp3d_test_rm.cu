#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/spreadinterp.h"
#include "../finufft/utils.h"
#include "../finufft/spreadinterp.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2, nf3;
	FLT sigma = 2.0;
	int N1, N2, N3, M;
	if (argc<5) {
		fprintf(stderr,"Usage: ./spreadinterp [method [nupts_distribute [N1 N2 [M [tol]]]]]\n");
		return 1;
	}  
	int method;
	sscanf(argv[1],"%d",&method);

	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);

	double w;
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[5],"%lf",&w); nf3 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	N3 = (int) nf3/sigma;
	M = N1*N2*N3;// let density always be 1
	if(argc>6){
		sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		if(M==0) M=N1*N2*N3;
	}

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int ier;
	int ns=std::ceil(-log10(tol/10.0));
	cufinufft_plan dplan;
	FLT upsampfac=2.0;
	cout<<scientific<<setprecision(6);


	FLT *x, *y, *z;
	CPX *c;
	CPX *fws, *fwfinufft;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));

	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fws,       nf1*nf2*nf3*sizeof(CPX));
	hipHostMalloc(&fwfinufft, nf1*nf2*nf3*sizeof(CPX));
#if 0
	// spread a single source, only for reference accuracy check...
	c[0].real(1.0; c[0].imag(0.0));   // unit strength
	x[0] = y[0] = nf1/2.0;                  // at center
	ier = cnufftspread(nf1,nf2,1,(FLT*) fwfinufft,1,x,y,NULL,(FLT*) c,opts);
	if (ier!=0) {
		printf("error when spreading M=1 pt for ref acc check (ier=%d)!\n",ier);
		return ier;
	}
	FLT kersumre = 0.0, kersumim = 0.0;  // sum kernel on uniform grid
	for (int i=0;i<nf1*nf2;++i) {
		kersumre += fwfinufft[i].real();
		kersumim += fwfinufft[i].imag();    // in case the kernel isn't real!
	}
#endif
	FLT strre = 0.0, strim = 0.0;          // also sum the strengths
	switch(nupts_distribute){
		// Making data
		case 1: //uniform
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*randm11(), nf2, 1);
					z[i] = RESCALE(M_PI*randm11(), nf3, 1);
					c[i].real(randm11());
					c[i].imag(randm11());
					strre += c[i].real();
					strim += c[i].imag();
					//cout <<x[i]<<","<<y[i]<<","<<z[i]<<endl;
				}
			}
			break;
		case 2: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*rand01()/(nf2*2/32), nf2, 1);
					z[i] = RESCALE(M_PI*rand01()/(nf3*2/32), nf3, 1);
					c[i].real(randm11());
					c[i].imag(randm11());
					strre += c[i].real();
					strim += c[i].imag();
				}
			}
			break;
	}

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() <<" s"<<endl<<endl;
#endif

	// Direction 1: Spreading
	printf("[info  ] Type 1: Spreading\n");
#if 0
	FLT sumre = 0.0, sumim = 0.0;   // check spreading accuracy, wrapping
	for (int i=0;i<nf1*nf2;++i) {
		sumre += fwi[i].real();
		sumim += fwi[i].imag();
	}
	FLT pre = kersumre*strre - kersumim*strim;   // pred ans, complex mult
	FLT pim = kersumim*strre + kersumre*strim;
	FLT maxerr = std::max(fabs(sumre-pre), fabs(sumim-pim));
	FLT ansmod = sqrt(sumre*sumre+sumim*sumim);
	printf("    rel err in total over grid:      %.3g\n",maxerr/ansmod);
	// note this is weaker than below dir=2 test, but is good indicator that
	// periodic wrapping is correct
#endif

	/* -------------------------------------- */
	// Method 5: Subprob                     //
	/* -------------------------------------- */
	ier = cufinufft_default_opts(type1, 3, dplan.opts);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	ier = setup_spreader_for_nufft(dplan.spopts, tol, dplan.opts);
	dplan.opts.upsampfac=upsampfac;
	dplan.opts.gpu_method=method;
	dplan.opts.gpu_kerevalmeth=1;
	dplan.opts.gpu_sort=1;
	dplan.spopts.pirange=0;
	switch(dplan.opts.gpu_method){
		case 4:
		{
			dplan.opts.gpu_binsizex=4;
			dplan.opts.gpu_binsizey=4;
			dplan.opts.gpu_binsizez=4;
			dplan.opts.gpu_obinsizex=8;
			dplan.opts.gpu_obinsizey=8;
			dplan.opts.gpu_obinsizez=8;
			dplan.opts.gpu_maxsubprobsize=1024;
		}
		break;
		case 2:
		{
			dplan.opts.gpu_binsizex=8;
			dplan.opts.gpu_binsizey=8;
			dplan.opts.gpu_binsizez=2;
			dplan.opts.gpu_maxsubprobsize=1024;
		}
		break;
		case 1:
		{
			dplan.opts.gpu_binsizex=8;
			dplan.opts.gpu_binsizey=8;
			dplan.opts.gpu_binsizez=2;
		}
		break;
	}
	timer.restart();
	ier = cufinufft_spread3d(N1, N2, N3, nf1, nf2, nf3, fws, M, x, y, z, c, tol, 
		&dplan);
	FLT tsubprob=timer.elapsedsec();
	if(ier != 0 ){
		cout<<"error: cnufftspread3d_gpu_subprob"<<endl;
		return 0;
	}
	printf("[method %d] %ld NU pts to (%ld,%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			method,M,N1,N2,N3,nf1*nf2*nf3,tsubprob,M/tsubprob);
	/* -------------------------------------- */
	// FINUTFFT cpu spreader                  //
	/* -------------------------------------- */
	timer.start();
	spread_opts spopts;
	setup_spreader(spopts,(FLT)tol,upsampfac,1);
	spopts.pirange=0;
	spopts.chkbnds=1;
	spopts.spread_direction=1;
	spopts.flags=0;//ker always return 1
	spopts.kerpad=1;
	spopts.sort_threads=0;
	spopts.sort=2;
	spopts.debug=1;

	ier = spreadinterp(nf1,nf2,nf3,(FLT*) fwfinufft,M,x,y,z,(FLT*) c,spopts);
	FLT t=timer.elapsedsec();
	if (ier!=0) {
		printf("error (ier=%d)!\n",ier);
		return ier;
	}
	printf("[finufft] %ld NU pts to (%ld,%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			M,N1,N2,N3,nf1*nf2*nf3,t,M/t);
	//printf("    %.3g NU pts in %.3g s \t%.3g pts/s \t%.3g spread pts/s\n",(double)M,t,M/t,pow(opts.nspread,2)*M/t);
	/* ------------------------------------------------------------------------------------------------------*/

	cout<<endl;
	FLT err=relerrtwonorm(nf1*nf2*nf3,fws,fwfinufft);
	printf("|| fws  - fwfinufft ||_2 / || fws  ||_2 =  %.6g\n", err);

#if 0
	cout<<"[result-input]"<<endl;
	for(int k=0; k<nf3; k++){
		for(int j=0; j<nf2; j++){
			//if( j % opts.bin_size_y == 0)
			//	printf("\n");
			for (int i=0; i<nf1; i++){
				if( i % opts.bin_size_x == 0 && i!=0)
					printf(" |");
				printf(" (%2.3g,%2.3g)",fws[i+j*nf1+k*nf2*nf1].real(),
					fws[i+j*nf1+k*nf2*nf1].imag() );
			}
			cout<<endl;
		}
		cout<<"----------------------------------------------------------------"<<endl;
	}
#endif
#if 0
	cout<<"[result-input]"<<endl;
	for(int k=0; k<nf3; k++){
		for(int j=0; j<nf2; j++){
			//if( j % opts.bin_size_y == 0)
			//	printf("\n");
			for (int i=0; i<nf1; i++){
				if( i % opts.bin_size_x == 0 && i!=0)
					printf(" |");
				printf(" (%2.3g,%2.3g)",fwfinufft[i+j*nf1+k*nf2*nf1].real(),
					fwfinufft[i+j*nf1+k*nf2*nf1].imag() );
			}
			cout<<endl;
		}
		cout<<"----------------------------------------------------------------"<<endl;
	}
#endif
#if 1
	cout<<"[resultdiff]"<<endl;
	FLT fwfinufft_infnorm=infnorm(nf1*nf2*nf3, fwfinufft);
	int nn=0;
	for(int k=0; k<nf3; k++){
		for(int j=0; j<nf2; j++){
			for (int i=0; i<nf1; i++){
				if( norm(fws[i+j*nf1+k*nf1*nf2]-fwfinufft[i+j*nf1+k*nf1*nf2])/
					fwfinufft_infnorm > tol & nn<10){
					cout<<"(i,j,k)=("<<i<<","<<j<<","<<k<<"), "<<
						fws[i+j*nf1+k*nf1*nf2] <<","<<
						fwfinufft[i+j*nf1+k*nf1*nf2]<<endl;
					nn++;
				}
			}
		}
	}
	cout<<endl;
#endif
	// Direction 2: Interpolation
	printf("\n[info  ] Type 2: Interpolation\n");
	ier = cufinufft_default_opts(type2, 3, dplan.opts);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	ier = setup_spreader_for_nufft(dplan.spopts, tol, dplan.opts);
	
	dplan.opts.upsampfac=upsampfac;
	dplan.opts.gpu_method=method;
	dplan.opts.gpu_kerevalmeth=1;
	dplan.opts.gpu_sort=1;
	dplan.spopts.pirange=0;
	switch(dplan.opts.gpu_method){
		case 4:
		{
			dplan.opts.gpu_binsizex=4;
			dplan.opts.gpu_binsizey=4;
			dplan.opts.gpu_binsizez=4;
			dplan.opts.gpu_obinsizex=8;
			dplan.opts.gpu_obinsizey=8;
			dplan.opts.gpu_obinsizez=8;
			dplan.opts.gpu_maxsubprobsize=1024;
		}
		break;
		case 2:
		{
			dplan.opts.gpu_binsizex=8;
			dplan.opts.gpu_binsizey=8;
			dplan.opts.gpu_binsizez=2;
			dplan.opts.gpu_maxsubprobsize=1024;
		}
		break;
		case 1:
		{
			dplan.opts.gpu_binsizex=8;
			dplan.opts.gpu_binsizey=8;
			dplan.opts.gpu_binsizez=2;
		}
		break;
	}

	CPX *fw;
	CPX *cfinufft, *cs;
	hipHostMalloc(&fw, nf1*nf2*nf3*sizeof(CPX));
	hipHostMalloc(&cfinufft, M*sizeof(CPX));
	hipHostMalloc(&cs,       M*sizeof(CPX));

	for(int i=0; i<nf1*nf2*nf3; i++){
		fw[i].real(1.0);
		fw[i].imag(0.0);
	}
	/* -------------------------------------- */
	// Method 1: Subprob                      //
	/* -------------------------------------- */
	timer.restart();
	ier = cufinufft_interp3d(N1, N2, N3, nf1, nf2, nf3, fw, M, x, y, z, cs, tol,
		&dplan);
	FLT tts=timer.elapsedsec();
	if(ier != 0 ){
		cout<<"error: cnufftinterp2d_gpu_subprob"<<endl;
		return 0;
	}
	printf("[method %d] Interp (%ld,%ld,%ld) modes to %ld NU pts in %.3g s \t%.3g NU pts/s\n",
			  method,nf1,nf2,nf3,M,tts,M/tts);
	/* -------------------------------------- */
	// FINUTFFT cpu spreader                  //
	/* -------------------------------------- */
	timer.start();
	setup_spreader(spopts,(FLT)tol,upsampfac,1);
	spopts.pirange=0;
	spopts.chkbnds=1;
	spopts.spread_direction=2;
	spopts.flags=0;//ker always return 1
	spopts.kerpad=1;
	spopts.sort_threads=0;
	spopts.sort=2;
	spopts.debug=0;

	ier = spreadinterp(nf1,nf2,nf3,(FLT*) fw,M,x,y,z,(FLT*) cfinufft,spopts);
	FLT tt=timer.elapsedsec();
	if (ier!=0) {
		printf("error (ier=%d)!\n",ier);
		return ier;
	}
	printf("[finufft] Interp (%ld,%ld,%ld) modes to %ld NU pts in %.3g s \t%.3g NU pts/s\n",
			  nf1,nf2,nf3,M,tt,M/tt);
	err=relerrtwonorm(M,cs,cfinufft);
	printf("|| cs  - cfinufft ||_2 / || cs  ||_2 =  %.6g\n", err);
	FLT cfinufft_infnorm=infnorm(M, cfinufft);

	cout<<"[resultdiff]"<<endl;
	nn = 0;
	for(int i=0; i<M; i++){
		if( norm(cs[i]-cfinufft[i])/cfinufft_infnorm > tol & nn<10){
			cout << cs[i]<<","<<cfinufft[i]<<endl;
			nn++;
		}
	}
	cout<<endl;	
#if 0
	cout<<"[result-hybrid]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % opts.bin_size_y == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % opts.bin_size_x == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fwi[i+j*nf1].real(),fwi[i+j*nf1].imag() );
			//cout<<" "<<setw(8)<<fwfinufft[i+j*nf1];
		}
		cout<<endl;
	}
	cout<<endl;
#endif
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(z);
	hipHostFree(c);
	hipHostFree(fws);
	hipHostFree(fwfinufft);
	return 0;
}
