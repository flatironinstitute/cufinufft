#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/spreadinterp.h"
#include "../finufft/utils.h"
#include "../finufft/spreadinterp.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT sigma = 2.0;
	int N1, N2, M;
	if (argc<5) {
		fprintf(stderr,"Usage: ./spreadinterp [method [nupts_distribute [N1 N2 [M [tol]]]]]\n");
		return 1;
	}  
	int method;
	sscanf(argv[1],"%d",&method);

	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);

	double w;
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	M = N1*N2;// let density always be 1
	if(argc>5){
		sscanf(argv[5],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		if(M==0) M=N1*N2;
	}

	FLT tol=1e-6;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}


	int ier;
	int dim=2;
	int ns=std::ceil(-log10(tol/10.0));
	cufinufft_plan dplan;
	FLT upsampfac=2.0;
	cout<<scientific<<setprecision(3);


	FLT *x, *y;
	CPX *c;
	CPX *fws, *fwfinufft;
	hipHostMalloc(&x, M*sizeof(CPX));
	hipHostMalloc(&y, M*sizeof(CPX));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fws,       nf1*nf2*sizeof(CPX));
	hipHostMalloc(&fwfinufft, nf1*nf2*sizeof(CPX));
#if 0
	// spread a single source, only for reference accuracy check...
	c[0].real(1.0); c[0].imag(0.0);   // unit strength
	x[0] = y[0] = nf1/2.0;                  // at center
	ier = cnufftspread(nf1,nf2,1,(FLT*) fwfinufft,1,x,y,NULL,(FLT*) c,opts);
	if (ier!=0) {
		printf("error when spreading M=1 pt for ref acc check (ier=%d)!\n",ier);
		return ier;
	}
	FLT kersumre = 0.0, kersumim = 0.0;  // sum kernel on uniform grid
	for (int i=0;i<nf1*nf2;++i) {
		kersumre += fwfinufft[i].real();
		kersumim += fwfinufft[i].imag();    // in case the kernel isn't real!
	}
#endif
	FLT strre = 0.0, strim = 0.0;          // also sum the strengths
	switch(nupts_distribute){
		// Making data
		case 1: //uniform
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*randm11(), nf2, 1);
					c[i].real(randm11());
					c[i].imag(randm11());
					strre += c[i].real();
					strim += c[i].imag();
				}
			}
			break;
		case 2: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf2, 1);
					c[i].real(randm11());
					c[i].imag(randm11());
					strre += c[i].real();
					strim += c[i].imag();
				}
			}
			break;
	}

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() <<" s"<<endl<<endl;
#endif

	// Direction 1: Spreading
	printf("[info  ] Type 1: Spreading\n");
#if 0
	FLT sumre = 0.0, sumim = 0.0;   // check spreading accuracy, wrapping
	for (int i=0;i<nf1*nf2;++i) {
		sumre += fwi[i].real();
		sumim += fwi[i].imag();
	}
	FLT pre = kersumre*strre - kersumim*strim;   // pred ans, complex mult
	FLT pim = kersumim*strre + kersumre*strim;
	FLT maxerr = std::max(fabs(sumre-pre), fabs(sumim-pim));
	FLT ansmod = sqrt(sumre*sumre+sumim*sumim);
	printf("    rel err in total over grid:      %.3g\n",maxerr/ansmod);
	// note this is weaker than below dir=2 test, but is good indicator that
	// periodic wrapping is correct
#endif

	/* -------------------------------------- */
	// Method 5: Subprob                     //
	/* -------------------------------------- */
	ier = cufinufft_default_opts(type1, dim, dplan.opts);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	ier = setup_spreader_for_nufft(dplan.spopts, tol, dplan.opts);

	dplan.opts.upsampfac=upsampfac;
	dplan.opts.gpu_method=method;
	dplan.opts.gpu_kerevalmeth=1;
	dplan.spopts.pirange=0;
	switch(method){
		case 2:
		case 3:
		{
			dplan.opts.gpu_binsizex=32;
			dplan.opts.gpu_binsizey=32;
		}
	}
	timer.restart();
	ier = cufinufft_spread2d(N1, N2, nf1, nf2, fws, M, x, y, c, &dplan);
	FLT tsubprob=timer.elapsedsec();
	if(ier != 0 ){
		cout<<"error: cnufftspread2d_gpu_subprob"<<endl;
		return 0;
	}
	printf("[method %d] %ld NU pts to (%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			method,M,N1,N2,nf1*nf2,tsubprob,M/tsubprob);

	/* -------------------------------------- */
	// FINUTFFT cpu spreader                  //
	/* -------------------------------------- */
	timer.start();
	spread_opts spopts;
	setup_spreader(spopts,(FLT)tol,upsampfac,1);
	spopts.pirange=0;
	spopts.chkbnds=1;
	spopts.spread_direction=1;
	spopts.flags=0;//ker always return 1
	spopts.kerpad=1;
	spopts.sort_threads=0;
	spopts.sort=2;
	spopts.debug=0;

	ier = spreadinterp(nf1,nf2,1,(FLT*) fwfinufft,M,x,y,NULL,(FLT*) c,spopts);
	FLT t=timer.elapsedsec();
	if (ier!=0) {
		printf("error (ier=%d)!\n",ier);
		return ier;
	}
	printf("[finufft] %ld NU pts to (%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			M,N1,N2,nf1*nf2,t,M/t);
	//printf("    %.3g NU pts in %.3g s \t%.3g pts/s \t%.3g spread pts/s\n",(double)M,t,M/t,pow(opts.nspread,2)*M/t);
	/* ------------------------------------------------------------------------------------------------------*/

	cout<<endl;
	FLT err=relerrtwonorm(nf1*nf2,fws,fwfinufft);
	err=relerrtwonorm(nf1*nf2,fws,fwfinufft);
	printf("|| fws  - fwfinufft ||_2 / || fws  ||_2 =  %.6g\n", err);

#if 0
	cout<<"[resultdiff]"<<endl;
	FLT fwfinufft_infnorm=infnorm(nf1*nf2, fwfinufft);
	int nn=0;
	for(int j=0; j<nf2; j++){
		for (int i=0; i<nf1; i++){
			if( norm(fwi[i+j*nf1]-fwfinufft[i+j*nf1])/fwfinufft_infnorm > 1e-5 & nn<10){
				cout<<norm(fwi[i+j*nf1]-fwh[i+j*nf1])/fwfinufft_infnorm<<" ";
				cout<<"(i,j)=("<<i<<","<<j<<"), "<<fwi[i+j*nf1] <<","<<fwfinufft[i+j*nf1]<<endl;
				nn++;
			}
		}
	}
	cout<<endl;
#endif
#if 1
	// Direction 2: Interpolation
	printf("\n[info  ] Type 2: Interpolation\n");
	ier = cufinufft_default_opts(type2, dim, dplan.opts);
	ier = setup_spreader_for_nufft(dplan.spopts, tol, dplan.opts);
	dplan.opts.upsampfac=upsampfac;
	dplan.opts.gpu_method=method;
	dplan.opts.gpu_kerevalmeth=1;
	dplan.spopts.pirange=0;
	switch(method){
		case 2:
		case 3:
		{
			dplan.opts.gpu_binsizex=32;
			dplan.opts.gpu_binsizey=32;
		}
	}

	CPX *fw;
	CPX *cfinufft, *cs;
	hipHostMalloc(&fw, nf1*nf2*sizeof(CPX));
	hipHostMalloc(&cfinufft, M*sizeof(CPX));
	hipHostMalloc(&cs,       M*sizeof(CPX));

	for(int i=0; i<nf1*nf2; i++){
	  fw[i].real(1.0);
	  fw[i].imag(0.0);
	}
	/* -------------------------------------- */
	// Method 1: Subprob                      //
	/* -------------------------------------- */
	timer.restart();
	ier = cufinufft_interp2d(N1, N2, nf1, nf2, fw, M, x, y, cs, &dplan);
	FLT tts=timer.elapsedsec();
	if(ier != 0 ){
		cout<<"error: cnufftinterp2d_gpu_subprob"<<endl;
		return 0;
	}
	printf("[method %d] Interp (%ld,%ld) modes to %ld NU pts in %.3g s \t%.3g U pts/s\n",
			  method,nf1,nf2,M,tts,nf1*nf2/tts);
	/* -------------------------------------- */
	// FINUTFFT cpu spreader                  //
	/* -------------------------------------- */
	timer.start();
	setup_spreader(spopts,(FLT)tol,upsampfac,1);
	spopts.pirange=0;
	spopts.chkbnds=1;
	spopts.spread_direction=2;
	spopts.flags=0;//ker always return 1
	spopts.kerpad=1;
	spopts.sort_threads=0;
	spopts.sort=2;
	spopts.debug=0;

	ier = spreadinterp(nf1,nf2,1,(FLT*) fw,M,x,y,NULL,(FLT*) cfinufft,spopts);
	FLT tt=timer.elapsedsec();
	if (ier!=0) {
		printf("error (ier=%d)!\n",ier);
		return ier;
	}
	printf("[finufft] Interp (%ld,%ld) modes to %ld NU pts in %.3g s \t%.3g U pts/s\n",
			  nf1,nf2,M,tt,nf1*nf2/tt);
	err=relerrtwonorm(M,cs,cfinufft);
	printf("|| cs  - cfinufft ||_2 / || cs  ||_2 =  %.6g\n", err);
#if 0
	cout<<"[result-hybrid]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % opts.gpu_binsizey == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % opts.gpu_binsizex == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fwi[i+j*nf1].real(),fwi[i+j*nf1].imag() );
			//cout<<" "<<setw(8)<<fwfinufft[i+j*nf1];
		}
		cout<<endl;
	}
	cout<<endl;
#endif
#endif
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fws);
	hipHostFree(fwfinufft);
	return 0;
}
