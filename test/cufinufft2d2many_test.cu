#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>

#include <cufinufft.h>
#include <profile.h>
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, M;
	int ntransf, maxbatchsize;
	if (argc<4) {
		fprintf(stderr,
			"Usage: cufinufft2d2many_test method N1 N2 [ntransf [maxbatchsize [M [tol]]]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven, or\n"
			"    2: sub-problem.\n"
			"  N1, N2: The size of the 2D array.\n"
			"  ntransf: Number of inputs (default 2 ^ 27 / (N1 * N2)).\n"
			"  maxbatchsize: Number of simultaneous transforms (default min(8, ntransf)).\n"
			"  M: The number of non-uniform points (default N1 * N2).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	M = 2*N1*N2;// let density always be 2
	ntransf = pow(2,28)/M;
	if(argc>4){
		sscanf(argv[4],"%d",&ntransf);
	}

	maxbatchsize = min(8, ntransf);
	if(argc>5){
		sscanf(argv[5],"%d",&maxbatchsize);
	}

	if(argc>6){
		sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;
	


	cout<<scientific<<setprecision(3);
	int ier;

	printf("#modes = %d, #inputs = %d, #NUpts = %d\n", N1*N2, ntransf, M);

	FLT *x, *y;
	CPX *c, *fk;
#if 1
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, ntransf*M*sizeof(CPX));
	hipHostMalloc(&fk,ntransf*N1*N2*sizeof(CPX));
#else
	x = (FLT*) malloc(M*sizeof(FLT));
	y = (FLT*) malloc(M*sizeof(FLT));
	c = (CPX*) malloc(ntransf*M*sizeof(CPX));
	fk = (CPX*) malloc(ntransf*N1*N2*sizeof(CPX));
#endif
	FLT *d_x, *d_y;
	CUCPX *d_c, *d_fk;
	checkCudaErrors(hipMalloc(&d_x,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_y,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_c,ntransf*M*sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_fk,ntransf*N1*N2*sizeof(CUCPX)));

	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
	}

	for(int i=0; i<ntransf*N1*N2; i++){
		fk[i].real(randm11());
		fk[i].imag(randm11());
	}

	checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fk,fk,N1*N2*ntransf*sizeof(CUCPX),hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*warm up gpu*/
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("Warm Up",1);
		char *a;
		checkCudaErrors(hipMalloc(&a,1));
	}
	float milliseconds = 0;
	double totaltime = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tWarm up GPU \t\t %.3g s\n", milliseconds/1000);

	cufinufft_plan dplan;
	int dim = 2;
	int type = 2;
	ier=cufinufft_default_opts(type, dim, &dplan.opts);
	dplan.opts.gpu_method=method;
	dplan.opts.gpu_kerevalmeth=1;

	int nmodes[3];
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = 1;
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_plan",2);
		ier=cufinufft_makeplan(type, dim, nmodes, iflag, ntransf, tol, 
			maxbatchsize, &dplan);
		if (ier!=0){
			printf("err: cufinufft2d_plan\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_setNUpts",3);
		ier=cufinufft_setNUpts(M, d_x, d_y, NULL, 0, NULL, NULL, NULL, &dplan);
		if (ier!=0){
			printf("err: cufinufft2d_setNUpts\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_exec",4);
		ier=cufinufft_exec(d_c, d_fk, &dplan);
		if (ier!=0){
			printf("err: cufinufft2d2_exec\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_destroy",5);
		ier=cufinufft_destroy(&dplan);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);
	// This must be here, since in gpu code, x, y gets modified if pirange=1
	checkCudaErrors(hipMemcpy(c,d_c,M*ntransf*sizeof(CUCPX),hipMemcpyDeviceToHost));
#if 1 
	CPX* fkstart; 
	CPX* cstart;
	for(int t=0; t<ntransf; t++){
		fkstart = fk + t*N1*N2;
		cstart = c + t*M;
		int jt = M/2;          // check arbitrary choice of one targ pt
		CPX J = IMA*(FLT)iflag;
		CPX ct = CPX(0,0);
		int m=0;
		for (int m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
			for (int m1=-(N1/2); m1<=(N1-1)/2; ++m1)
				ct += fkstart[m++] * exp(J*(m1*x[jt] + m2*y[jt]));   // crude direct
		
		printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,abs(cstart[jt]-ct)/infnorm(M,c));
	}
#endif
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		//        if( j % opts.gpu_binsizey == 0)
		//                printf("\n");
		for (int i=0; i<nf1; i++){
			//                if( i % opts.gpu_binsizex == 0 && i!=0)
			//                        printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
#endif	
	printf("[totaltime] %.3g us, speed %.3g NUpts/s\n", totaltime*1000, M*ntransf/totaltime*1000);
#if 1
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fk);
#else
	free(x);
	free(y);
	free(c);
	free(fk);
#endif
	checkCudaErrors(hipDeviceReset());
	return 0;
}
