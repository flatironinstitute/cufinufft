#include "hip/hip_runtime.h"
/* This test should excercise the API
   close to how a user might use the code

   Note this single precision version changes
   doubles ~~> float    and
   cufinufft_* ~~> cufinufftf_* function names.

*/

#include <iostream>
#include <iomanip>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <complex>

#include <cufinufft.h>

#include <profile.h>
#include "../contrib/utils.h"

using namespace std;

typedef std::complex<float> CPX;

int main(int argc, char* argv[])
{
  int N1 = 256;
  int N2 = 256;
  int M = N1*N2;

  float tol=1e-6;

  int iflag=1;

  cout<<scientific<<setprecision(3);
  int ier;

  // malloc host arrays
  float *x, *y;
  CPX *c, *fk;
  checkCudaErrors(hipHostMalloc(&x, M*sizeof(float)));
  checkCudaErrors(hipHostMalloc(&y, M*sizeof(float)));
  checkCudaErrors(hipHostMalloc(&c, M*sizeof(CPX)));
  checkCudaErrors(hipHostMalloc(&fk,N1*N2*sizeof(CPX)));

  // malloc device arrays
  float *d_x, *d_y;
  hipFloatComplex *d_c, *d_fk;
  checkCudaErrors(hipMalloc(&d_x,M*sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y,M*sizeof(float)));
  checkCudaErrors(hipMalloc(&d_c,M*sizeof(hipFloatComplex)));
  checkCudaErrors(hipMalloc(&d_fk,N1*N2*sizeof(hipFloatComplex)));

  // Making data
  for (int i = 0; i < M; i++) {
    x[i] = M_PI*randm11();  // x in [-pi,pi)
    y[i] = M_PI*randm11();
  }
  for(int i=0; i<N1*N2; i++){
    fk[i].real(1.0);
    fk[i].imag(1.0);
  }

  // Copy data to device memory, real users might just populate in memory.
  checkCudaErrors(hipMemcpy(d_x,x,M*sizeof(float),hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y,y,M*sizeof(float),hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_fk, fk, N1*N2*sizeof(CPX),
                             hipMemcpyHostToDevice));


  // construct plan
  cufinufftf_plan dplan;
  int dim = 2;
  int type = 2;

  int nmodes[3];
  int ntransf = 1;
  int maxbatchsize = 1;
  nmodes[0] = N1;
  nmodes[1] = N2;
  nmodes[2] = 1;

  ier=cufinufftf_makeplan(type, dim, nmodes, iflag, ntransf, tol,
                         maxbatchsize, &dplan, NULL);
  if (ier!=0){
    printf("err: cufinufft2d_plan\n");
    return ier;
  }


  // Set Non uniform points
  ier=cufinufftf_setpts(M, d_x, d_y, NULL, 0, NULL, NULL, NULL, dplan);
  if (ier!=0){
    printf("err: cufinufft_setpts\n");
    return ier;
  }

  // Execute the plan on the data
  ier=cufinufftf_execute(d_c, d_fk, dplan);
  if (ier!=0){
    printf("err: cufinufft2d2_exec\n");
    return ier;
  }

  // Destroy the plan when done processing
  ier=cufinufftf_destroy(dplan);
  if (ier!=0){
    printf("err: cufinufft_destroyc\n");
    return ier;
  }

  // Copy test data back to host and compare
  checkCudaErrors(hipMemcpy(c,d_c,M*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
  int jt = M/2;          // check arbitrary choice of one targ pt
  CPX J = IMA*(float)iflag;
  CPX ct = CPX(0,0);
  int m=0;
  for (int m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
    for (int m1=-(N1/2); m1<=(N1-1)/2; ++m1)
      ct += fk[m++] * exp(J*(m1*x[jt] + m2*y[jt]));   // crude direct
  printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,abs(c[jt]-ct)/infnorm(M,c));


  // Cleanup
  checkCudaErrors(hipHostFree(x));
  checkCudaErrors(hipHostFree(y));
  checkCudaErrors(hipHostFree(c));
  checkCudaErrors(hipHostFree(fk));
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_c));
  checkCudaErrors(hipFree(d_fk));

  return 0;
}
