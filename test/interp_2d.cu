#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/cuspreadinterp.h"
#include "../contrib/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT upsampfac=2.0;
	int N1, N2, M;
	if (argc<5) {
		fprintf(stderr,
			"Usage: interp2d method nupts_distr nf1 nf2 [M [tol [kerevalmeth]]]\n"
			"Arguments:\n"
			"  method: One of\n"
			"    1: nupts driven, or\n"
			"    2: sub-problem.\n"
			"  nupts_distr: The distribution of the points; one of\n"
			"    0: uniform, or\n"
			"    1: concentrated in a small region.\n"
			"  nf1, nf2: The size of the 2D array.\n"
			"  M: The number of non-uniform points (default nf1 * nf2 / 4).\n"
			"  tol: NUFFT tolerance (default 1e-6).\n"
			"  kerevalmeth: Kernel evaluation method; one of\n"
			"     0: Exponential of square root, or\n"
			"     1: Horner evaluation (default).\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/upsampfac;
	N2 = (int) nf2/upsampfac;
	M = N1*N2;// let density always be 1
	if(argc>5){
		sscanf(argv[5],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		if(M == 0) M=N1*N2;
	}

	FLT tol=1e-6;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int kerevalmeth=1;
	if(argc>7){
		sscanf(argv[7],"%d",&kerevalmeth);
	}
	int ier;

	int ns=std::ceil(-log10(tol/10.0));
	cufinufft_plan dplan;

	int dim=2;
	ier = cufinufft_default_opts(2, dim, &dplan.opts);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	ier = setup_spreader_for_nufft(dplan.spopts, tol, dplan.opts);
	dplan.opts.gpu_method=method;
	dplan.spopts.pirange=0;
	cout<<scientific<<setprecision(3);


	FLT *x, *y;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*sizeof(CPX));

	dplan.opts.gpu_kerevalmeth=kerevalmeth;
	switch(nupts_distribute){
		// Making data
		case 1: //uniform
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*randm11(), nf2, 1);
				}
			}
			break;
		case 2: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf2, 1);
				}
			}
			break;
	}
	for(int i=0; i<nf1*nf2; i++){
		fw[i].real(1.0);
		fw[i].imag(0.0);
	}

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() <<" s"<<endl<<endl;
#endif

#ifdef INFO
	cout<<"[info  ] Interpolating  ["<<nf1<<"x"<<nf2<<"] uniform points to "<<M<<"nupts"<<endl;
#endif
	timer.restart();
	ier = cufinufft_interp2d(N1, N2, nf1, nf2, fw, M, x, y, c, &dplan);
	if(ier != 0 ){
		cout<<"error: cnufftinterp2d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld U pts to #%d NU pts in %.3g s (\t%.3g U pts/s)\n",
			dplan.opts.gpu_method,nf1*nf2,M,t,nf1*nf2/t);
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<M; j++){
		printf(" (%2.3g,%2.3g)",c[j].real(),c[j].imag() );
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
