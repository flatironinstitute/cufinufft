#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp> 
#include "../../contrib/utils.h"
#include <cuspreadinterp.h>

using namespace std;

#define MAXBINSIZE 1024

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
static __inline__ __device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val +
					__longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN 
		// (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
#endif

static __forceinline__ __device__
FLT evaluate_kernel(FLT x, FLT es_c, FLT es_beta)
	/* ES ("exp sqrt") kernel evaluation at single real argument:
	   phi(x) = exp(beta.sqrt(1 - (2x/n_s)^2)),    for |x| < nspread/2
	   related to an asymptotic approximation to the Kaiser--Bessel, itself an
	   approximation to prolate spheroidal wavefunction (PSWF) of order 0.
	   This is the "reference implementation", used by eg common/onedim_* 
	    2/17/17 */
{
	return exp(es_beta * (sqrt(1.0 - es_c*x*x)));
	//return x;
	//return 1.0;
}

static __inline__ __device__
void eval_kernel_vec_Horner(FLT *ker, const FLT x, const int w, 
	const double upsampfac)
	/* Fill ker[] with Horner piecewise poly approx to [-w/2,w/2] ES kernel eval at
	   x_j = x + j,  for j=0,..,w-1.  Thus x in [-w/2,-w/2+1].   w is aka ns.
	   This is the current evaluation method, since it's faster (except i7 w=16).
	   Two upsampfacs implemented. Params must match ref formula. Barnett 4/24/18 */
{
	FLT z = 2*x + w - 1.0;         // scale so local grid offset z in [-1,1]
	// insert the auto-generated code which expects z, w args, writes to ker...
	if (upsampfac==2.0) {     // floating point equality is fine here
#include "../../contrib/ker_horner_allw_loop.c"
	}
}

static __inline__ __device__
void eval_kernel_vec(FLT *ker, const FLT x, const double w, const double es_c, 
					 const double es_beta)
{
	for(int i=0; i<w; i++){
		ker[i] = evaluate_kernel(abs(x+i), es_c, es_beta);		
	}
}
#if 0
__global__
void RescaleXY_2d(int M, int nf1, int nf2, FLT* x, FLT* y)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
		x[i] = RESCALE(x[i], nf1, 1);
		y[i] = RESCALE(y[i], nf2, 1);
	}
}
#endif
/* ------------------------ 2d Spreading Kernels ----------------------------*/
/* Kernels for NUptsdriven Method */
__global__ 
void TrivialGlobalSortIdx_2d(int M, int* index)
{
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<M; i+=gridDim.x*blockDim.x){
		index[i] = i;
	}
}

__global__
void Spread_2d_NUptsdriven(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, 
		const int ns, int nf1, int nf2, FLT es_c, FLT es_beta, int *idxnupts, 
		int pirange)
{
	int xstart,ystart,xend,yend;
	int xx, yy, ix, iy;
	int outidx;
	FLT ker1[MAX_NSPREAD];
	FLT ker2[MAX_NSPREAD];

	FLT x_rescaled, y_rescaled;
	FLT kervalue1, kervalue2;
	CUCPX cnow;
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
		x_rescaled=RESCALE(x[idxnupts[i]], nf1, pirange);
		y_rescaled=RESCALE(y[idxnupts[i]], nf2, pirange);
		cnow = c[idxnupts[i]];

		xstart = ceil(x_rescaled - ns/2.0);
		ystart = ceil(y_rescaled - ns/2.0);
		xend = floor(x_rescaled + ns/2.0);
		yend = floor(y_rescaled + ns/2.0);

		FLT x1=(FLT)xstart-x_rescaled;
		FLT y1=(FLT)ystart-y_rescaled;
		eval_kernel_vec(ker1,x1,ns,es_c,es_beta);
		eval_kernel_vec(ker2,y1,ns,es_c,es_beta);
		for(yy=ystart; yy<=yend; yy++){
			for(xx=xstart; xx<=xend; xx++){
				ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
				iy = yy < 0 ? yy+nf2 : (yy>nf2-1 ? yy-nf2 : yy);
				outidx = ix+iy*nf1;
				kervalue1=ker1[xx-xstart];
				kervalue2=ker2[yy-ystart];
				atomicAdd(&fw[outidx].x, cnow.x*kervalue1*kervalue2);
				atomicAdd(&fw[outidx].y, cnow.y*kervalue1*kervalue2);
			}
		}

	}

}

__global__
void Spread_2d_NUptsdriven_Horner(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, 
	const int ns, int nf1, int nf2, FLT sigma, int* idxnupts, int pirange)
{
	int xx, yy, ix, iy;
	int outidx;
	FLT ker1[MAX_NSPREAD];
	FLT ker2[MAX_NSPREAD];
	FLT ker1val, ker2val;

	FLT x_rescaled, y_rescaled;
	CUCPX cnow;
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
		x_rescaled=RESCALE(x[idxnupts[i]], nf1, pirange);
		y_rescaled=RESCALE(y[idxnupts[i]], nf2, pirange);
		cnow = c[idxnupts[i]];
		int xstart = ceil(x_rescaled - ns/2.0);
		int ystart = ceil(y_rescaled - ns/2.0);
		int xend = floor(x_rescaled + ns/2.0);
		int yend = floor(y_rescaled + ns/2.0);

		FLT x1=(FLT)xstart-x_rescaled;
		FLT y1=(FLT)ystart-y_rescaled;
		eval_kernel_vec_Horner(ker1,x1,ns,sigma);
		eval_kernel_vec_Horner(ker2,y1,ns,sigma);
		for(yy=ystart; yy<=yend; yy++){
			for(xx=xstart; xx<=xend; xx++){
				ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
				iy = yy < 0 ? yy+nf2 : (yy>nf2-1 ? yy-nf2 : yy);
				outidx = ix+iy*nf1;
				ker1val=ker1[xx-xstart];
				ker2val=ker2[yy-ystart];
				FLT kervalue=ker1val*ker2val;
				atomicAdd(&fw[outidx].x, cnow.x*kervalue);
				atomicAdd(&fw[outidx].y, cnow.y*kervalue);
			}
		}
	}
}

/* Kernels for SubProb Method */
// SubProb properties
__global__
void CalcBinSize_noghost_2d(int M, int nf1, int nf2, int  bin_size_x, 
	int bin_size_y, int nbinx, int nbiny, int* bin_size, FLT *x, FLT *y, 
	int* sortidx, int pirange)
{
	int binidx, binx, biny;
	int oldidx;
	FLT x_rescaled,y_rescaled;
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<M; i+=gridDim.x*blockDim.x){
		x_rescaled=RESCALE(x[i], nf1, pirange);
		y_rescaled=RESCALE(y[i], nf2, pirange);
		binx = floor(x_rescaled/bin_size_x);
		binx = binx >= nbinx ? binx-1 : binx;
		biny = floor(y_rescaled/bin_size_y);
		biny = biny >= nbiny ? biny-1 : biny;
		binidx = binx+biny*nbinx;
		oldidx = atomicAdd(&bin_size[binidx], 1);
		sortidx[i] = oldidx;
		if(binx >= nbinx || biny >= nbiny){
			sortidx[i] = -biny;
		}
	}
}

__global__
void CalcInvertofGlobalSortIdx_2d(int M, int bin_size_x, int bin_size_y, 
	int nbinx,int nbiny, int* bin_startpts, int* sortidx, FLT *x, FLT *y, 
	int* index, int pirange, int nf1, int nf2)
{
	int binx, biny;
	int binidx;
	FLT x_rescaled, y_rescaled;
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<M; i+=gridDim.x*blockDim.x){
		x_rescaled=RESCALE(x[i], nf1, pirange);
		y_rescaled=RESCALE(y[i], nf2, pirange);
		binx = floor(x_rescaled/bin_size_x);
		binx = binx >= nbinx ? binx-1 : binx;
		biny = floor(y_rescaled/bin_size_y);
		biny = biny >= nbiny ? biny-1 : biny;
		binidx = binx+biny*nbinx;

		index[bin_startpts[binidx]+sortidx[i]] = i;
	}
}

__global__
void MapBintoSubProb_2d(int* d_subprob_to_bin,int* d_subprobstartpts, 
	int* d_numsubprob,int numbins)
{
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<numbins; 
		i+=gridDim.x*blockDim.x){
		for(int j=0; j<d_numsubprob[i]; j++){
			d_subprob_to_bin[d_subprobstartpts[i]+j]=i;
		}
	}
}

__global__
void CalcSubProb_2d(int* bin_size, int* num_subprob, int maxsubprobsize, 
	int numbins)
{
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<numbins; 
		i+=gridDim.x*blockDim.x){
		num_subprob[i]=ceil(bin_size[i]/(float) maxsubprobsize);
	}
}

__global__
void Spread_2d_Subprob(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
	int nf1, int nf2, FLT es_c, FLT es_beta, FLT sigma, int* binstartpts,
	int* bin_size, int bin_size_x, int bin_size_y, int* subprob_to_bin,
	int* subprobstartpts, int* numsubprob, int maxsubprobsize, int nbinx, 
	int nbiny, int* idxnupts, int pirange)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,ystart,xend,yend;
	int subpidx=blockIdx.x;
	int bidx=subprob_to_bin[subpidx];
	int binsubp_idx=subpidx-subprobstartpts[bidx];
	int ix, iy;
	int outidx;
	int ptstart=binstartpts[bidx]+binsubp_idx*maxsubprobsize;
	int nupts=min(maxsubprobsize, bin_size[bidx]-binsubp_idx*maxsubprobsize);

	int xoffset=(bidx % nbinx)*bin_size_x;
	int yoffset=(bidx / nbinx)*bin_size_y;

	int N = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0));
	FLT ker1[MAX_NSPREAD];
	FLT ker2[MAX_NSPREAD];
	
	for(int i=threadIdx.x; i<N; i+=blockDim.x){
		fwshared[i].x = 0.0;
		fwshared[i].y = 0.0;
	}
	__syncthreads();

	FLT x_rescaled, y_rescaled;
	CUCPX cnow;
	for(int i=threadIdx.x; i<nupts; i+=blockDim.x){
		int idx = ptstart+i;
		x_rescaled=RESCALE(x[idxnupts[idx]], nf1, pirange);
		y_rescaled=RESCALE(y[idxnupts[idx]], nf2, pirange);
		cnow = c[idxnupts[idx]];

		xstart = ceil(x_rescaled - ns/2.0)-xoffset;
		ystart = ceil(y_rescaled - ns/2.0)-yoffset;
		xend   = floor(x_rescaled + ns/2.0)-xoffset;
		yend   = floor(y_rescaled + ns/2.0)-yoffset;

		FLT x1=(FLT)xstart+xoffset - x_rescaled;
		FLT y1=(FLT)ystart+yoffset - y_rescaled;
		eval_kernel_vec(ker1,x1,ns,es_c,es_beta);
		eval_kernel_vec(ker2,y1,ns,es_c,es_beta);

		for(int yy=ystart; yy<=yend; yy++){
			for(int xx=xstart; xx<=xend; xx++){
				ix = xx+ceil(ns/2.0);
				iy = yy+ceil(ns/2.0);
				if(ix < (bin_size_x + (int) ceil(ns/2.0)*2) && 
				   iy < (bin_size_y + (int) ceil(ns/2.0)*2)){
					outidx = ix+iy*(bin_size_x+ceil(ns/2.0)*2);
					FLT kervalue1 = ker1[xx-xstart];
					FLT kervalue2 = ker2[yy-ystart];
					atomicAdd(&fwshared[outidx].x, cnow.x*kervalue1*kervalue2);
					atomicAdd(&fwshared[outidx].y, cnow.y*kervalue1*kervalue2);
				}
			}
		}
	}
	__syncthreads();
	/* write to global memory */
	for(int k=threadIdx.x; k<N; k+=blockDim.x){
		int i = k % (int) (bin_size_x+2*ceil(ns/2.0) );
		int j = k /( bin_size_x+2*ceil(ns/2.0) );
		ix = xoffset-ceil(ns/2.0)+i;
		iy = yoffset-ceil(ns/2.0)+j;
		if(ix < (nf1+ceil(ns/2.0)) && iy < (nf2+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			iy = iy < 0 ? iy+nf2 : (iy>nf2-1 ? iy-nf2 : iy);
			outidx = ix+iy*nf1;
			int sharedidx=i+j*(bin_size_x+ceil(ns/2.0)*2);
			atomicAdd(&fw[outidx].x, fwshared[sharedidx].x);
			atomicAdd(&fw[outidx].y, fwshared[sharedidx].y);
		}
	}
}

__global__
void Spread_2d_Subprob_Horner(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, 
	const int ns, int nf1, int nf2, FLT sigma, int* binstartpts, int* bin_size, 
	int bin_size_x, int bin_size_y, int* subprob_to_bin, int* subprobstartpts, 
	int* numsubprob, int maxsubprobsize, int nbinx, int nbiny, int* idxnupts, 
	int pirange)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,ystart,xend,yend;
	int subpidx=blockIdx.x;
	int bidx=subprob_to_bin[subpidx];
	int binsubp_idx=subpidx-subprobstartpts[bidx];
	int ix, iy, outidx;
	int ptstart=binstartpts[bidx]+binsubp_idx*maxsubprobsize;
	int nupts=min(maxsubprobsize, bin_size[bidx]-binsubp_idx*maxsubprobsize);

	int xoffset=(bidx % nbinx)*bin_size_x;
	int yoffset=(bidx / nbinx)*bin_size_y;

	int N = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0));
	
	FLT ker1[MAX_NSPREAD];
	FLT ker2[MAX_NSPREAD];


	for(int i=threadIdx.x; i<N; i+=blockDim.x){
		fwshared[i].x = 0.0;
		fwshared[i].y = 0.0;
	}
	__syncthreads();

	FLT x_rescaled, y_rescaled;
	CUCPX cnow;
	for(int i=threadIdx.x; i<nupts; i+=blockDim.x){
		int idx = ptstart+i;
		x_rescaled=RESCALE(x[idxnupts[idx]], nf1, pirange);
		y_rescaled=RESCALE(y[idxnupts[idx]], nf2, pirange);
		cnow = c[idxnupts[idx]];

		xstart = ceil(x_rescaled - ns/2.0)-xoffset;
		ystart = ceil(y_rescaled - ns/2.0)-yoffset;
		xend   = floor(x_rescaled + ns/2.0)-xoffset;
		yend   = floor(y_rescaled + ns/2.0)-yoffset;

		eval_kernel_vec_Horner(ker1,xstart+xoffset-x_rescaled,ns,sigma);
		eval_kernel_vec_Horner(ker2,ystart+yoffset-y_rescaled,ns,sigma);

		for(int yy=ystart; yy<=yend; yy++){
			FLT kervalue2 = ker2[yy-ystart];
			for(int xx=xstart; xx<=xend; xx++){
				ix = xx+ (int) ceil(ns/2.0);
				iy = yy+ (int) ceil(ns/2.0);
				if(ix < (bin_size_x + (int) ceil(ns/2.0)*2) && 
				   iy < (bin_size_y + (int) ceil(ns/2.0)*2)){
					outidx = ix+iy*(bin_size_x+ (int) ceil(ns/2.0)*2);
					FLT kervalue1 = ker1[xx-xstart];
					atomicAdd(&fwshared[outidx].x, cnow.x*kervalue1*kervalue2);
					atomicAdd(&fwshared[outidx].y, cnow.y*kervalue1*kervalue2);
				}
			}
		}
	}
	__syncthreads();

	/* write to global memory */
	for(int k=threadIdx.x; k<N; k+=blockDim.x){
		int i = k % (int) (bin_size_x+2*ceil(ns/2.0) );
		int j = k /( bin_size_x+2*ceil(ns/2.0) );
		ix = xoffset-ceil(ns/2.0)+i;
		iy = yoffset-ceil(ns/2.0)+j;
		if(ix < (nf1+ceil(ns/2.0)) && iy < (nf2+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			iy = iy < 0 ? iy+nf2 : (iy>nf2-1 ? iy-nf2 : iy);
			outidx = ix+iy*nf1;
			int sharedidx=i+j*(bin_size_x+ceil(ns/2.0)*2);
			atomicAdd(&fw[outidx].x, fwshared[sharedidx].x);
			atomicAdd(&fw[outidx].y, fwshared[sharedidx].y);
		}
	}
}

/* Kernels for Paul's Method */
__global__
void LocateFineGridPos_Paul(int M, int nf1, int nf2, int  bin_size_x, 
	int bin_size_y, int nbinx, int nbiny, int* bin_size, int ns, FLT *x, FLT *y, 
	int* sortidx, int* finegridsize, int pirange)
{
	int binidx, binx, biny;
	int oldidx;
	int xidx, yidx, finegrididx;
	FLT x_rescaled,y_rescaled;
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<M; i+=gridDim.x*blockDim.x){
		if(ns%2 == 0){
			x_rescaled=RESCALE(x[i], nf1, pirange);
			y_rescaled=RESCALE(y[i], nf2, pirange);
			binx = floor(floor(x_rescaled)/bin_size_x);
			biny = floor(floor(y_rescaled)/bin_size_y);
			binidx = binx+biny*nbinx;
			xidx = floor(x_rescaled) - binx*bin_size_x;
			yidx = floor(y_rescaled) - biny*bin_size_y;
			finegrididx = binidx*bin_size_x*bin_size_y + xidx + yidx*bin_size_x;
		}else{
			x_rescaled=RESCALE(x[i], nf1, pirange);
			y_rescaled=RESCALE(y[i], nf2, pirange);
			xidx = ceil(x_rescaled - 0.5);
			yidx = ceil(y_rescaled - 0.5);
			
			//xidx = (xidx == nf1) ? (xidx-nf1) : xidx;
			//yidx = (yidx == nf2) ? (yidx-nf2) : yidx;

			binx = floor(xidx/(float) bin_size_x);
			biny = floor(yidx/(float) bin_size_y);
			binidx = binx+biny*nbinx;

			xidx = xidx - binx*bin_size_x;
			yidx = yidx - biny*bin_size_y;
			finegrididx = binidx*bin_size_x*bin_size_y + xidx + yidx*bin_size_x;
		}
		oldidx = atomicAdd(&finegridsize[finegrididx], 1);
		sortidx[i] = oldidx;
	}
}

__global__
void CalcInvertofGlobalSortIdx_Paul(int nf1, int nf2, int M, int bin_size_x, 
		int bin_size_y, int nbinx,int nbiny,int ns, FLT *x, FLT *y, 
		int* finegridstartpts, int* sortidx, int* index, int pirange)
{
	FLT x_rescaled, y_rescaled;
	int binx, biny, binidx, xidx, yidx, finegrididx;
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<M; i+=gridDim.x*blockDim.x){
		if(ns%2 == 0){
			x_rescaled=RESCALE(x[i], nf1, pirange);
			y_rescaled=RESCALE(y[i], nf2, pirange);
			binx = floor(floor(x_rescaled)/bin_size_x);
			biny = floor(floor(y_rescaled)/bin_size_y);
			binidx = binx+biny*nbinx;
			xidx = floor(x_rescaled) - binx*bin_size_x;
			yidx = floor(y_rescaled) - biny*bin_size_y;
			finegrididx = binidx*bin_size_x*bin_size_y + xidx + yidx*bin_size_x;
		}else{
			x_rescaled=RESCALE(x[i], nf1, pirange);
			y_rescaled=RESCALE(y[i], nf2, pirange);
			xidx = ceil(x_rescaled - 0.5);
			yidx = ceil(y_rescaled - 0.5);
			
			xidx = (xidx == nf1) ? xidx - nf1 : xidx;
			yidx = (yidx == nf2) ? yidx - nf2 : yidx;

			binx = floor(xidx/(float) bin_size_x);
			biny = floor(yidx/(float) bin_size_y);
			binidx = binx+biny*nbinx;

			xidx = xidx - binx*bin_size_x;
			yidx = yidx - biny*bin_size_y;
			finegrididx = binidx*bin_size_x*bin_size_y + xidx + yidx*bin_size_x;
		}
		index[finegridstartpts[finegrididx]+sortidx[i]] = i;
	}
}

__global__
void CalcSubProb_2d_Paul(int* finegridsize, int* num_subprob, 
	int maxsubprobsize)
{
	typedef hipcub::BlockReduce<int, 1024> BlockReduce; // how to fix this...?
	__shared__ typename BlockReduce::TempStorage temp_storage;
	
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	int aggregate = BlockReduce(temp_storage).Reduce(finegridsize[i], 
			hipcub::Max());
	
	num_subprob[blockIdx.x] = (int)ceil(aggregate/(float) maxsubprobsize);
	//num_subprob[blockIdx.x] = aggregate;
}

__global__
void Spread_2d_Subprob_Paul(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, 
	const int ns, int nf1, int nf2, FLT es_c, FLT es_beta, FLT sigma, 
	int* binstartpts, int* bin_size, int bin_size_x, int bin_size_y, 
	int* subprob_to_bin, int* subprobstartpts, int* numsubprob, 
	int maxsubprobsize, int nbinx, int nbiny, int* idxnupts, int* fgstartpts,
	int* finegridsize, int pirange)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,ystart,xend,yend;
	int subpidx=blockIdx.x;
	int bidx=subprob_to_bin[subpidx];
	int binsubp_idx=subpidx-subprobstartpts[bidx];

	int ix,iy,outidx;

	int xoffset=(bidx % nbinx)*bin_size_x;
	int yoffset=(bidx / nbinx)*bin_size_y;

	int N = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0));
#if 0
	FLT ker1[MAX_NSPREAD*10];
    FLT ker2[MAX_NSPREAD*10];
#endif
	for(int i=threadIdx.x; i<N; i+=blockDim.x){
		fwshared[i].x = 0.0;
		fwshared[i].y = 0.0;
	}
	__syncthreads();

	FLT x_rescaled, y_rescaled;
	for(int i=threadIdx.x; i<bin_size_x*bin_size_y; i+=blockDim.x){
		int fineidx = bidx*bin_size_x*bin_size_y+i;
		int idxstart = fgstartpts[fineidx]+binsubp_idx*maxsubprobsize;
		int nupts = min(maxsubprobsize,finegridsize[fineidx]-binsubp_idx*
			maxsubprobsize);
		if(nupts > 0){
			x_rescaled = x[idxnupts[idxstart]];
			y_rescaled = y[idxnupts[idxstart]];

			xstart = ceil(x_rescaled - ns/2.0)-xoffset;
			ystart = ceil(y_rescaled - ns/2.0)-yoffset;
			xend   = floor(x_rescaled + ns/2.0)-xoffset;
			yend   = floor(y_rescaled + ns/2.0)-yoffset;
#if 0
			for(int m=0; m<nupts; m++){
				int idx = idxstart+m;
				x_rescaled=RESCALE(x[idxnupts[idx]], nf1, pirange);
				y_rescaled=RESCALE(y[idxnupts[idx]], nf2, pirange);

				eval_kernel_vec_Horner(ker1+m*MAX_NSPREAD,xstart+xoffset-
					x_rescaled,ns,sigma);
				eval_kernel_vec_Horner(ker2+m*MAX_NSPREAD,ystart+yoffset-
					y_rescaled,ns,sigma);
			}
#endif
			for(int yy=ystart; yy<=yend; yy++){
				FLT kervalue2[10];
				for(int m=0; m<nupts; m++){
					int idx = idxstart+m;
#if 1 
					y_rescaled=RESCALE(y[idxnupts[idx]], nf2, pirange);
					FLT disy = abs(y_rescaled-(yy+yoffset));
					kervalue2[m] = evaluate_kernel(disy, es_c, es_beta);
#else
					kervalue2[m] = ker2[m*MAX_NSPREAD+yy-ystart];
#endif
				}
				for(int xx=xstart; xx<=xend; xx++){
					ix = xx+ceil(ns/2.0);
					iy = yy+ceil(ns/2.0);
					outidx = ix+iy*(bin_size_x+ceil(ns/2.0)*2);
					CUCPX updatevalue;
					updatevalue.x = 0.0;
					updatevalue.y = 0.0;
					for(int m=0; m<nupts; m++){
						int idx = idxstart+m;
#if 1
						x_rescaled=RESCALE(x[idxnupts[idx]], nf1, pirange);
						FLT disx = abs(x_rescaled-(xx+xoffset));
						FLT kervalue1 = evaluate_kernel(disx, es_c, es_beta);

						updatevalue.x += kervalue2[m]*kervalue1*
										 c[idxnupts[idx]].x;
						updatevalue.y += kervalue2[m]*kervalue1*
										 c[idxnupts[idx]].y;
#else
						FLT kervalue1 = ker1[m*MAX_NSPREAD+xx-xstart];
						updatevalue.x += kervalue1*kervalue2[m]*
							c[idxnupts[idx]].x;
						updatevalue.y += kervalue1*kervalue2[m]*
							c[idxnupts[idx]].y;
#endif
					}
					atomicAdd(&fwshared[outidx].x, updatevalue.x);
					atomicAdd(&fwshared[outidx].y, updatevalue.y);
				}
			}
		}
	}
	__syncthreads();

	/* write to global memory */
	for(int k=threadIdx.x; k<N; k+=blockDim.x){
		int i = k % (int) (bin_size_x+2*ceil(ns/2.0) );
		int j = k /( bin_size_x+2*ceil(ns/2.0) );
		ix = xoffset-ceil(ns/2.0)+i;
		iy = yoffset-ceil(ns/2.0)+j;
		if(ix < (nf1+ceil(ns/2.0)) && iy < (nf2+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			iy = iy < 0 ? iy+nf2 : (iy>nf2-1 ? iy-nf2 : iy);
			outidx = ix+iy*nf1;
			int sharedidx=i+j*(bin_size_x+ceil(ns/2.0)*2);
			atomicAdd(&fw[outidx].x, fwshared[sharedidx].x);
			atomicAdd(&fw[outidx].y, fwshared[sharedidx].y);
		}
	}
}
/* --------------------- 2d Interpolation Kernels ----------------------------*/
/* Kernels for NUptsdriven Method */
__global__
void Interp_2d_NUptsdriven(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
		       int nf1, int nf2, FLT es_c, FLT es_beta, int* idxnupts, int pirange)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){

		FLT x_rescaled=RESCALE(x[idxnupts[i]], nf1, pirange);
		FLT y_rescaled=RESCALE(y[idxnupts[i]], nf2, pirange);
		int xstart = ceil(x_rescaled - ns/2.0);
		int ystart = ceil(y_rescaled - ns/2.0);
		int xend = floor(x_rescaled + ns/2.0);
		int yend = floor(y_rescaled + ns/2.0);
		CUCPX cnow;
		cnow.x = 0.0;
		cnow.y = 0.0;
		for(int yy=ystart; yy<=yend; yy++){
			FLT disy=abs(y_rescaled-yy);
			FLT kervalue2 = evaluate_kernel(disy, es_c, es_beta);
			for(int xx=xstart; xx<=xend; xx++){
				int ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
				int iy = yy < 0 ? yy+nf2 : (yy>nf2-1 ? yy-nf2 : yy);
				int inidx = ix+iy*nf1;
				FLT disx=abs(x_rescaled-xx);
				FLT kervalue1 = evaluate_kernel(disx, es_c, es_beta);
				cnow.x += fw[inidx].x*kervalue1*kervalue2;
				cnow.y += fw[inidx].y*kervalue1*kervalue2;
			}
		}
		c[idxnupts[i]].x = cnow.x;
		c[idxnupts[i]].y = cnow.y;
	}

}

__global__
void Interp_2d_NUptsdriven_Horner(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, 
	const int ns, int nf1, int nf2, FLT sigma, int* idxnupts, int pirange)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
		FLT x_rescaled=RESCALE(x[idxnupts[i]], nf1, pirange);
		FLT y_rescaled=RESCALE(y[idxnupts[i]], nf2, pirange);

		int xstart = ceil(x_rescaled - ns/2.0);
		int ystart = ceil(y_rescaled - ns/2.0);
		int xend = floor(x_rescaled + ns/2.0);
		int yend = floor(y_rescaled + ns/2.0);

		CUCPX cnow;
		cnow.x = 0.0;
		cnow.y = 0.0;
		FLT ker1[MAX_NSPREAD];
		FLT ker2[MAX_NSPREAD];

		eval_kernel_vec_Horner(ker1,xstart-x_rescaled,ns,sigma);
        eval_kernel_vec_Horner(ker2,ystart-y_rescaled,ns,sigma);

		for(int yy=ystart; yy<=yend; yy++){
			FLT disy=abs(y_rescaled-yy);
			FLT kervalue2 = ker2[yy-ystart];
			for(int xx=xstart; xx<=xend; xx++){
				int ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
				int iy = yy < 0 ? yy+nf2 : (yy>nf2-1 ? yy-nf2 : yy);
				int inidx = ix+iy*nf1;
				FLT disx=abs(x_rescaled-xx);
				FLT kervalue1 = ker1[xx-xstart];
				cnow.x += fw[inidx].x*kervalue1*kervalue2;
				cnow.y += fw[inidx].y*kervalue1*kervalue2;
			}
		}
		c[idxnupts[i]].x = cnow.x;
		c[idxnupts[i]].y = cnow.y;
	}

}

/* Kernels for Subprob Method */
__global__
void Interp_2d_Subprob(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
	int nf1, int nf2, FLT es_c, FLT es_beta, FLT sigma, int* binstartpts,
	int* bin_size, int bin_size_x, int bin_size_y, int* subprob_to_bin,
	int* subprobstartpts, int* numsubprob, int maxsubprobsize, int nbinx, 
	int nbiny, int* idxnupts, int pirange)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,ystart,xend,yend;
	int subpidx=blockIdx.x;
	int bidx=subprob_to_bin[subpidx];
	int binsubp_idx=subpidx-subprobstartpts[bidx];
	int ix, iy;
	int outidx;
	int ptstart=binstartpts[bidx]+binsubp_idx*maxsubprobsize;
	int nupts=min(maxsubprobsize, bin_size[bidx]-binsubp_idx*maxsubprobsize);

	int xoffset=(bidx % nbinx)*bin_size_x;
	int yoffset=(bidx / nbinx)*bin_size_y;
	int N = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0));

	for(int k=threadIdx.x;k<N; k+=blockDim.x){
		int i = k % (int) (bin_size_x+2*ceil(ns/2.0) );
		int j = k /( bin_size_x+2*ceil(ns/2.0) );
		ix = xoffset-ceil(ns/2.0)+i;
		iy = yoffset-ceil(ns/2.0)+j;
		if(ix < (nf1+ceil(ns/2.0)) && iy < (nf2+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			iy = iy < 0 ? iy+nf2 : (iy>nf2-1 ? iy-nf2 : iy);
			outidx = ix+iy*nf1;
			int sharedidx=i+j*(bin_size_x+ceil(ns/2.0)*2);
			fwshared[sharedidx].x = fw[outidx].x;
			fwshared[sharedidx].y = fw[outidx].y;
		}
	}
	__syncthreads();

	FLT x_rescaled, y_rescaled;
	CUCPX cnow;
	for(int i=threadIdx.x; i<nupts; i+=blockDim.x){
		int idx = ptstart+i;
		x_rescaled=RESCALE(x[idxnupts[idx]], nf1, pirange);
		y_rescaled=RESCALE(y[idxnupts[idx]], nf2, pirange);
		cnow.x = 0.0;
		cnow.y = 0.0;

		xstart = ceil(x_rescaled - ns/2.0)-xoffset;
		ystart = ceil(y_rescaled - ns/2.0)-yoffset;
		xend   = floor(x_rescaled + ns/2.0)-xoffset;
		yend   = floor(y_rescaled + ns/2.0)-yoffset;

		for(int yy=ystart; yy<=yend; yy++){
			FLT disy=abs(y_rescaled-(yy+yoffset));
			FLT kervalue2 = evaluate_kernel(disy, es_c, es_beta);
			for(int xx=xstart; xx<=xend; xx++){
				ix = xx+ceil(ns/2.0);
				iy = yy+ceil(ns/2.0);
				outidx = ix+iy*(bin_size_x+ceil(ns/2.0)*2);
				FLT disx=abs(x_rescaled-(xx+xoffset));
				//FLT kervalue1 = ker1[xx-xstart];
				FLT kervalue1 = evaluate_kernel(disx, es_c, es_beta);
				cnow.x += fwshared[outidx].x*kervalue1*kervalue2;
				cnow.y += fwshared[outidx].y*kervalue1*kervalue2;
			}
		}
		c[idxnupts[idx]] = cnow;
	}
}

__global__
void Interp_2d_Subprob_Horner(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, 
	const int ns, int nf1, int nf2, FLT sigma, int* binstartpts, int* bin_size, 
	int bin_size_x, int bin_size_y, int* subprob_to_bin, int* subprobstartpts, 
	int* numsubprob, int maxsubprobsize, int nbinx, int nbiny, int* idxnupts, 
	int pirange)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,ystart,xend,yend;
	int subpidx=blockIdx.x;
	int bidx=subprob_to_bin[subpidx];
	int binsubp_idx=subpidx-subprobstartpts[bidx];
	int ix, iy;
	int outidx;
	int ptstart=binstartpts[bidx]+binsubp_idx*maxsubprobsize;
	int nupts=min(maxsubprobsize, bin_size[bidx]-binsubp_idx*maxsubprobsize);

	int xoffset=(bidx % nbinx)*bin_size_x;
	int yoffset=(bidx / nbinx)*bin_size_y;

	int N = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0));


	for(int k=threadIdx.x;k<N; k+=blockDim.x){
		int i = k % (int) (bin_size_x+2*ceil(ns/2.0) );
		int j = k /( bin_size_x+2*ceil(ns/2.0) );
		ix = xoffset-ceil(ns/2.0)+i;
		iy = yoffset-ceil(ns/2.0)+j;
		if(ix < (nf1+ceil(ns/2.0)) && iy < (nf2+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			iy = iy < 0 ? iy+nf2 : (iy>nf2-1 ? iy-nf2 : iy);
			outidx = ix+iy*nf1;
			int sharedidx=i+j*(bin_size_x+ceil(ns/2.0)*2);
			fwshared[sharedidx].x = fw[outidx].x;
			fwshared[sharedidx].y = fw[outidx].y;
		}
	}
	__syncthreads();

	FLT ker1[MAX_NSPREAD];
	FLT ker2[MAX_NSPREAD];

	FLT x_rescaled, y_rescaled;
	CUCPX cnow;
	for(int i=threadIdx.x; i<nupts; i+=blockDim.x){
		int idx = ptstart+i;
		x_rescaled=RESCALE(x[idxnupts[idx]], nf1, pirange);
		y_rescaled=RESCALE(y[idxnupts[idx]], nf2, pirange);
		cnow.x = 0.0;
		cnow.y = 0.0;

		xstart = ceil(x_rescaled - ns/2.0)-xoffset;
		ystart = ceil(y_rescaled - ns/2.0)-yoffset;
		xend   = floor(x_rescaled + ns/2.0)-xoffset;
		yend   = floor(y_rescaled + ns/2.0)-yoffset;

		eval_kernel_vec_Horner(ker1,xstart+xoffset-x_rescaled,ns,sigma);
		eval_kernel_vec_Horner(ker2,ystart+yoffset-y_rescaled,ns,sigma);
		
		for(int yy=ystart; yy<=yend; yy++){
			FLT disy=abs(y_rescaled-(yy+yoffset));
			FLT kervalue2 = ker2[yy-ystart];
			for(int xx=xstart; xx<=xend; xx++){
				ix = xx+ceil(ns/2.0);
				iy = yy+ceil(ns/2.0);
				outidx = ix+iy*(bin_size_x+ceil(ns/2.0)*2);
		
				FLT kervalue1 = ker1[xx-xstart];
				cnow.x += fwshared[outidx].x*kervalue1*kervalue2;
				cnow.y += fwshared[outidx].y*kervalue1*kervalue2;
			}
		}
		c[idxnupts[idx]] = cnow;
	}
}
#if 0
__global__
void PtsRearrange_noghost_2d(int M, int nf1, int nf2, int bin_size_x, 
		int bin_size_y, int nbinx,int nbiny, int* bin_startpts, int* sortidx, 
		FLT *x, FLT *x_sorted, FLT *y, FLT *y_sorted, CUCPX *c, CUCPX *c_sorted)
{
	//int i = blockDim.x*blockIdx.x + threadIdx.x;
	int binx, biny;
	int binidx;
	FLT x_rescaled, y_rescaled;
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<M; i+=gridDim.x*blockDim.x){
		x_rescaled=RESCALE(x[i], nf1, 1);
		y_rescaled=RESCALE(y[i], nf2, 1);
		binx = floor(x_rescaled/bin_size_x);
		biny = floor(y_rescaled/bin_size_y);
		binidx = binx+biny*nbinx;

		x_sorted[bin_startpts[binidx]+sortidx[i]] = x_rescaled;
		y_sorted[bin_startpts[binidx]+sortidx[i]] = y_rescaled;
		c_sorted[bin_startpts[binidx]+sortidx[i]] = c[i];
	}
}

__global__
void CreateSortIdx(int M, int nf1, int nf2, FLT *x, FLT *y, int* sortidx)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	FLT x_rescaled,y_rescaled;
	if (i < M){
		//x_rescaled = RESCALE(x[i],nf1,1);
		//y_rescaled = RESCALE(y[i],nf2,1);
		x_rescaled=RESCALE(x[i], nf1, 1);
		y_rescaled=RESCALE(y[i], nf2, 1);
		sortidx[i] = floor(x_rescaled) + floor(y_rescaled)*nf1;
	}
}

__global__
void Spread_2d_Simple(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
		int nf1, int nf2, FLT es_c, FLT es_beta, int bin_size,int bin_size_x, 
		int bin_size_y, int binx, int biny)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,ystart,xend,yend;
	int xx, yy, ix, iy;
	int outidx;
	int ptstart=0;

	int xoffset=binx*bin_size_x;
	int yoffset=biny*bin_size_y;

	int N = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0));
	for(int i=threadIdx.x+threadIdx.y*blockDim.x; i<N; 
		i+=blockDim.x*blockDim.y){
		fwshared[i].x = 0.0;
		fwshared[i].y = 0.0;
	}
	__syncthreads();

	FLT x_rescaled, y_rescaled;
	for(int i=threadIdx.x+threadIdx.y*blockDim.x; i<bin_size; 
		i+=blockDim.x*blockDim.y){
		int idx=ptstart+i;
		x_rescaled=x[idx];
		y_rescaled=y[idx];
		xstart = ceil(x_rescaled - ns/2.0)-xoffset;
		ystart = ceil(y_rescaled - ns/2.0)-yoffset;
		xend = floor(x_rescaled + ns/2.0)-xoffset;
		yend = floor(y_rescaled + ns/2.0)-yoffset;
		for(yy=ystart; yy<=yend; yy++){
			FLT disy=abs(y_rescaled-(yy+yoffset));
			FLT kervalue2 = evaluate_kernel(disy, es_c, es_beta);
			for(xx=xstart; xx<=xend; xx++){
				ix = xx+ceil(ns/2.0);
				iy = yy+ceil(ns/2.0);
				outidx = ix+iy*(bin_size_x+ceil(ns/2.0)*2);
				FLT disx=abs(x_rescaled-(xx+xoffset));
				FLT kervalue1 = evaluate_kernel(disx, es_c, es_beta);
				atomicAdd(&fwshared[outidx].x, c[idx].x*kervalue1*kervalue2);
				atomicAdd(&fwshared[outidx].y, c[idx].y*kervalue1*kervalue2);
			}
		}
	}
	__syncthreads();

	/* write to global memory */
	for(int k=threadIdx.x+threadIdx.y*blockDim.x; k<N; 
		k+=blockDim.x*blockDim.y){
		int i = k % (int) (bin_size_x+2*ceil(ns/2.0) );
		int j = k /( bin_size_x+2*ceil(ns/2.0) );
		ix = xoffset+i-ceil(ns/2.0);
		iy = yoffset+j-ceil(ns/2.0);
		if(ix < (nf1+ceil(ns/2.0)) && iy < (nf2+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			iy = iy < 0 ? iy+nf2 : (iy>nf2-1 ? iy-nf2 : iy);
			outidx = ix+iy*nf1;
			int sharedidx=i+j*(bin_size_x+ceil(ns/2.0)*2);
			atomicAdd(&fw[outidx].x, fwshared[sharedidx].x);
			atomicAdd(&fw[outidx].y, fwshared[sharedidx].y);
			//atomicAdd(&fw[outidx].x, y_rescaled);
			//atomicAdd(&fw[outidx].y, ystart+ceil(ns/2.0));
		}
	}
}

__global__
void Spread_2d_Hybrid(FLT *x, FLT *y, CUCPX *c, CUCPX *fw, int M, const int ns,
		int nf1, int nf2, FLT es_c, FLT es_beta, int* binstartpts,
		int* bin_size, int bin_size_x, int bin_size_y)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,ystart,xend,yend;
	int bidx=blockIdx.x+blockIdx.y*gridDim.x;
	int xx, yy, ix, iy;
	int outidx;
	int ptstart=binstartpts[bidx];

	int xoffset=blockIdx.x*bin_size_x;
	int yoffset=blockIdx.y*bin_size_y;

	int N = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0));
	for(int i=threadIdx.x+threadIdx.y*blockDim.x; i<N; 
		i+=blockDim.x*blockDim.y){
		fwshared[i].x = 0.0;
		fwshared[i].y = 0.0;
	}
	__syncthreads();

	FLT x_rescaled, y_rescaled;
	for(int i=threadIdx.x+threadIdx.y*blockDim.x; i<bin_size[bidx]; 
		i+=blockDim.x*blockDim.y){
		int idx=ptstart+i;
		x_rescaled=x[idx];
		y_rescaled=y[idx];
		xstart = ceil(x_rescaled - ns/2.0)-xoffset;
		ystart = ceil(y_rescaled - ns/2.0)-yoffset;
		xend = floor(x_rescaled + ns/2.0)-xoffset;
		yend = floor(y_rescaled + ns/2.0)-yoffset;

		for(yy=ystart; yy<=yend; yy++){
			FLT disy=abs(y_rescaled-(yy+yoffset));
			FLT kervalue2 = evaluate_kernel(disy, es_c, es_beta);
			for(xx=xstart; xx<=xend; xx++){
				ix = xx+ceil(ns/2.0);
				iy = yy+ceil(ns/2.0);
				outidx = ix+iy*(bin_size_x+ceil(ns/2.0)*2);
				FLT disx=abs(x_rescaled-(xx+xoffset));
				FLT kervalue1 = evaluate_kernel(disx, es_c, es_beta);
				//fwshared[outidx].x += kervalue1*kervalue2;
				//fwshared[outidx].y += kervalue1*kervalue2;
				atomicAdd(&fwshared[outidx].x, c[idx].x*kervalue1*kervalue2);
				atomicAdd(&fwshared[outidx].y, c[idx].y*kervalue1*kervalue2);
				//atomicAdd(&fwshared[outidx].x, kervalue1*kervalue2);
				//atomicAdd(&fwshared[outidx].y, kervalue1*kervalue2);
			}
		}
	}
	__syncthreads();
	/* write to global memory */
	for(int k=threadIdx.x+threadIdx.y*blockDim.x; k<N; 
		k+=blockDim.x*blockDim.y){
		int i = k % (int) (bin_size_x+2*ceil(ns/2.0) );
		int j = k /( bin_size_x+2*ceil(ns/2.0) );
		ix = xoffset+i-ceil(ns/2.0);
		iy = yoffset+j-ceil(ns/2.0);
		if(ix < (nf1+ceil(ns/2.0)) && iy < (nf2+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			iy = iy < 0 ? iy+nf2 : (iy>nf2-1 ? iy-nf2 : iy);
			outidx = ix+iy*nf1;
			int sharedidx=i+j*(bin_size_x+ceil(ns/2.0)*2);
			atomicAdd(&fw[outidx].x, fwshared[sharedidx].x);
			atomicAdd(&fw[outidx].y, fwshared[sharedidx].y);
		}
	}
}
#endif


