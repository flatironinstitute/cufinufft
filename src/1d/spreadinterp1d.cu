#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <thrust/extrema.h>
#include "../../contrib/utils.h"
#include "../../contrib/utils_fp.h"
#include "../cuspreadinterp.h"
#include "../../include/utils.h"

using namespace std;

/* ------------------------ 1d Spreading Kernels ----------------------------*/
/* Kernels for NUptsdriven Method */

__global__
void Spread_1d_NUptsdriven(FLT *x, CUCPX *c, CUCPX *fw, int M, const int ns, 
	int nf1, FLT es_c, FLT es_beta, int *idxnupts, int pirange)
{
	int xstart,xend;
	int xx, ix;
	FLT ker1[MAX_NSPREAD];

	FLT x_rescaled;
	FLT kervalue1;
	CUCPX cnow;
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
		x_rescaled=RESCALE(x[idxnupts[i]], nf1, pirange);
		cnow = c[idxnupts[i]];

		xstart = ceil(x_rescaled - ns/2.0);
		xend  = floor(x_rescaled + ns/2.0);

		FLT x1=(FLT)xstart-x_rescaled;
		eval_kernel_vec(ker1,x1,ns,es_c,es_beta);
		for(xx=xstart; xx<=xend; xx++){
			ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
			kervalue1=ker1[xx-xstart];
			atomicAdd(&fw[ix].x, cnow.x*kervalue1);
			atomicAdd(&fw[ix].y, cnow.y*kervalue1);
		}
	}

}

__global__
void Spread_1d_NUptsdriven_Horner(FLT *x, CUCPX *c, CUCPX *fw, int M, 
	const int ns, int nf1, FLT sigma, int* idxnupts, int pirange)
{
	int xx, ix;
	FLT ker1[MAX_NSPREAD];

	FLT x_rescaled;
	CUCPX cnow;
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
		x_rescaled=RESCALE(x[idxnupts[i]], nf1, pirange);
		cnow = c[idxnupts[i]];
		int xstart = ceil(x_rescaled - ns/2.0);
		int xend  = floor(x_rescaled + ns/2.0);

		FLT x1=(FLT)xstart-x_rescaled;
		eval_kernel_vec_Horner(ker1,x1,ns,sigma);
		for(xx=xstart; xx<=xend; xx++){
			ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
			FLT kervalue=ker1[xx-xstart];
			atomicAdd(&fw[ix].x, cnow.x*kervalue);
			atomicAdd(&fw[ix].y, cnow.y*kervalue);
		}
	}
}

/* Kernels for SubProb Method */
// SubProb properties
__global__
void CalcBinSize_noghost_1d(int M, int nf1, int  bin_size_x, int nbinx, 
	int* bin_size, FLT *x, int* sortidx, int pirange)
{
	int binx;
	int oldidx;
	FLT x_rescaled;
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<M; i+=gridDim.x*blockDim.x){
		x_rescaled=RESCALE(x[i], nf1, pirange);
		binx = floor(x_rescaled/bin_size_x);
		binx = binx >= nbinx ? binx-1 : binx;
		binx = binx < 0 ? 0 : binx;
		oldidx = atomicAdd(&bin_size[binx], 1);
		sortidx[i] = oldidx;
		if(binx >= nbinx){
			sortidx[i] = -binx;
		}
	}
}

__global__
void CalcInvertofGlobalSortIdx_1d(int M, int bin_size_x, int nbinx, 
	int* bin_startpts, int* sortidx, FLT *x, int* index, int pirange, int nf1)
{
	int binx;
	FLT x_rescaled;
	for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<M; i+=gridDim.x*blockDim.x){
		x_rescaled=RESCALE(x[i], nf1, pirange);
		binx = floor(x_rescaled/bin_size_x);
		binx = binx >= nbinx ? binx-1 : binx;
		binx = binx < 0 ? 0 : binx;

		index[bin_startpts[binx]+sortidx[i]] = i;
	}
}


__global__
void Spread_1d_Subprob(FLT *x, CUCPX *c, CUCPX *fw, int M, const int ns,
	int nf1, FLT es_c, FLT es_beta, FLT sigma, int* binstartpts,
	int* bin_size, int bin_size_x, int* subprob_to_bin,
	int* subprobstartpts, int* numsubprob, int maxsubprobsize, int nbinx, 
	int* idxnupts, int pirange)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,xend;
	int subpidx=blockIdx.x;
	int bidx=subprob_to_bin[subpidx];
	int binsubp_idx=subpidx-subprobstartpts[bidx];
	int ix;
	int ptstart=binstartpts[bidx]+binsubp_idx*maxsubprobsize;
	int nupts=min(maxsubprobsize, bin_size[bidx]-binsubp_idx*maxsubprobsize);

	int xoffset=(bidx % nbinx)*bin_size_x;

	int N = (bin_size_x+2*ceil(ns/2.0));
	FLT ker1[MAX_NSPREAD];
	
	for(int i=threadIdx.x; i<N; i+=blockDim.x){
		fwshared[i].x = 0.0;
		fwshared[i].y = 0.0;
	}
	__syncthreads();

	FLT x_rescaled;
	CUCPX cnow;
	for(int i=threadIdx.x; i<nupts; i+=blockDim.x){
		int idx = ptstart+i;
		x_rescaled=RESCALE(x[idxnupts[idx]], nf1, pirange);
		cnow = c[idxnupts[idx]];

		xstart = ceil(x_rescaled - ns/2.0)-xoffset;
		xend   = floor(x_rescaled + ns/2.0)-xoffset;

		FLT x1=(FLT)xstart+xoffset - x_rescaled;
		eval_kernel_vec(ker1,x1,ns,es_c,es_beta);

		for(int xx=xstart; xx<=xend; xx++){
			ix = xx+ceil(ns/2.0);
			if(ix >= (bin_size_x + (int) ceil(ns/2.0)*2) || ix<0) break;
			atomicAdd(&fwshared[ix].x, cnow.x*ker1[xx-xstart]);
			atomicAdd(&fwshared[ix].y, cnow.y*ker1[xx-xstart]);
		}
	}
	__syncthreads();
	/* write to global memory */
	for(int k=threadIdx.x; k<N; k+=blockDim.x){
		ix = xoffset-ceil(ns/2.0)+k;
		if(ix < (nf1+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			atomicAdd(&fw[ix].x, fwshared[k].x);
			atomicAdd(&fw[ix].y, fwshared[k].y);
		}
	}
}

__global__
void Spread_1d_Subprob_Horner(FLT *x, CUCPX *c, CUCPX *fw, int M, 
	const int ns, int nf1, FLT sigma, int* binstartpts, int* bin_size, 
	int bin_size_x, int* subprob_to_bin, int* subprobstartpts, 
	int* numsubprob, int maxsubprobsize, int nbinx, int* idxnupts, int pirange)
{
	extern __shared__ CUCPX fwshared[];

	int xstart,xend;
	int subpidx=blockIdx.x;
	int bidx=subprob_to_bin[subpidx];
	int binsubp_idx=subpidx-subprobstartpts[bidx];
	int ix;
	int ptstart=binstartpts[bidx]+binsubp_idx*maxsubprobsize;
	int nupts=min(maxsubprobsize, bin_size[bidx]-binsubp_idx*maxsubprobsize);

	int xoffset=(bidx % nbinx)*bin_size_x;

	int N = (bin_size_x+2*ceil(ns/2.0));
	
	FLT ker1[MAX_NSPREAD];

	for(int i=threadIdx.x; i<N; i+=blockDim.x){
		fwshared[i].x = 0.0;
		fwshared[i].y = 0.0;
	}
	__syncthreads();

	FLT x_rescaled;
	CUCPX cnow;
	for(int i=threadIdx.x; i<nupts; i+=blockDim.x){
		int idx = ptstart+i;
		x_rescaled=RESCALE(x[idxnupts[idx]], nf1, pirange);
		cnow = c[idxnupts[idx]];

		xstart = ceil(x_rescaled - ns/2.0)-xoffset;
		xend  = floor(x_rescaled + ns/2.0)-xoffset;

		eval_kernel_vec_Horner(ker1,xstart+xoffset-x_rescaled,ns,sigma);

		for(int xx=xstart; xx<=xend; xx++){
			ix = xx+ceil(ns/2.0);
			if(ix >= (bin_size_x + (int) ceil(ns/2.0)*2) || ix<0) break;
			atomicAdd(&fwshared[ix].x, cnow.x*ker1[xx-xstart]);
			atomicAdd(&fwshared[ix].y, cnow.y*ker1[xx-xstart]);
		}
	}
	__syncthreads();

	/* write to global memory */
	for(int k=threadIdx.x; k<N; k+=blockDim.x){
		ix = xoffset-ceil(ns/2.0)+k;
		if(ix < (nf1+ceil(ns/2.0))){
			ix = ix < 0 ? ix+nf1 : (ix>nf1-1 ? ix-nf1 : ix);
			atomicAdd(&fw[ix].x, fwshared[k].x);
			atomicAdd(&fw[ix].y, fwshared[k].y);
		}
	}
}

/* --------------------- 1d Interpolation Kernels ----------------------------*/
/* Kernels for NUptsdriven Method */
__global__
void Interp_1d_NUptsdriven(FLT *x, CUCPX *c, CUCPX *fw, int M, const int ns,
		       int nf1, FLT es_c, FLT es_beta, int* idxnupts, int pirange)
{
	FLT ker1[MAX_NSPREAD];
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
		FLT x_rescaled=RESCALE(x[idxnupts[i]], nf1, pirange);
        
		int xstart = ceil(x_rescaled - ns/2.0);
		int xend  = floor(x_rescaled + ns/2.0);
		CUCPX cnow;
		cnow.x = 0.0;
		cnow.y = 0.0;

		FLT x1=(FLT)xstart-x_rescaled;
		eval_kernel_vec(ker1,x1,ns,es_c,es_beta);
		for(int xx=xstart; xx<=xend; xx++){
			int ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
			FLT kervalue1 = ker1[xx-xstart];
			cnow.x += fw[ix].x*kervalue1;
			cnow.y += fw[ix].y*kervalue1;
		}
		c[idxnupts[i]].x = cnow.x;
		c[idxnupts[i]].y = cnow.y;
	}
}

__global__
void Interp_1d_NUptsdriven_Horner(FLT *x, CUCPX *c, CUCPX *fw, int M, 
	const int ns, int nf1, FLT sigma, int* idxnupts, int pirange)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
		FLT x_rescaled=RESCALE(x[idxnupts[i]], nf1, pirange);

		int xstart = ceil(x_rescaled - ns/2.0);
		int xend  = floor(x_rescaled + ns/2.0);

		CUCPX cnow;
		cnow.x = 0.0;
		cnow.y = 0.0;
		FLT ker1[MAX_NSPREAD];

		eval_kernel_vec_Horner(ker1,xstart-x_rescaled,ns,sigma);

		for(int xx=xstart; xx<=xend; xx++){
			int ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
			cnow.x += fw[ix].x*ker1[xx-xstart];
			cnow.y += fw[ix].y*ker1[xx-xstart];
		}
		c[idxnupts[i]].x = cnow.x;
		c[idxnupts[i]].y = cnow.y;
	}

}
