#include "hip/hip_runtime.h"
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <assert.h>

#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "../cuspreadinterp.h"
#include "../memtransfer.h"

using namespace std;

int CUFINUFFT_SPREAD1D(int nf1, CUCPX* d_fw, int M, FLT *d_kx, CUCPX *d_c, 
	CUFINUFFT_PLAN d_plan)
/*
	This c function is written for only doing 1D spreading. See
	test/spread1d_test.cu for usage.

	note: not allocate,transfer and free memories on gpu.
	Melody Shih 11/21/21
*/
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	d_plan->kx = d_kx;
	d_plan->c  = d_c;
	d_plan->fw = d_fw;

	int ier;
	d_plan->nf1 = nf1;
	d_plan->M = M;
	d_plan->maxbatchsize = 1;

	hipEventRecord(start);
	ier = ALLOCGPUMEM1D_PLAN(d_plan);
	ier = ALLOCGPUMEM1D_NUPTS(d_plan);

	if(d_plan->opts.gpu_method == 1){
		ier = CUSPREAD1D_NUPTSDRIVEN_PROP(nf1,M,d_plan);
		if(ier != 0 ){
			printf("error: cuspread1d_nuptsdriven_prop, method(%d)\n",
				d_plan->opts.gpu_method);
			return ier;
		}
	}

	if(d_plan->opts.gpu_method == 2){
		ier = CUSPREAD1D_SUBPROB_PROP(nf1,M,d_plan);
		if(ier != 0 ){
			printf("error: cuspread1d_subprob_prop, method(%d)\n",
				d_plan->opts.gpu_method);
			return ier;
		}
	}

#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Obtain Spread Prop\t %.3g ms\n", milliseconds);
#endif

	hipEventRecord(start);
	ier = CUSPREAD1D(d_plan,1);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Spread (%d)\t\t %5.3f ms\n", d_plan->opts.gpu_method,
		milliseconds);
#endif

	hipEventRecord(start);
	FREEGPUMEMORY1D(d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
	return ier;
}

int CUSPREAD1D(CUFINUFFT_PLAN d_plan, int blksize)
/*
	A wrapper for different spreading methods.

	Methods available:
	(1) Non-uniform points driven
	(2) Subproblem

	Melody Shih 11/21/21
*/
{
	int nf1 = d_plan->nf1;
	int M = d_plan->M;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	switch(d_plan->opts.gpu_method)
	{
		case 1:
			{
				hipEventRecord(start);
				ier = CUSPREAD1D_NUPTSDRIVEN(nf1, M, d_plan, blksize);
				if(ier != 0 ){
					cout<<"error: cnufftspread1d_gpu_nuptsdriven"<<endl;
					return 1;
				}
			}
			break;
		case 2:
			{
				hipEventRecord(start);
				ier = CUSPREAD1D_SUBPROB(nf1, M, d_plan, blksize);
				if(ier != 0 ){
					cout<<"error: cnufftspread1d_gpu_subprob"<<endl;
					return 1;
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 1,2"<<endl;
			return 2;
	}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Spread " << milliseconds <<" ms"<<endl;
#endif
	return ier;
}

int CUSPREAD1D_NUPTSDRIVEN_PROP(int nf1, int M, CUFINUFFT_PLAN d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if(d_plan->opts.gpu_sort){

		int bin_size_x=d_plan->opts.gpu_binsizex;
		if(bin_size_x < 0){
			cout<<"error: invalid binsize (binsizex) = ("<<bin_size_x<<")"<<endl;
			return 1; 
		}

		int numbins = ceil((FLT) nf1/bin_size_x);
#ifdef DEBUG
		cout<<"[debug ] Dividing the uniform grids to bin size["
			<<d_plan->opts.gpu_binsizex<<"]"<<endl;
		cout<<"[debug ] numbins = ["<<numbins<<"]"<<endl;
#endif

		FLT*   d_kx = d_plan->kx;
#ifdef DEBUG
		FLT *h_kx;
		h_kx = (FLT*)malloc(M*sizeof(FLT));

		checkCudaErrors(hipMemcpy(h_kx,d_kx,M*sizeof(FLT),
			hipMemcpyDeviceToHost));
		for(int i=0; i<M; i++){
			cout<<"[debug ] ";
			cout <<"("<<setw(3)<<h_kx[i]<<")"<<endl;
		}
#endif
		int *d_binsize = d_plan->binsize;
		int *d_binstartpts = d_plan->binstartpts;
		int *d_sortidx = d_plan->sortidx;
		int *d_idxnupts = d_plan->idxnupts;

		int pirange = d_plan->spopts.pirange;

		hipEventRecord(start);
		checkCudaErrors(hipMemset(d_binsize,0,numbins*sizeof(int)));
		CalcBinSize_noghost_1d<<<(M+1024-1)/1024, 1024>>>(M,nf1,
			bin_size_x,numbins,d_binsize,d_kx,d_sortidx,pirange);
#ifdef SPREADTIME
		float milliseconds = 0;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel CalcBinSize_noghost_1d \t\t%.3g ms\n",
			milliseconds);
#endif
#ifdef DEBUG
		int *h_binsize;// For debug
		h_binsize     = (int*)malloc(numbins*sizeof(int));
		checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins*sizeof(int),
			hipMemcpyDeviceToHost));
		cout<<"[debug ] bin size:"<<endl;
		cout<<"[debug ] ";
		for(int i=0; i<numbins; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(1)<<i<<"]="<<h_binsize[i];
		}
		cout<<endl;
		free(h_binsize);
		cout<<"[debug ] ------------------------------------------------"<<endl;

		int *h_sortidx;
		h_sortidx = (int*)malloc(M*sizeof(int));

		checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),
			hipMemcpyDeviceToHost));

		for(int i=0; i<M; i++){
			if(h_sortidx[i] < 0){
				cout<<"[debug ] ";
				cout <<"point["<<setw(3)<<i<<"]="<<setw(3)<<h_sortidx[i]<<endl;
				cout<<"[debug ] ";
				printf("(%10.10f) ", RESCALE(h_kx[i],nf1,pirange));
				printf("(%10.10f) ", RESCALE(h_kx[i],nf1,pirange)/32);
				printf("(%f)\n", floor(RESCALE(h_kx[i],nf1,pirange)/32));
			}
		}
#endif
		hipEventRecord(start);
		int n=numbins;
		thrust::device_ptr<int> d_ptr(d_binsize);
		thrust::device_ptr<int> d_result(d_binstartpts);
		thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel BinStartPts_1d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
		int *h_binstartpts;
		h_binstartpts = (int*)malloc((numbins)*sizeof(int));
		checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins)
			*sizeof(int),hipMemcpyDeviceToHost));
		cout<<"[debug ] Result of scan bin_size array:"<<endl;
		cout<<"[debug ] ";
		for(int i=0; i<numbins; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(1)<<i<<"]="<<h_binstartpts[i];
		}
		cout<<endl;
		free(h_binstartpts);
		cout<<"[debug ] ------------------------------------------------"<<endl;
#endif
		hipEventRecord(start);
		CalcInvertofGlobalSortIdx_1d<<<(M+1024-1)/1024,1024>>>(M,bin_size_x,
			numbins,d_binstartpts,d_sortidx,d_kx,d_idxnupts,pirange,nf1);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel CalcInvertofGlobalSortIdx_1d \t%.3g ms\n",
			milliseconds);
#endif
#ifdef DEBUG
		int *h_idxnupts;
		h_idxnupts = (int*)malloc(M*sizeof(int));
		checkCudaErrors(hipMemcpy(h_idxnupts,d_idxnupts,M*sizeof(int),
					hipMemcpyDeviceToHost));
		for (int i=0; i<M; i++){
			cout <<"[debug ] idx="<< h_idxnupts[i]<<endl;
		}
		free(h_idxnupts);
#endif
	}else{
		int *d_idxnupts = d_plan->idxnupts;

		hipEventRecord(start);
		TrivialGlobalSortIdx_1d<<<(M+1024-1)/1024, 1024>>>(M,d_idxnupts);
#ifdef SPREADTIME
		float milliseconds = 0;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel TrivialGlobalSortIDx_1d \t\t%.3g ms\n",
			milliseconds);
#endif
	}
	return 0;
}

int CUSPREAD1D_NUPTSDRIVEN(int nf1, int M, CUFINUFFT_PLAN d_plan, int blksize)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=d_plan->spopts.nspread;   // psi's support in terms of number of cells
	int pirange=d_plan->spopts.pirange;
	int *d_idxnupts=d_plan->idxnupts;
	FLT es_c=d_plan->spopts.ES_c;
	FLT es_beta=d_plan->spopts.ES_beta;
	FLT sigma=d_plan->spopts.upsampfac;

	FLT* d_kx = d_plan->kx;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;
	hipEventRecord(start);
	if(d_plan->opts.gpu_kerevalmeth){
		for(int t=0; t<blksize; t++){
			Spread_1d_NUptsdriven_Horner<<<blocks, threadsPerBlock>>>(d_kx,
				d_c+t*M, d_fw+t*nf1, M, ns, nf1, sigma, d_idxnupts, pirange);
		}
	}else{
		for(int t=0; t<blksize; t++){
			Spread_1d_NUptsdriven<<<blocks, threadsPerBlock>>>(d_kx, d_c+t*M, 
				d_fw+t*nf1, M, ns, nf1, es_c, es_beta, d_idxnupts, pirange);
		}
	}

#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_1d_NUptsdriven (%d)\t%.3g ms\n",
		milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
	return 0;
}
int CUSPREAD1D_SUBPROB_PROP(int nf1, int M, CUFINUFFT_PLAN d_plan)
/*
	This function determines the properties for spreading that are independent
	of the strength of the nodes,  only relates to the locations of the nodes,
	which only needs to be done once.
*/
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int maxsubprobsize=d_plan->opts.gpu_maxsubprobsize;
	int bin_size_x=d_plan->opts.gpu_binsizex;
	if(bin_size_x < 0){
		cout<<"error: invalid binsize (binsizex) = (";
		cout<<bin_size_x<<")"<<endl;
		return 1; 
	}
	int numbins = ceil((FLT) nf1/bin_size_x);
#ifdef DEBUG
	cout<<"[debug  ] Dividing the uniform grids to bin size["
		<<d_plan->opts.gpu_binsizex<<"]"<<endl;
	cout<<"[debug  ] numbins = ["<<numbins<<"]"<<endl;
#endif

	FLT*   d_kx = d_plan->kx;

#ifdef DEBUG
	FLT *h_kx;
	h_kx = (FLT*)malloc(M*sizeof(FLT));

	checkCudaErrors(hipMemcpy(h_kx,d_kx,M*sizeof(FLT),hipMemcpyDeviceToHost));
	for(int i=0; i<M; i++){
		cout<<"[debug ]";
		cout <<"("<<setw(3)<<h_kx[i]<<")"<<endl;
	}
#endif
	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_sortidx = d_plan->sortidx;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;

	int *d_subprob_to_bin = NULL;

	int pirange=d_plan->spopts.pirange;

	hipEventRecord(start);
	checkCudaErrors(hipMemset(d_binsize,0,numbins*sizeof(int)));
	CalcBinSize_noghost_1d<<<(M+1024-1)/1024, 1024>>>(M,nf1,bin_size_x,
		numbins,d_binsize,d_kx,d_sortidx,pirange);
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_noghost_1d \t\t%.3g ms\n",
		milliseconds);
#endif
#ifdef DEBUG
	int *h_binsize;// For debug
	h_binsize = (int*)malloc(numbins*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins*sizeof(int),hipMemcpyDeviceToHost));
	cout<<"[debug ] bin size:"<<endl;
	cout<<"[debug ] ";
	for(int i=0; i<numbins; i++){
		if(i!=0) cout<<" ";
		cout <<"bin["<<setw(3)<<i<<"]="<<h_binsize[i];
	}
	free(h_binsize);
	cout<<"[debug ] ----------------------------------------------------"<<endl;
#endif
#ifdef DEBUG
	int *h_sortidx;
	h_sortidx = (int*)malloc(M*sizeof(int));
	checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),
		hipMemcpyDeviceToHost));
	cout<<"[debug ]";
	for(int i=0; i<M; i++){
		cout <<"[debug] point["<<setw(3)<<i<<"]="<<setw(3)<<h_sortidx[i]<<endl;
	}

#endif

	hipEventRecord(start);
	int n=numbins;
	thrust::device_ptr<int> d_ptr(d_binsize);
	thrust::device_ptr<int> d_result(d_binstartpts);
	thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_1d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binstartpts;
	h_binstartpts = (int*)malloc(numbins*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,numbins*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Result of scan bin_size array:"<<endl;
	cout<<"[debug ] ";
	for(int i=0; i<numbins; i++){
		if(i!=0) cout<<" ";
		cout <<"bin["<<setw(3)<<i<<"] = "<<setw(2)<<h_binstartpts[i];
	}
	free(h_binstartpts);
	cout<<"[debug ] ---------------------------------------------------"<<endl;
#endif
	hipEventRecord(start);
	CalcInvertofGlobalSortIdx_1d<<<(M+1024-1)/1024,1024>>>(M,bin_size_x,
		numbins,d_binstartpts,d_sortidx,d_kx,d_idxnupts,pirange,nf1);
#ifdef DEBUG
	int *h_idxnupts;
	h_idxnupts = (int*)malloc(M*sizeof(int));
	checkCudaErrors(hipMemcpy(h_idxnupts,d_idxnupts,M*sizeof(int),
				hipMemcpyDeviceToHost));
	for (int i=0; i<M; i++){
		cout <<"[debug ] idx="<< h_idxnupts[i]<<endl;
	}
	free(h_idxnupts);
#endif
	hipEventRecord(start);
	CalcSubProb_1d<<<(M+1024-1)/1024, 1024>>>(d_binsize,d_numsubprob,
		maxsubprobsize,numbins);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcSubProb_1d\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int* h_numsubprob;
	h_numsubprob = (int*) malloc(n*sizeof(int));
	checkCudaErrors(hipMemcpy(h_numsubprob,d_numsubprob,numbins*
				sizeof(int),hipMemcpyDeviceToHost));
	cout<<"[debug ] ";
	for(int i=0; i<numbins; i++){
		if(i!=0) cout<<" ";
		cout <<"nsub["<<setw(3)<<i<<"] = "<<setw(2)<<h_numsubprob[i];
	}
	cout << endl;
	free(h_numsubprob);
#endif
	d_ptr    = thrust::device_pointer_cast(d_numsubprob);
	d_result = thrust::device_pointer_cast(d_subprobstartpts+1);
	thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
	checkCudaErrors(hipMemset(d_subprobstartpts,0,sizeof(int)));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Scan Subprob array\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
	printf("[debug ] Subproblem start points\n");
	int* h_subprobstartpts;
	h_subprobstartpts = (int*) malloc((n+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprobstartpts,d_subprobstartpts,
				(n+1)*sizeof(int),hipMemcpyDeviceToHost));
	cout<<"[debug ] ";
	for(int i=0; i<numbins; i++){
		if(i!=0) cout<<" ";
		cout <<"nsub["<<setw(3)<<i<<"] = "<<setw(2)<<h_subprobstartpts[i];
	}
	cout << endl;
	printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
	free(h_subprobstartpts);
#endif
	hipEventRecord(start);
	int totalnumsubprob;
	checkCudaErrors(hipMemcpy(&totalnumsubprob,&d_subprobstartpts[n],
		sizeof(int),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMalloc(&d_subprob_to_bin,totalnumsubprob*sizeof(int)));
	MapBintoSubProb_1d<<<(numbins+1024-1)/1024, 1024>>>(
			d_subprob_to_bin,d_subprobstartpts,d_numsubprob,numbins);
	assert(d_subprob_to_bin != NULL);
	if (d_plan->subprob_to_bin != NULL) hipFree(d_plan->subprob_to_bin);
	d_plan->subprob_to_bin = d_subprob_to_bin;
	assert(d_plan->subprob_to_bin != NULL);
	d_plan->totalnumsubprob = totalnumsubprob;
#ifdef DEBUG
	printf("[debug ] Map Subproblem to Bins\n");
	int* h_subprob_to_bin;
	h_subprob_to_bin = (int*) malloc((totalnumsubprob)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprob_to_bin,d_subprob_to_bin,
				(totalnumsubprob)*sizeof(int),hipMemcpyDeviceToHost));
	for(int j=0; j<totalnumsubprob; j++){
		cout<<"[debug ] ";
		cout <<"nsub["<<j<<"] = "<<setw(2)<<h_subprob_to_bin[j];
		cout<<endl;
	}
	free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int CUSPREAD1D_SUBPROB(int nf1, int M, CUFINUFFT_PLAN d_plan, int blksize)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ns=d_plan->spopts.nspread;// psi's support in terms of number of cells
	FLT es_c=d_plan->spopts.ES_c;
	FLT es_beta=d_plan->spopts.ES_beta;
	int maxsubprobsize=d_plan->opts.gpu_maxsubprobsize;

	// assume that bin_size_x > ns/2;
	int bin_size_x=d_plan->opts.gpu_binsizex;
	int numbins = ceil((FLT) nf1/bin_size_x);

	FLT* d_kx = d_plan->kx;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;

	int totalnumsubprob=d_plan->totalnumsubprob;
	int *d_subprob_to_bin = d_plan->subprob_to_bin;

	int pirange=d_plan->spopts.pirange;

	FLT sigma=d_plan->opts.upsampfac;
	hipEventRecord(start);

	size_t sharedplanorysize = (bin_size_x+2*(int)ceil(ns/2.0))*sizeof(CUCPX);
	if(sharedplanorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}

	if(d_plan->opts.gpu_kerevalmeth){
		for(int t=0; t<blksize; t++){
			Spread_1d_Subprob_Horner<<<totalnumsubprob, 256,
				sharedplanorysize>>>(d_kx, d_c+t*M, d_fw+t*nf1, M,
				ns, nf1, sigma, d_binstartpts, d_binsize, bin_size_x,
				d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
				numbins, d_idxnupts, pirange);
		}
	}else{
		for(int t=0; t<blksize; t++){
			Spread_1d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
				d_kx, d_c+t*M, d_fw+t*nf1, M, ns, nf1, es_c, es_beta, sigma, 
				d_binstartpts, d_binsize, bin_size_x, d_subprob_to_bin, 
				d_subprobstartpts, d_numsubprob, maxsubprobsize, numbins,
				d_idxnupts, pirange);
		}
	}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_1d_Subprob (%d)\t\t%.3g ms\n",
		milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
	return 0;
}
