#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include "cuda_hip_wrapper.h"
#include <hip/hip_runtime_api.h>
#include <complex>

#include <cufinufft_eitherprec.h>
#include "../cuspreadinterp.h"
#include "../cudeconvolve.h"
#include "../memtransfer.h"

using namespace std;

int CUFINUFFT1D1_EXEC(CUCPX* d_c, CUCPX* d_fk, CUFINUFFT_PLAN d_plan)
/*  
	1D Type-1 NUFFT

	This function is called in "exec" stage (See ../cufinufft.cu).
	It includes (copied from doc in finufft library)
		Step 1: spread data to oversampled regular mesh using kernel
		Step 2: compute FFT on uniform mesh
		Step 3: deconvolve by division of each Fourier mode independently by the
		        Fourier series coefficient of the kernel.

	Melody Shih 11/21/21
*/
{
	assert(d_plan->spopts.spread_direction == 1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	int blksize;
	int ier;
	CUCPX* d_fkstart;
	CUCPX* d_cstart;
	for(int i=0; i*d_plan->maxbatchsize < d_plan->ntransf; i++){
		blksize = min(d_plan->ntransf - i*d_plan->maxbatchsize, 
			d_plan->maxbatchsize);
		d_cstart   = d_c + i*d_plan->maxbatchsize*d_plan->M;
		d_fkstart  = d_fk + i*d_plan->maxbatchsize*d_plan->ms;
		d_plan->c  = d_cstart;
		d_plan->fk = d_fkstart;

		checkCudaErrors(hipMemset(d_plan->fw,0,d_plan->maxbatchsize*
					d_plan->nf1*sizeof(CUCPX)));// this is needed
#ifdef TIME
		float milliseconds = 0;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tInitialize fw to 0\t %.3g s\n", 
			milliseconds/1000);
#endif
		// Step 1: Spread
		hipEventRecord(start);
		ier = CUSPREAD1D(d_plan,blksize);
		if(ier != 0 ){
			printf("error: cuspread1d, method(%d)\n", d_plan->opts.gpu_method);
			return ier;
		}
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tSpread (%d)\t\t %.3g s\n", milliseconds/1000, 
			d_plan->opts.gpu_method);
#endif
		// Step 2: FFT
		hipEventRecord(start);
		CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds/1000);
#endif

		// Step 3: deconvolve and shuffle
		hipEventRecord(start);
		CUDECONVOLVE1D(d_plan,blksize);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tDeconvolve\t\t %.3g s\n", milliseconds/1000);
#endif
	}
	return ier;
}

int CUFINUFFT1D2_EXEC(CUCPX* d_c, CUCPX* d_fk, CUFINUFFT_PLAN d_plan)
/*  
	1D Type-2 NUFFT

	This function is called in "exec" stage (See ../cufinufft.cu).
	It includes (copied from doc in finufft library)
		Step 1: deconvolve (amplify) each Fourier mode, dividing by kernel 
		        Fourier coeff
		Step 2: compute FFT on uniform mesh
		Step 3: interpolate data to regular mesh

	Melody Shih 11/21/21
*/
{
	assert(d_plan->spopts.spread_direction == 2);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	int blksize;
	int ier;
	CUCPX* d_fkstart;
	CUCPX* d_cstart;
	for(int i=0; i*d_plan->maxbatchsize < d_plan->ntransf; i++){
		blksize = min(d_plan->ntransf - i*d_plan->maxbatchsize, 
			d_plan->maxbatchsize);
		d_cstart  = d_c  + i*d_plan->maxbatchsize*d_plan->M;
		d_fkstart = d_fk + i*d_plan->maxbatchsize*d_plan->ms;

		d_plan->c = d_cstart;
		d_plan->fk = d_fkstart;

		// Step 1: amplify Fourier coeffs fk and copy into upsampled array fw
		hipEventRecord(start);
		CUDECONVOLVE1D(d_plan,blksize);
#ifdef TIME
		float milliseconds = 0;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tAmplify & Copy fktofw\t %.3g s\n", milliseconds/1000);
#endif
		// Step 2: FFT
		hipDeviceSynchronize();
		hipEventRecord(start);
		CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds/1000);
#endif

		// Step 3: deconvolve and shuffle
		hipEventRecord(start);
		ier = CUINTERP1D(d_plan, blksize);
		if(ier != 0 ){
			printf("error: cuinterp1d, method(%d)\n", d_plan->opts.gpu_method);
			return ier;
		}
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tUnspread (%d)\t\t %.3g s\n", milliseconds/1000,
			d_plan->opts.gpu_method);
#endif
	}
	return ier;
}

