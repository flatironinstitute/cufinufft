#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "memtransfer.h"
#include <multi_gpu_policy.h>

using namespace std;

int __ALLOCGPUMEM2D_PLAN(CUFINUFFT_PLAN d_plan)
/*
	wrapper for gpu memory allocation in "plan" stage.

	Melody Shih 07/25/19
*/
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int maxbatchsize = d_plan->maxbatchsize;

	d_plan->byte_now=0;
	// No extra memory is needed in nuptsdriven method (case 1)
	switch(d_plan->opts.gpu_method)
	{
		case 1:
			{
				if(d_plan->opts.gpu_sort){
					int numbins[2];
					numbins[0] = ceil((FLT) nf1/d_plan->opts.gpu_binsizex);
					numbins[1] = ceil((FLT) nf2/d_plan->opts.gpu_binsizey);
					checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
					checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				}
			}
			break;
		case 2:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/d_plan->opts.gpu_binsizex);
				numbins[1] = ceil((FLT) nf2/d_plan->opts.gpu_binsizey);
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
						(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
		case 3:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/d_plan->opts.gpu_binsizex);
				numbins[1] = ceil((FLT) nf2/d_plan->opts.gpu_binsizey);
				checkCudaErrors(hipMalloc(&d_plan->finegridsize,nf1*nf2*
						sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->fgstartpts,nf1*nf2*
						sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
						(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method " << endl;
	}

	if(!d_plan->opts.gpu_spreadinterponly){
		checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize*nf1*nf2*
				sizeof(CUCPX)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1,(nf1/2+1)*sizeof(FLT)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2,(nf2/2+1)*sizeof(FLT)));
	}

	hipStream_t* streams =(hipStream_t*) malloc(d_plan->opts.gpu_nstreams*
		sizeof(hipStream_t));
	for(int i=0; i<d_plan->opts.gpu_nstreams; i++)
		checkCudaErrors(hipStreamCreate(&streams[i]));
	d_plan->streams = streams;

	return 0;
}


int ALLOCGPUMEM2D_PLAN(CUFINUFFT_PLAN d_plan) {

    int ierr;
    int orig_device;

    if (policy_set_device(& d_plan->opts) == 1) {
        hipGetDevice(& orig_device);
        hipSetDevice(d_plan->opts.gpu_device_id);
    }

    ierr =  __ALLOCGPUMEM2D_PLAN(d_plan);

    if (policy_set_device(& d_plan->opts) == 1) {
        hipSetDevice(orig_device);
    }

    return ierr;
}



int __ALLOCGPUMEM2D_NUPTS(CUFINUFFT_PLAN d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
	int M = d_plan->M;

	if(d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if(d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch(d_plan->opts.gpu_method)
	{
		case 1:
			{
				if(d_plan->opts.gpu_sort)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case 2:
		case 3:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		default:
			cerr<<"err: invalid method" << endl;
	}

	return 0;
}


int ALLOCGPUMEM2D_NUPTS(CUFINUFFT_PLAN d_plan) {

    int ierr;
    int orig_device;

    if (policy_set_device(& d_plan->opts) == 1) {
        hipGetDevice(& orig_device);
        hipSetDevice(d_plan->opts.gpu_device_id);
    }

    ierr = __ALLOCGPUMEM2D_NUPTS(d_plan);

    if (policy_set_device(& d_plan->opts) == 1) {
        hipSetDevice(orig_device);
    }

    return ierr;
}



void __FREEGPUMEMORY2D(CUFINUFFT_PLAN d_plan)
/*
	wrapper for freeing gpu memory.

	Melody Shih 07/25/19
*/
{
	if(!d_plan->opts.gpu_spreadinterponly){
		checkCudaErrors(hipFree(d_plan->fw));
		checkCudaErrors(hipFree(d_plan->fwkerhalf1));
		checkCudaErrors(hipFree(d_plan->fwkerhalf2));
	}
	switch(d_plan->opts.gpu_method)
	{
		case 1:
			{
				if(d_plan->opts.gpu_sort){
					checkCudaErrors(hipFree(d_plan->idxnupts));
					checkCudaErrors(hipFree(d_plan->sortidx));
					checkCudaErrors(hipFree(d_plan->binsize));
					checkCudaErrors(hipFree(d_plan->binstartpts));
				}else{
					checkCudaErrors(hipFree(d_plan->idxnupts));
				}
			}
			break;
		case 2:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case 3:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->finegridsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}

	for(int i=0; i<d_plan->opts.gpu_nstreams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));
}


void FREEGPUMEMORY2D(CUFINUFFT_PLAN d_plan) {

    int orig_device;

    if (policy_set_device(& d_plan->opts) == 1) {
        hipGetDevice(& orig_device);
        hipSetDevice(d_plan->opts.gpu_device_id);
    }

    __FREEGPUMEMORY2D(d_plan);

    if (policy_set_device(& d_plan->opts) == 1) {
        hipSetDevice(orig_device);
    }
}



int ALLOCGPUMEM1D_PLAN(CUFINUFFT_PLAN d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
int ALLOCGPUMEM1D_NUPTS(CUFINUFFT_PLAN d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
void FREEGPUMEMORY1D(CUFINUFFT_PLAN d_plan)
{
	cerr<<"Not yet implemented"<<endl;
}

int __ALLOCGPUMEM3D_PLAN(CUFINUFFT_PLAN d_plan)
/*
	wrapper for gpu memory allocation in "plan" stage.

	Melody Shih 07/25/19
*/
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int nf3 = d_plan->nf3;
	int maxbatchsize = d_plan->maxbatchsize;

	d_plan->byte_now=0;

	switch(d_plan->opts.gpu_method)
	{
		case 1:
			{
				if(d_plan->opts.gpu_sort){
					int numbins[3];
					numbins[0] = ceil((FLT) nf1/d_plan->opts.gpu_binsizex);
					numbins[1] = ceil((FLT) nf2/d_plan->opts.gpu_binsizey);
					numbins[2] = ceil((FLT) nf3/d_plan->opts.gpu_binsizez);
					checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*numbins[2]*sizeof(int)));
					checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*numbins[2]*sizeof(int)));
				}
			}
			break;
		case 2:
			{
				int numbins[3];
				numbins[0] = ceil((FLT) nf1/d_plan->opts.gpu_binsizex);
				numbins[1] = ceil((FLT) nf2/d_plan->opts.gpu_binsizey);
				numbins[2] = ceil((FLT) nf3/d_plan->opts.gpu_binsizez);
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
					(numbins[0]*numbins[1]*numbins[2]+1)*sizeof(int)));
			}
			break;
		case 4:
			{
				int numobins[3], numbins[3];
				int binsperobins[3];
				numobins[0] = ceil((FLT) nf1/d_plan->opts.gpu_obinsizex);
				numobins[1] = ceil((FLT) nf2/d_plan->opts.gpu_obinsizey);
				numobins[2] = ceil((FLT) nf3/d_plan->opts.gpu_obinsizez);

				binsperobins[0] = d_plan->opts.gpu_obinsizex/
					d_plan->opts.gpu_binsizex;
				binsperobins[1] = d_plan->opts.gpu_obinsizey/
					d_plan->opts.gpu_binsizey;
				binsperobins[2] = d_plan->opts.gpu_obinsizez/
					d_plan->opts.gpu_binsizez;

				numbins[0] = numobins[0]*(binsperobins[0]+2);
				numbins[1] = numobins[1]*(binsperobins[1]+2);
				numbins[2] = numobins[2]*(binsperobins[2]+2);

				checkCudaErrors(hipMalloc(&d_plan->numsubprob,
					numobins[0]*numobins[1]*numobins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,
					numbins[0]*numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,
					(numbins[0]*numbins[1]*numbins[2]+1)*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,(numobins[0]
					*numobins[1]*numobins[2]+1)*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method" << endl;
	}

	if(!d_plan->opts.gpu_spreadinterponly){
		checkCudaErrors(hipMalloc(&d_plan->fw, maxbatchsize*nf1*nf2*nf3*
			sizeof(CUCPX)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1,(nf1/2+1)*sizeof(FLT)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2,(nf2/2+1)*sizeof(FLT)));
		checkCudaErrors(hipMalloc(&d_plan->fwkerhalf3,(nf3/2+1)*sizeof(FLT)));
	}

	return 0;
}


int ALLOCGPUMEM3D_PLAN(CUFINUFFT_PLAN d_plan) {

    int ierr;
    int orig_device;

    if (policy_set_device(& d_plan->opts) == 1) {
        hipGetDevice(& orig_device);
        hipSetDevice(d_plan->opts.gpu_device_id);
    }

    ierr = __ALLOCGPUMEM3D_PLAN(d_plan);

    if (policy_set_device(& d_plan->opts) == 1) {
        hipSetDevice(orig_device);
    }

    return ierr;
}



int __ALLOCGPUMEM3D_NUPTS(CUFINUFFT_PLAN d_plan)
/*
	wrapper for gpu memory allocation in "setNUpts" stage.

	Melody Shih 07/25/19
*/
{
	int M = d_plan->M;

	d_plan->byte_now=0;

	if(d_plan->sortidx ) checkCudaErrors(hipFree(d_plan->sortidx));
	if(d_plan->idxnupts) checkCudaErrors(hipFree(d_plan->idxnupts));

	switch(d_plan->opts.gpu_method)
	{
		case 1:
			{
				if(d_plan->opts.gpu_sort)
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
			}
			break;
		case 2:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
		case 4:
			{
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
			}
			break;
		default:
			cerr << "err: invalid method" << endl;
	}

	return 0;
}


int ALLOCGPUMEM3D_NUPTS(CUFINUFFT_PLAN d_plan) {

    int ierr;
    int orig_device;

    if (policy_set_device(& d_plan->opts) == 1) {
        hipGetDevice(& orig_device);
        hipSetDevice(d_plan->opts.gpu_device_id);
    }

    ierr = __ALLOCGPUMEM3D_NUPTS(d_plan);

    if (policy_set_device(& d_plan->opts) == 1) {
        hipSetDevice(orig_device);
    }

    return ierr;
}



void __FREEGPUMEMORY3D(CUFINUFFT_PLAN d_plan)
/*
	wrapper for freeing gpu memory.

	Melody Shih 07/25/19
*/
{
	if(!d_plan->opts.gpu_spreadinterponly){
		hipFree(d_plan->fw);
		hipFree(d_plan->fwkerhalf1);
		hipFree(d_plan->fwkerhalf2);
		hipFree(d_plan->fwkerhalf3);
	}

	switch(d_plan->opts.gpu_method)
	{
		case 1:
			{
				if(d_plan->opts.gpu_sort){
					checkCudaErrors(hipFree(d_plan->idxnupts));
					checkCudaErrors(hipFree(d_plan->sortidx));
					checkCudaErrors(hipFree(d_plan->binsize));
					checkCudaErrors(hipFree(d_plan->binstartpts));
				}else{
					checkCudaErrors(hipFree(d_plan->idxnupts));
				}
			}
			break;
		case 2:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case 4:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}

	for(int i=0; i<d_plan->opts.gpu_nstreams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));
}



void FREEGPUMEMORY3D(CUFINUFFT_PLAN d_plan) {

    int orig_device;

    if (policy_set_device(& d_plan->opts) == 1) {
        hipGetDevice(& orig_device);
        hipSetDevice(d_plan->opts.gpu_device_id);
    }

    __FREEGPUMEMORY3D(d_plan);

    if (policy_set_device(& d_plan->opts) == 1) {
        hipSetDevice(orig_device);
    }
}
