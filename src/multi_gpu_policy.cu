#include <multi_gpu_policy.h>
#include <cufinufft_opts.h>
#include <hip/hip_runtime.h>



hipError_t get_current_device(CtxProfile * ctx_profile) {

    hipError_t cuda_err;
    hipError_t ierr;

    // GET current device bound to this thread (this will work for _both_ the
    // cuda runtim API and the cuda driver)
    cuda_err = hipGetDevice(& ctx_profile->i_dev);
    if (cuda_err != hipSuccess) {
        ctx_profile->i_dev      = -1;
        ctx_profile->is_primary = -1;
        ctx_profile->is_clean   = -1;
        return hipErrorInvalidContext;
    }

    // GET the state of the primary context
    unsigned int flags;
    int active;
    hipDevicePrimaryCtxGetState(ctx_profile->i_dev, & flags, & active);
    if (active == 1){
        ctx_profile->is_primary = 1;
        ctx_profile->is_clean   = 0;
        return hipSuccess;
    }

    // The PRIMARY CONTEXT could be the only context on the device bound to
    // this thread -- and just not be active because nothing has been called
    // it, or another context is the current context

    hipDevice_t device;

    ierr = hipCtxGetDevice(& device);
    if (ierr == hipErrorInvalidContext) {
        ctx_profile->is_primary = 1;
        ctx_profile->is_clean   = 1;
    } else if (ierr != hipSuccess) {
        return ierr;
    }

    // There is defintely a device bound to this thread -- so there MUST be a
    // current context. The only thing to figure out now is if this context is
    // the primary context (which just hasn't been used yet), or another.

    hipCtx_t context;

    ierr = hipCtxGetCurrent(& context);
    if (ierr != hipSuccess)
        return ierr;

    hipCtx_t primary_context;
    // NOTE: this will make the primary context active -- we need to release it
    // again below:
    ierr = hipDevicePrimaryCtxRetain(& primary_context, device);
    if (ierr != hipSuccess)
        return ierr;
    // Restore the pre-existing context by de-activating the primary context
    // that cudaPrimaryCtxRetain activated
    ierr = hipDevicePrimaryCtxRelease(device);
    if (ierr != hipSuccess)
        return ierr;

    if (primary_context == context) {
        ctx_profile->is_primary = 1;
        ctx_profile->is_clean   = 0;
        return hipSuccess;
    }

    ctx_profile->is_primary = 0;
    ctx_profile->is_clean   = 0;
    return hipSuccess;
}



int use_set_device(CtxProfile * ctx_profile, cufinufft_opts * opts) {

    if (ctx_profile->is_primary == 1 || ctx_profile->is_clean == 1 || opts->gpu_force_primary_ctx == 1) {
        return 1;
    }

    return 0;
}



int policy_set_device(cufinufft_opts * opts) {
    return opts->gpu_primary_ctx;
}
