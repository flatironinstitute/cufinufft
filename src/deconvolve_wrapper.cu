#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "cudeconvolve.h"

using namespace std;

/* Kernel for copying fw to fk with amplication by prefac/ker */
// Note: assume modeord=0: CMCL-compatible mode ordering in fk (from -N/2 up 
// to N/2-1)
__global__
void Deconvolve_2d(int ms, int mt, int nf1, int nf2, CUCPX* fw, CUCPX *fk, 
	FLT *fwkerhalf1, FLT *fwkerhalf2)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<ms*mt; i+=blockDim.x*gridDim.x){
		int k1 = i % ms;
		int k2 = i / ms;
		int outidx = k1 + k2*ms;
		int w1 = k1-ms/2 >= 0 ? k1-ms/2 : nf1+k1-ms/2;
		int w2 = k2-mt/2 >= 0 ? k2-mt/2 : nf2+k2-mt/2;
		int inidx = w1 + w2*nf1;

		FLT kervalue = fwkerhalf1[abs(k1-ms/2)]*fwkerhalf2[abs(k2-mt/2)];
		fk[outidx].x = fw[inidx].x/kervalue;
		fk[outidx].y = fw[inidx].y/kervalue;
	}
}

__global__
void Deconvolve_3d(int ms, int mt, int mu, int nf1, int nf2, int nf3, CUCPX* fw, 
	CUCPX *fk, FLT *fwkerhalf1, FLT *fwkerhalf2, FLT *fwkerhalf3)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<ms*mt*mu; i+=blockDim.x*
		gridDim.x){
		int k1 = i % ms;
		int k2 = (i / ms) % mt;
		int k3 = (i / ms / mt);
		int outidx = k1 + k2*ms + k3*ms*mt;
		int w1 = k1-ms/2 >= 0 ? k1-ms/2 : nf1+k1-ms/2;
		int w2 = k2-mt/2 >= 0 ? k2-mt/2 : nf2+k2-mt/2;
		int w3 = k3-mu/2 >= 0 ? k3-mu/2 : nf3+k3-mu/2;
		int inidx = w1 + w2*nf1 + w3*nf1*nf2;

		FLT kervalue = fwkerhalf1[abs(k1-ms/2)]*fwkerhalf2[abs(k2-mt/2)]*
			fwkerhalf3[abs(k3-mu/2)];
		fk[outidx].x = fw[inidx].x/kervalue;
		fk[outidx].y = fw[inidx].y/kervalue;
		//fk[outidx].x = kervalue;
		//fk[outidx].y = kervalue;
	}
}

/* Kernel for copying fk to fw with same amplication */
__global__
void Amplify_2d(int ms, int mt, int nf1, int nf2, CUCPX* fw, CUCPX *fk, 
	FLT *fwkerhalf1, FLT *fwkerhalf2)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<ms*mt; i+=blockDim.x*gridDim.x){
		int k1 = i % ms;
		int k2 = i / ms;
		int inidx = k1 + k2*ms;
		int w1 = k1-ms/2 >= 0 ? k1-ms/2 : nf1+k1-ms/2;
		int w2 = k2-mt/2 >= 0 ? k2-mt/2 : nf2+k2-mt/2;
		int outidx = w1 + w2*nf1;

		FLT kervalue = fwkerhalf1[abs(k1-ms/2)]*fwkerhalf2[abs(k2-mt/2)];
		fw[outidx].x = fk[inidx].x/kervalue;
		fw[outidx].y = fk[inidx].y/kervalue;
	}
}

__global__
void Amplify_3d(int ms, int mt, int mu, int nf1, int nf2, int nf3, CUCPX* fw, 
	CUCPX *fk, FLT *fwkerhalf1, FLT *fwkerhalf2, FLT *fwkerhalf3)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<ms*mt*mu; 
		i+=blockDim.x*gridDim.x){
		int k1 = i % ms;
		int k2 = (i / ms) % mt;
		int k3 = (i / ms / mt);
		int inidx = k1 + k2*ms + k3*ms*mt;
		int w1 = k1-ms/2 >= 0 ? k1-ms/2 : nf1+k1-ms/2;
		int w2 = k2-mt/2 >= 0 ? k2-mt/2 : nf2+k2-mt/2;
		int w3 = k3-mu/2 >= 0 ? k3-mu/2 : nf3+k3-mu/2;
		int outidx = w1 + w2*nf1 + w3*nf1*nf2;

		FLT kervalue = fwkerhalf1[abs(k1-ms/2)]*fwkerhalf2[abs(k2-mt/2)]*
			fwkerhalf3[abs(k3-mu/2)];
		fw[outidx].x = fk[inidx].x/kervalue;
		fw[outidx].y = fk[inidx].y/kervalue;
		//fw[outidx].x = fk[inidx].x;
		//fw[outidx].y = fk[inidx].y;
	}
}


int cudeconvolve2d(cufinufft_plan *d_plan, int blksize)
/* 
	wrapper for deconvolution & amplication in 2D.

	Melody Shih 07/25/19
*/
{
	int ms=d_plan->ms;
	int mt=d_plan->mt;
	int nf1=d_plan->nf1;
	int nf2=d_plan->nf2;
	int nmodes=ms*mt;
	int maxbatchsize=d_plan->maxbatchsize;

	if(d_plan->spopts.spread_direction == 1){
		for(int t=0; t<blksize; t++){
			Deconvolve_2d<<<(nmodes+256-1)/256, 256>>>(ms, mt, nf1, nf2, 
				d_plan->fw+t*nf1*nf2,d_plan->fk+t*nmodes,d_plan->fwkerhalf1, 
				d_plan->fwkerhalf2);
		}
	}else{
		checkCudaErrors(hipMemset(d_plan->fw,0,maxbatchsize*nf1*nf2*
			sizeof(CUCPX)));
		for(int t=0; t<blksize; t++){
			Amplify_2d<<<(nmodes+256-1)/256, 256>>>(ms, 
				mt, nf1, nf2, d_plan->fw+t*nf1*nf2, d_plan->fk+t*nmodes,
				d_plan->fwkerhalf1, d_plan->fwkerhalf2);
#ifdef DEBUG
			CPX* h_fw;
			h_fw = (CPX*) malloc(nf1*nf2*sizeof(CPX));
			checkCudaErrors(hipMemcpy2D(h_fw,nf1*sizeof(CUCPX),d_plan->fw,
				nf1*sizeof(CUCPX),nf1*sizeof(CUCPX),nf2,
				hipMemcpyDeviceToHost));
			for(int j=0; j<nf2; j++){
				for(int i=0; i<nf1; i++){
					printf("(%g,%g)",h_fw[i+j*nf1].real(),h_fw[i+j*nf1].imag());
				}
				printf("\n");
			}
			free(h_fw);
#endif
		}
	}
	return 0;
}

int cudeconvolve3d(cufinufft_plan *d_plan, int blksize)
/* 
	wrapper for deconvolution & amplication in 3D.

	Melody Shih 07/25/19
*/
{
	int ms=d_plan->ms;
	int mt=d_plan->mt;
	int mu=d_plan->mu;
	int nf1=d_plan->nf1;
	int nf2=d_plan->nf2;
	int nf3=d_plan->nf3;
	int nmodes=ms*mt*mu;
	int maxbatchsize=d_plan->maxbatchsize;
	if(d_plan->spopts.spread_direction == 1){
		for(int t=0; t<blksize; t++){
			Deconvolve_3d<<<(nmodes+256-1)/256, 256>>>(ms, mt, mu, nf1, nf2, 
				nf3, d_plan->fw+t*nf1*nf2*nf3, d_plan->fk+t*nmodes, 
				d_plan->fwkerhalf1, d_plan->fwkerhalf2, d_plan->fwkerhalf3);
		}
	}else{
		checkCudaErrors(hipMemset(d_plan->fw,0,maxbatchsize*nf1*nf2*nf3*
			sizeof(CUCPX)));
		for(int t=0; t<blksize; t++){
			Amplify_3d<<<(nmodes+256-1)/256, 256>>>(ms, mt, mu, nf1, nf2, nf3,
				d_plan->fw+t*nf1*nf2*nf3, d_plan->fk+t*nmodes, 
				d_plan->fwkerhalf1, d_plan->fwkerhalf2, d_plan->fwkerhalf3);
#if 0
			CPX* h_fw;
			h_fw = (CPX*) malloc(nf1*nf2*nf3*sizeof(CPX));
			checkCudaErrors(hipMemcpy(h_fw,d_plan->fw,nf1*nf2*nf3*sizeof(CUCPX),
				hipMemcpyDeviceToHost));
			for(int k=0; k<nf3; k++){
				for(int j=0; j<nf2; j++){
					for(int i=0; i<nf1; i++){
						printf("(%g,%g,%g)",h_fw[i+j*nf1+k*nf1*nf2].real(),
							h_fw[i+j*nf1+k*nf1*nf2].imag());
					}
					printf("\n");
				}
				printf("\n");
			}
			free(h_fw);
#endif
		}
	}
	return 0;
}
