#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "../cuspreadinterp.h"
#include "../memtransfer.h"
#include <profile.h>

using namespace std;

int cufinufft_interp3d(int ms, int mt, int mu, int nf1, int nf2, int nf3, 
	CPX* h_fw, int M, FLT *h_kx, FLT *h_ky, FLT *h_kz, CPX *h_c, FLT eps, 
	cufinufft_plan* d_plan)
/*
	This c function is written for only doing 3D interpolation. It includes 
	allocating, transfering and freeing the memories on gpu. See 
	test/interp_3d.cu for usage.

	Melody Shih 07/25/19
*/
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	//ier = setup_spreader_for_nufft(d_plan->spopts, eps, d_plan->opts);

	d_plan->ms = ms;
	d_plan->mt = mt;
	d_plan->mu = mu;
	d_plan->nf1 = nf1;
	d_plan->nf2 = nf2;
	d_plan->nf3 = nf3;
	d_plan->M = M;
	d_plan->maxbatchsize = 1;

	hipEventRecord(start);
	ier = allocgpumem3d_plan(d_plan);
	ier = allocgpumem3d_nupts(d_plan);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Allocate GPU memory\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	checkCudaErrors(hipMalloc(&d_plan->kx,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->ky,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->kz,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->c,M*sizeof(CUCPX)));

	checkCudaErrors(hipMemcpy(d_plan->kx,h_kx,M*sizeof(FLT),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->ky,h_ky,M*sizeof(FLT),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->kz,h_kz,M*sizeof(FLT),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->fw,h_fw,nf1*nf2*nf3*sizeof(CUCPX),
		hipMemcpyHostToDevice));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory HtoD\t %.3g ms\n", milliseconds);
#endif
	if(d_plan->opts.gpu_method == 1){
		ier = cuspread3d_nuptsdriven_prop(nf1,nf2,nf3,M,d_plan);
		if(ier != 0 ){
			printf("error: cuinterp3d_nuptsdriven_prop, method(%d)\n", 
				d_plan->opts.gpu_method);
			return 0;
		}
	}
	if(d_plan->opts.gpu_method == 2){
		ier = cuspread3d_subprob_prop(nf1,nf2,nf3,M,d_plan);
		if(ier != 0 ){
			printf("error: cuspread3d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
			return 0;
		}
	}
	hipEventRecord(start);
	ier = cuinterp3d(d_plan, 1);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Interp (%d)\t\t %.3g ms\n", d_plan->opts.gpu_method, milliseconds);
#endif
	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(h_c,d_plan->c,M*sizeof(CUCPX),hipMemcpyDeviceToHost));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory DtoH\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	freegpumemory3d(d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
	hipFree(d_plan->kx);
	hipFree(d_plan->ky);
	hipFree(d_plan->kz);
	hipFree(d_plan->c);
	return ier;
}

int cuinterp3d(cufinufft_plan* d_plan, int blksize)
/*
	A wrapper for different interpolation methods. 

	Methods available:
	(1) Non-uniform points driven
	(2) Subproblem

	Melody Shih 07/25/19
*/
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int nf3 = d_plan->nf3;
	int M = d_plan->M;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	switch(d_plan->opts.gpu_method)
	{
		case 1:
			{
				hipEventRecord(start);
				{
					PROFILE_CUDA_GROUP("Interpolation", 6);
					ier = cuinterp3d_nuptsdriven(nf1, nf2, nf3, M, d_plan, blksize);
					if(ier != 0 ){
						cout<<"error: cnufftspread3d_gpu_nuptsdriven"<<endl;
						return 1;
					}
				}
			}
			break;
		case 2:
			{
				hipEventRecord(start);
				{
					PROFILE_CUDA_GROUP("Interpolation", 6);
					ier = cuinterp3d_subprob(nf1, nf2, nf3, M, d_plan, blksize);
					if(ier != 0 ){
						cout<<"error: cnufftspread3d_gpu_subprob"<<endl;
						return 1;
					}
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 1,2"<<endl;
			return 2;
	}
#ifdef SPREADTIME
	float milliseconds;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Interp " << milliseconds <<" ms"<<endl;
#endif
	return ier;
}


int cuinterp3d_nuptsdriven(int nf1, int nf2, int nf3, int M, cufinufft_plan *d_plan,
	int blksize)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=d_plan->spopts.nspread;   // psi's support in terms of number of cells
	FLT es_c=d_plan->spopts.ES_c;
	FLT es_beta=d_plan->spopts.ES_beta;
	FLT sigma=d_plan->spopts.upsampfac;
	int pirange=d_plan->spopts.pirange;

	int *d_idxnupts = d_plan->idxnupts;

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	FLT* d_kz = d_plan->kz;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;

	hipEventRecord(start);
	if(d_plan->opts.gpu_kerevalmeth){
#if 0
		hipStream_t *streams = d_plan->streams;
		int nstreams = d_plan->nstreams;
		for(int t=0; t<d_plan->maxbatchsize; t++){
			Interp_3d_NUptsdriven_Horner<<<blocks, threadsPerBlock, 0, 
				streams[t%nstreams]>>>(d_kx, d_ky, d_kz, d_c+t*M, 
				d_fw+t*nf1*nf2*nf3, M, ns, nf1, nf2, nf3, sigma);
		}
#else 
		for(int t=0; t<blksize; t++){
			Interp_3d_NUptsdriven_Horner<<<blocks, threadsPerBlock, 0, 
				0>>>(d_kx, d_ky, d_kz, d_c+t*M, 
				d_fw+t*nf1*nf2*nf3, M, ns, nf1, nf2, nf3, sigma, d_idxnupts,
				pirange);
		}
#endif
	}else{
#if 0
		hipStream_t *streams = d_plan->streams;
		int nstreams = d_plan->nstreams;
		for(int t=0; t<d_plan->maxbatchsize; t++){
			Interp_3d_NUptsdriven<<<blocks, threadsPerBlock, 0, streams[t%nstreams]
				>>>(d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, M, ns, 
				nf1, nf2, nf3,es_c, es_beta);
		}
#else
		for(int t=0; t<blksize; t++){
			Interp_3d_NUptsdriven<<<blocks, threadsPerBlock, 0, 0 
				>>>(d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, M, ns, 
				nf1, nf2, nf3,es_c, es_beta, d_idxnupts,pirange);
		}
#endif
	}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Interp_3d_NUptsdriven (%d) \t%.3g ms\n", 
		milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
	return 0;
}

int cuinterp3d_subprob(int nf1, int nf2, int nf3, int M, cufinufft_plan *d_plan,
	int blksize)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=d_plan->spopts.nspread;   // psi's support in terms of number of cells
	int maxsubprobsize=d_plan->opts.gpu_maxsubprobsize;

	// assume that bin_size_x > ns/2;
	int bin_size_x=d_plan->opts.gpu_binsizex;
	int bin_size_y=d_plan->opts.gpu_binsizey;
	int bin_size_z=d_plan->opts.gpu_binsizez;
	int numbins[3];
	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
	numbins[2] = ceil((FLT) nf3/bin_size_z);
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<d_plan->opts.gpu_binsizex<<"x"<<d_plan->opts.gpu_binsizey<<"x"<<d_plan->opts.gpu_binsizez<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"x"<<numbins[2]
	<<"]"<<endl;
#endif

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	FLT* d_kz = d_plan->kz;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;
	int *d_subprob_to_bin = d_plan->subprob_to_bin;
	int totalnumsubprob=d_plan->totalnumsubprob;

	FLT sigma=d_plan->spopts.upsampfac;
	FLT es_c=d_plan->spopts.ES_c;
	FLT es_beta=d_plan->spopts.ES_beta;
	int pirange=d_plan->spopts.pirange;
	hipEventRecord(start);
	size_t sharedplanorysize = (bin_size_x+2*ceil(ns/2.0))*
		(bin_size_y+2*ceil(ns/2.0))*(bin_size_z+2*ceil(ns/2.0))*sizeof(CUCPX);
	if(sharedplanorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}

	for(int t=0; t<blksize; t++){
		if(d_plan->opts.gpu_kerevalmeth==1){
			Interp_3d_Subprob_Horner<<<totalnumsubprob, 256,
				sharedplanorysize>>>(d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, 
				M, ns, nf1, nf2, nf3, sigma, d_binstartpts, d_binsize, bin_size_x,
				bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
				d_numsubprob, maxsubprobsize,numbins[0], numbins[1], numbins[2],
				d_idxnupts,pirange);
		}else{
			Interp_3d_Subprob<<<totalnumsubprob, 256,
				sharedplanorysize>>>(d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, 
				M, ns, nf1, nf2, nf3, es_c, es_beta, d_binstartpts, d_binsize, 
				bin_size_x, bin_size_y, bin_size_z, d_subprob_to_bin, 
				d_subprobstartpts, d_numsubprob, maxsubprobsize,numbins[0], 
				numbins[1], numbins[2],d_idxnupts,pirange);
		}
	}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Interp_3d_Subprob (%d) \t%.3g ms\n", milliseconds,
	d_plan->opts.gpu_kerevalmeth);
#endif
	return 0;
}
