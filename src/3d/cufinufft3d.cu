#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>

#include <cufinufft.h>
#include "../cuspreadinterp.h"
#include "../cudeconvolve.h"
#include "../memtransfer.h"

using namespace std;

int cufinufft3d1_exec(CUCPX* d_c, CUCPX* d_fk, cufinufft_plan *d_plan)
/*  
	3D Type-1 NUFFT

	This function is called in "exec" stage (See ../cufinufft.cu).
	It includes (copied from doc in finufft library)
		Step 1: spread data to oversampled regular mesh using kernel
		Step 2: compute FFT on uniform mesh
		Step 3: deconvolve by division of each Fourier mode independently by the
		        Fourier series coefficient of the kernel.

	Melody Shih 07/25/19		
*/
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	int blksize; 
	int ier;
	CUCPX* d_fkstart;
	CUCPX* d_cstart;
	for(int i=0; i*d_plan->maxbatchsize < d_plan->ntransf; i++){
		blksize = min(d_plan->ntransf - i*d_plan->maxbatchsize, 
			d_plan->maxbatchsize);
		d_cstart = d_c + i*d_plan->maxbatchsize*d_plan->M;
		d_fkstart = d_fk + i*d_plan->maxbatchsize*d_plan->ms*d_plan->mt*
			d_plan->mu;

		d_plan->c = d_cstart;
		d_plan->fk = d_fkstart;

		checkCudaErrors(hipMemset(d_plan->fw,0,d_plan->maxbatchsize*
					d_plan->nf1*d_plan->nf2*d_plan->nf3*sizeof(CUCPX)));
#ifdef TIME
		float milliseconds = 0;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tInitialize fw\t\t %.3g s\n", milliseconds/1000);
#endif
		// Step 1: Spread
		hipEventRecord(start);
		ier = cuspread3d(d_plan, blksize);
		if(ier != 0 ){
			printf("error: cuspread3d, method(%d)\n", d_plan->opts.gpu_method);
			return 0;
		}
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tSpread (%d)\t\t %.3g s\n", milliseconds/1000, 
			d_plan->opts.gpu_method);
#endif
		// Step 2: FFT
		hipEventRecord(start);
		CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds/1000);
#endif

		// Step 3: deconvolve and shuffle
		hipEventRecord(start);
		cudeconvolve3d(d_plan, blksize);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tDeconvolve\t\t %.3g s\n", milliseconds/1000);
#endif
	}
	return ier;
}

int cufinufft3d2_exec(CUCPX* d_c, CUCPX* d_fk, cufinufft_plan *d_plan)
/*  
	3D Type-2 NUFFT

	This function is called in "exec" stage (See ../cufinufft.cu).
	It includes (copied from doc in finufft library)
		Step 1: deconvolve (amplify) each Fourier mode, dividing by kernel 
		        Fourier coeff
		Step 2: compute FFT on uniform mesh
		Step 3: interpolate data to regular mesh

	Melody Shih 07/25/19		
*/
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int blksize;
	int ier;
	CUCPX* d_fkstart;
	CUCPX* d_cstart;
	for(int i=0; i*d_plan->maxbatchsize < d_plan->ntransf; i++){
		blksize = min(d_plan->ntransf - i*d_plan->maxbatchsize, 
			d_plan->maxbatchsize);
		d_cstart  = d_c  + i*d_plan->maxbatchsize*d_plan->M;
		d_fkstart = d_fk + i*d_plan->maxbatchsize*d_plan->ms*d_plan->mt*
			d_plan->mu;

		d_plan->c = d_cstart;
		d_plan->fk = d_fkstart;

		// Step 1: amplify Fourier coeffs fk and copy into upsampled array fw
		hipEventRecord(start);
		cudeconvolve3d(d_plan, blksize);
#ifdef TIME
		float milliseconds = 0;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tAmplify & Copy fktofw\t %.3g s\n", milliseconds/1000);
#endif
		// Step 2: FFT
		hipEventRecord(start);
		hipDeviceSynchronize();
		CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds/1000);
#endif

		// Step 3: deconvolve and shuffle
		hipEventRecord(start);
		ier = cuinterp3d(d_plan, blksize);
		if(ier != 0 ){
			printf("error: cuinterp3d, method(%d)\n", d_plan->opts.gpu_method);
			return 0;
		}
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tUnspread (%d)\t\t %.3g s\n", milliseconds/1000,
			d_plan->opts.gpu_method);
#endif

		hipEventRecord(start);
#if 0
		if(d_plan->nstreams != 1)
			hipDeviceSynchronize();
#endif
	}
	return ier;
}

#if 0
int cufinufft3d_plan(int M, int ms, int mt, int mu, int ntransf, 
	int maxbatchsize, int iflag, const cufinufft_opts opts, 
	cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	//ier=cufinufft_default_opts(opts,eps,upsampfac);
	int nf1 = (int) d_plan->opts.gpu_upsampfac*ms;
	int nf2 = (int) d_plan->opts.gpu_upsampfac*mt;
	int nf3 = (int) d_plan->opts.gpu_upsampfac*mu;
	int fftsign = (iflag>=0) ? 1 : -1;

	d_plan->ms = ms;
	d_plan->mt = mt;
	d_plan->mu = mu;
	d_plan->nf1 = nf1;
	d_plan->nf2 = nf2;
	d_plan->nf3 = nf3;
	d_plan->M = M;
	d_plan->iflag = fftsign;
	d_plan->ntransf = ntransf;
	d_plan->maxbatchsize = maxbatchsize;
#ifdef INFO
	printf("[info  ] 3d: (ms,mt,mu)=(%d,%d) (nf1, nf2, nf3)=(%d,%d,%d) nj=%d, ntransform = %d\n",
		ms, mt, mu, d_plan->nf1, d_plan->nf2, d_plan->nf3, d_plan->M, 
		d_plan->ntransf);
#endif

	// this may move to gpu
	CNTime timer; timer.start();
	FLT *fwkerhalf1 = (FLT*)malloc(sizeof(FLT)*(nf1/2+1));
	FLT *fwkerhalf2 = (FLT*)malloc(sizeof(FLT)*(nf2/2+1));
	FLT *fwkerhalf3 = (FLT*)malloc(sizeof(FLT)*(nf3/2+1));
	onedim_fseries_kernel(nf1, fwkerhalf1, opts);
	onedim_fseries_kernel(nf2, fwkerhalf2, opts);
	onedim_fseries_kernel(nf3, fwkerhalf3, opts);
#ifdef DEBUG
	printf("[time  ] \tkernel fser (ns=%d):\t %.3g s\n", d_plan->opts.gpu_nspread,timer.elapsedsec());
#endif

	hipEventRecord(start);
	ier = allocgpumemory3d(opts, d_plan);
#ifdef DEBUG
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocate GPU memory\t %.3g s\n", milliseconds/1000);
#endif

	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(d_plan->fwkerhalf1,fwkerhalf1,(nf1/2+1)*
		sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->fwkerhalf2,fwkerhalf2,(nf2/2+1)*
		sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->fwkerhalf3,fwkerhalf3,(nf3/2+1)*
		sizeof(FLT),hipMemcpyHostToDevice));
#ifdef DEBUG
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tCopy fwkerhalf1,2,3 HtoD %.3g s\n", milliseconds/1000);
#endif

	hipEventRecord(start);
	hipfftHandle fftplan;
	int dim = 3;
	int n[] = {nf3, nf2, nf1};
	int inembed[] = {nf3, nf2, nf1};
	int istride = 1;
	hipfftPlanMany(&fftplan,dim,n,inembed,istride,inembed[0]*inembed[1]*inembed[2],
		inembed,istride,inembed[0]*inembed[1]*inembed[2],CUFFT_TYPE,
		maxbatchsize);
	d_plan->fftplan = fftplan;
#ifdef DEBUG
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tCUFFT Plan\t\t %.3g s\n", milliseconds/1000);
#endif
	return ier;
}
int cufinufft3d_setNUpts(FLT* h_kx, FLT* h_ky, FLT *h_kz, cufinufft_opts &opts, cufinufft_plan *d_plan)
{
	int M = d_plan->M;
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int nf3 = d_plan->nf3;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Copy memory to device
	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(d_plan->kx,h_kx,d_plan->M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->ky,h_ky,d_plan->M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->kz,h_kz,d_plan->M*sizeof(FLT),hipMemcpyHostToDevice));
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tCopy kx,ky,kz HtoD\t %.3g s\n", milliseconds/1000);
#endif

	if(d_plan->opts.gpu_pirange==1){
		hipEventRecord(start);
		RescaleXY_3d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,nf3,d_plan->kx,
			d_plan->ky,d_plan->kz);
		d_plan->opts.gpu_pirange=0;
#ifdef SPREADTIME
		float milliseconds;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ]\tRescaleXY_3d\t\t %.3g ms\n", milliseconds);
#endif
	}

	hipEventRecord(start);
	if(d_plan->opts.gpu_method == 5){
		int ier = cuspread3d_subprob_prop(nf1,nf2,nf3,M,opts,d_plan);
		if(ier != 0 ){
			printf("error: cuspread3d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
			return 0;
		}
	}
	if(d_plan->opts.gpu_method == 1 || d_plan->opts.gpu_method ==  2 || d_plan->opts.gpu_method == 3){
		int ier = cuspread3d_blockgather_prop(nf1,nf2,nf3,M,opts,d_plan);
		if(ier != 0 ){
			printf("error: cuspread3d_blockgather_prop, method(%d)\n", d_plan->opts.gpu_method);
			return 0;
		}
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tSetup Subprob properties %.3g s\n", 
		milliseconds/1000);
#endif
	return 0;
}
#endif
